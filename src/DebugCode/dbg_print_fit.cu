#include "hip/hip_runtime.h"

extern "C" {
#include "../shape/head.h"
}
__device__ int dbg_ndop1, dbg_ndel1, dbg_xlim0, dbg_xlim1, dbg_ylim0, dbg_ylim1;

__global__ void dbg_print_fit_krnl1(struct dat_t *ddat, int s, int f){
	/* Single-threaded debug kernel */
	if (threadIdx.x == 0) {
		switch (ddat->set[s].type) {
		case DELAY:
			dbg_ndel1 = ddat->set[s].desc.deldop.frame[f].ndel;
			dbg_ndop1 = ddat->set[s].desc.deldop.frame[f].ndop;
			dbg_xlim0 = ddat->set[s].desc.deldop.frame[f].pos.xlim[0];
			dbg_xlim1 = ddat->set[s].desc.deldop.frame[f].pos.xlim[1];
			dbg_ylim0 = ddat->set[s].desc.deldop.frame[f].pos.ylim[0];
			dbg_ylim1 = ddat->set[s].desc.deldop.frame[f].pos.ylim[1];
			break;
		case DOPPLER:
			dbg_ndop1 = ddat->set[s].desc.doppler.frame[f].ndop;
			dbg_xlim0 = ddat->set[s].desc.doppler.frame[f].pos.xlim[0];
			dbg_xlim1 = ddat->set[s].desc.doppler.frame[f].pos.xlim[1];
			dbg_ylim0 = ddat->set[s].desc.doppler.frame[f].pos.ylim[0];
			dbg_ylim1 = ddat->set[s].desc.doppler.frame[f].pos.ylim[1];
			break;
		}
	}
}
__global__ void dbg_print_fit_krnl2(struct dat_t *ddat, float *fit, int s, int f) {
	/* ndop-threaded kernel */
	int idop = blockIdx.x * blockDim.x + threadIdx.x;// +1 ;

	if (idop < dbg_ndop1) {
		fit[idop] = ddat->set[s].desc.doppler.frame[f].fit_s[idop];
		printf("fit_s[%i]=%g\n", idop, ddat->set[s].desc.doppler.frame[f].fit_s[idop]);
	}
}
__global__ void dbg_print_lc_fit_krnl(struct dat_t *ddat, double *fit, int s, int n) {
	/* ndop-threaded kernel */
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (i <= n) {
		fit[i] = ddat->set[s].desc.lghtcrv.fit[i];
	}
}
__global__ void dbg_print_fit_deldop_krnl2_32(struct dat_t *ddat, float *fit, int s, int f){
	/* ndel*ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < (dbg_ndop1*dbg_ndel1))
		fit[offset] = ddat->set[s].desc.deldop.frame[f].fit_s[offset];
}
__global__ void dbg_print_fit_deldop_krnl2_64(struct dat_t *ddat, double *fit, int s, int f){
	/* ndel*ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int idel = offset % ddat->set[s].desc.deldop.frame[f].ndel + 1;
	int idop = offset / ddat->set[s].desc.deldop.frame[f].ndel + 1;

	if (offset < (dbg_ndop1*dbg_ndel1))
		fit[offset] = ddat->set[s].desc.deldop.frame[f].fit[idel][idop];
}

__host__ void dbg_print_fit(struct dat_t *ddat, int s, int f, const char
		*filename_fit, int gpuid) {
	/* Debug function that prints all Doppler frame fit values to csv */

	int idop, /*nThreads*/ndop, xlim[2], ylim[2];
	FILE *fp_fit;
	float *fit, *host_fit;
	dim3 BLK,THD;
	//gpuid = GPU0;
	hipSetDevice(gpuid);
	printf("\n %sfile created",filename_fit);


	/* Launch 1st debug kernel to get ndop and xlim/ylim	 */
	dbg_print_fit_krnl1<<<1,1>>>(ddat, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_krnl1");
	deviceSyncAfterKernelLaunch("dbg_print_fit_krnl2");
	gpuErrchk(hipMemcpyFromSymbol(&xlim[0], HIP_SYMBOL(dbg_xlim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&xlim[1], HIP_SYMBOL(dbg_xlim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[0], HIP_SYMBOL(dbg_ylim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[1], HIP_SYMBOL(dbg_ylim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(dbg_ndop1), sizeof(int),
			0, hipMemcpyDeviceToHost));

	//nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	cudaCalloc((void**)&fit, sizeof(float), ndop);
//	host_fit = (float *)malloc(sizeof(float) * ndop);
	int maxThreads = 128;
	BLK.x = floor((maxThreads - 1 + ndop)/maxThreads);
	THD.x = maxThreads; // Thread block dimensions

	dbg_print_fit_krnl2<<<BLK,THD>>>(ddat, fit, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_krnl_2");
	deviceSyncAfterKernelLaunch("dbg_print_fit_krnl_2");
//	gpuErrchk(hipMemcpy(&host_fit, fit, sizeof(float)*ndop,
//			hipMemcpyDeviceToHost));

//	fp_fit = fopen(filename_fit, "w+");
//	fprintf(fp_fit, "idop , ");
//	for (idop=0; idop<ndop; idop++)
//		fprintf(fp_fit,	"\n%i , %g", idop, fit[idop]);
//	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
//	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
//	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
//	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
//	fprintf(fp_fit, "\nthreads , %i", nThreads);
//	fclose(fp_fit);
	hipFree(fit);
//	free(host_fit);
}

__host__ void dbg_print_fit_host(struct dat_t *ddat, int s, int f, const char *filename_fit) {
	/* Debug function that prints all Doppler frame fit values to csv */

	int idop, nThreads;
	FILE *fp_fit;
	nThreads = (ddat->set[s].desc.doppler.frame[f].pos.xlim[1]-
			ddat->set[s].desc.doppler.frame[f].pos.xlim[0]+1)*
					(ddat->set[s].desc.doppler.frame[f].pos.ylim[1]-
							ddat->set[s].desc.doppler.frame[f].pos.ylim[0]+1);

	printf("\n %sfile created",filename_fit);
	fp_fit = fopen(filename_fit, "w+");

	fprintf(fp_fit, "idel , ");

	for (idop=1; idop<=ddat->set[s].desc.doppler.frame[f].ndop; idop++)
		fprintf(fp_fit,	"\n%i , %g", idop, ddat->set[s].desc.doppler.frame[f].fit[idop]);

	fprintf(fp_fit, "\nxlim0 , %i", ddat->set[s].desc.doppler.frame[f].pos.xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", ddat->set[s].desc.doppler.frame[f].pos.xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ddat->set[s].desc.doppler.frame[f].pos.ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ddat->set[s].desc.doppler.frame[f].pos.ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
}

__host__ void dbg_print_deldop_fit(struct dat_t *ddat, int s, int f, const char *filename_fit) {
	/* Debug function that prints all Doppler frame fit values to csv */

	int idop, ndop, idel, ndel, nbins, nThreads, offset, xlim[2], ylim[2];
	FILE *fp_fit;
	float *fit_dd32, *host_fit32;
	double *fit_dd64, *host_fit64;

	dim3 BLK,THD;
	printf("\n %sfile created",filename_fit);

	/* Launch 1st debug kernel to get ndop and xlim/ylim	 */
	dbg_print_fit_krnl1<<<1,1>>>(ddat, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_krnl1");
	//deviceSyncAfterKernelLaunch("dbg_print_fit_krnl2");
	gpuErrchk(hipMemcpyFromSymbol(&xlim[0], HIP_SYMBOL(dbg_xlim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&xlim[1], HIP_SYMBOL(dbg_xlim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[0], HIP_SYMBOL(dbg_ylim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[1], HIP_SYMBOL(dbg_ylim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(dbg_ndop1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ndel, HIP_SYMBOL(dbg_ndel1), sizeof(int),
				0, hipMemcpyDeviceToHost));

	nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	nbins = ndop * ndel;
	if (FP64) {
		cudaCalloc((void**)&fit_dd64, sizeof(double), nbins);
		host_fit64 = (double *)malloc(sizeof(double) * nbins);
	} else {
		cudaCalloc((void**)&fit_dd32, sizeof(float), nbins);
		host_fit32 = (float *)malloc(sizeof(float) * nbins);
	}
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nbins)/THD.x);

	if (FP64) {
		dbg_print_fit_deldop_krnl2_64<<<BLK,THD>>>(ddat, fit_dd64, s, f);
		checkErrorAfterKernelLaunch("dbg_print_fit_deldop_krnl_2_64");
		hipMemcpy(host_fit64, fit_dd64, sizeof(double)*nbins, hipMemcpyDeviceToHost);
	} else {
		dbg_print_fit_deldop_krnl2_32<<<BLK,THD>>>(ddat, fit_dd32, s, f);
		checkErrorAfterKernelLaunch("dbg_print_fit_deldop_krnl_2_32");
		hipMemcpy(host_fit32, fit_dd32, sizeof(float)*nbins, hipMemcpyDeviceToHost);
	}

	fp_fit = fopen(filename_fit, "w+");

	/* Print top corner idop/idel label */
	fprintf(fp_fit, "idop/idel , ");

	/* Print top row idel values */
	for (idel=1; idel<=ndel; idel++)
		fprintf(fp_fit, "%i , ", idel);

	/* Print first entry in every row (except 1st): idop */
	for (idop=1; idop<=ndop; idop++) {
		fprintf(fp_fit,	"\n%i , ", idop);

		/* Write the rest of the row values: fit[idel][idop] */
		for (idel=1; idel<=ndel; idel++) {
			offset = (idop-1)*ndel + (idel-1);
			if (FP64)
				fprintf(fp_fit, " %g , ", host_fit64[offset]);
			else
				fprintf(fp_fit, " %g , ", host_fit32[offset]);//fit_dd[offset]);
		}
	}
	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);

}

__host__ void dbg_print_deldop_fit_host(struct dat_t *ddat, int s, int f, const char *filename_fit) {
	/* Debug function that prints all Delay-Doppler frame fit values to csv */

	int idop, ndop, idel, ndel, nThreads, xlim[2], ylim[2];
	FILE *fp_fit;
	printf("\n %sfile created",filename_fit);

	for (idop=0;idop<2;idop++){
		xlim[idop] = ddat->set[s].desc.deldop.frame[f].pos.xlim[idop];
		ylim[idop] = ddat->set[s].desc.deldop.frame[f].pos.ylim[idop];}

	ndel = ddat->set[s].desc.deldop.frame[f].ndel;
	ndop = ddat->set[s].desc.deldop.frame[f].ndop;
	nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	fp_fit = fopen(filename_fit, "w+");

	/* Print top corner idop/idel label */
	fprintf(fp_fit, "idop/idel , ");

	/* Print top row idel values */
	for (idel=1; idel<=ndel; idel++)
		fprintf(fp_fit, "%i , ", idel);

	/* Print first entry in every row (except 1st): idop */
	for (idop=1; idop<=ndop; idop++) {
		fprintf(fp_fit,	"\n%i , ", idop);

		/* Write the rest of the row values: fit[idel][idop] */
		for (idel=1; idel<=ndel; idel++)
			fprintf(fp_fit, " %g , ", ddat->set[s].desc.deldop.frame[f].fit[idel][idop]);
	}
	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
}

__host__ void dbg_print_deldop_fit_host2(struct deldopfrm_t *frame, const char *filename_fit) {
	/* Debug function that prints all Delay-Doppler frame fit values to csv */

	int idop, ndop, idel, ndel, nThreads, xlim[2], ylim[2];
	FILE *fp_fit;
	printf("\n %sfile created",filename_fit);

	for (idop=0;idop<2;idop++){
		xlim[idop] = frame->pos.xlim[idop];
		ylim[idop] = frame->pos.ylim[idop];}

	ndel = frame->ndel;
	ndop = frame->ndop;
	nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	fp_fit = fopen(filename_fit, "w+");

	/* Print top corner idop/idel label */
	fprintf(fp_fit, "idop/idel , ");

	/* Print top row idel values */
	for (idel=1; idel<=ndel; idel++)
		fprintf(fp_fit, "%i , ", idel);

	/* Print first entry in every row (except 1st): idop */
	for (idop=1; idop<=ndop; idop++) {
		fprintf(fp_fit,	"\n%i , ", idop);

		/* Write the rest of the row values: fit[idel][idop] */
		for (idel=1; idel<=ndel; idel++)
			fprintf(fp_fit, " %g , ", frame->fit[idel][idop]);
	}
	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
}

__host__ void dbg_print_lc_fit(struct dat_t *ddat, int s, const char *filename_fit, int n) {
	/* Debug function that prints lightcurve fit values */

	int i;
	FILE *fp_fit;
	double *fit;
	dim3 BLK,THD;

	cudaCalloc((void**)&fit, sizeof(double), n);
	fit -= 1;
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + n)/THD.x);

	dbg_print_lc_fit_krnl<<<BLK,THD>>>(ddat, fit, s, n);
	checkErrorAfterKernelLaunch("dbg_print_lc_fit_krnl");
	deviceSyncAfterKernelLaunch("dbg_print_lc_fit_krnl");

	fp_fit = fopen(filename_fit, "w+");
	fprintf(fp_fit, "i , ");
	for (i=1; i<=n; i++)
		fprintf(fp_fit,	"\n%i , %g", i, fit[i]);
	fclose(fp_fit);
	//hipFree(fit);
}

__host__ void dbg_print_lc_fit_host(struct lghtcrv_t *lghtcrv, const char *filename_fit, int n) {
	/* Debug function that prints light curve fit values (host version) */

	int i;
	FILE *fp_fit;
	dim3 BLK,THD;
	fp_fit = fopen(filename_fit, "w+");
	fprintf(fp_fit, "i , ");
	for (i=1; i<=n; i++)
		fprintf(fp_fit,	"\n%i , %g", i, lghtcrv->fit[i]);
	fclose(fp_fit);
}
