#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                           realize_spin.c

Takes the initial spin state described in mod, computes the spin state at the epoch of
each data frame, and produces the various coordinate transformation matrices needed in
dat.  Also computes the total apparent spin vector at the epoch of each data frame.

Modified 2015 June 3 by CM:
    Implement smearing for the "fit" and "write" actions

Modified 2011 August 15 by CM:
    Determine which spin impulses must be applied to each frame or
        lightcurve point
    Pass the "int_abstol" parameter to the inteuler routine

Modified 2006 June 18 by CM:
    Eliminate range datasets

Modified 2005 January 20 by CM:
    For POS and range datasets, save the intrisic spin vector and total
        (intrinsic plus orbital) spin vector

Modified 2004 March 22 by CM:
    For lightcurve points, save the intrisic spin vector and total
        (intrinsic plus orbital) spin vector

Modified 2004 Feb 5 by CM:
    Implement "=" state for angle and spin offsets by creating
    routines realize_angleoff and realize_omegaoff

Modified 2003 May 4 by CM:
    Apply angle offsets to Doppler datasets, not just delay-Doppler
 *****************************************************************************************/
extern "C" {
#include "../shape/head.h"
}

__global__ void add_offsets_to_euler_krnl(struct mod_t *dmod,
		struct dat_t *ddat, double3 *angle_omega_save, int s)
{
	/* Single-threaded kernel */
	/*	angle_omega_save[0].x,y,z = original anglesave[3]
	 * 	angle_omega_save[1].x,y,z = original omegasave[3]
	 * 		 */

	if (threadIdx.x == 0) {

		angle_omega_save[0].x = dmod->spin.angle[0].val;
		angle_omega_save[0].y = dmod->spin.angle[1].val;
		angle_omega_save[0].z = dmod->spin.angle[2].val;
		angle_omega_save[1].x = dmod->spin.omega[0].val;
		angle_omega_save[1].y = dmod->spin.omega[1].val;
		angle_omega_save[1].z = dmod->spin.omega[2].val;
//		Original code:
//		anglesave[j] = dmod->spin.angle[j].val;
//		omegasave[j] = dmod->spin.omega[j].val;
		for (int j=0; j<=2; j++)
			dmod->spin.angle[j].val += ddat->set[s].angleoff[j].val;
	}
}
__global__ void add_offsets_to_euler_f_krnl(struct mod_t *dmod,
		struct dat_t *ddat, float3 *angle_omega_save, int s)
{
	/* Single-threaded kernel */
	/*	angle_omega_save[0].x,y,z = original anglesave[3]
	 * 	angle_omega_save[1].x,y,z = original omegasave[3]
	 * 		 */

	if (threadIdx.x == 0) {

		angle_omega_save[0].x = __double2float_rn(dmod->spin.angle[0].val);
		angle_omega_save[0].y = __double2float_rn(dmod->spin.angle[1].val);
		angle_omega_save[0].z = __double2float_rn(dmod->spin.angle[2].val);
		angle_omega_save[1].x = __double2float_rn(dmod->spin.omega[0].val);
		angle_omega_save[1].y = __double2float_rn(dmod->spin.omega[1].val);
		angle_omega_save[1].z = __double2float_rn(dmod->spin.omega[2].val);
//		Original code:
//		anglesave[j] = dmod->spin.angle[j].val;
//		omegasave[j] = dmod->spin.omega[j].val;
		for (int j=0; j<=2; j++)
			dmod->spin.angle[j].val += ddat->set[s].angleoff[j].val;
	}
}
__global__ void realize_spin_dop_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nviews, int s, int f)
{
	/* nview-threaded kernel */
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (k < nviews) {

		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.doppler.frame[f].view[k].t,
				ddat->set[s].desc.doppler.frame[f].t_integrate,
				ddat->set[s].desc.doppler.frame[f].impulse,
				&ddat->set[s].desc.doppler.frame[f].n_integrate,s,f,k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.doppler.frame[f].t_integrate,
				ddat->set[s].desc.doppler.frame[f].impulse,
				ddat->set[s].desc.doppler.frame[f].n_integrate,
				ddat->set[s].desc.doppler.frame[f].view[k].intspin,
				ddat->set[s].desc.doppler.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.doppler.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.doppler.frame[f].view[k].intspin,
				ddat->set[s].desc.doppler.frame[f].view[k].ae,
				ddat->set[s].desc.doppler.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.doppler.frame[f].view[k].spin[j] = ddat->set[s].desc.doppler.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.doppler.frame[f].view[k].intspin[j];
	}
}
__global__ void realize_spin_dop_f_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nframes, int s, int k)
{
	/* nview-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (f < nframes) {

		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.doppler.frame[f].view[k].t,
				ddat->set[s].desc.doppler.frame[f].t_integrate,
				ddat->set[s].desc.doppler.frame[f].impulse,
				&ddat->set[s].desc.doppler.frame[f].n_integrate,s,f,k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.doppler.frame[f].t_integrate,
				ddat->set[s].desc.doppler.frame[f].impulse,
				ddat->set[s].desc.doppler.frame[f].n_integrate,
				ddat->set[s].desc.doppler.frame[f].view[k].intspin,
				ddat->set[s].desc.doppler.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.doppler.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.doppler.frame[f].view[k].intspin,
				ddat->set[s].desc.doppler.frame[f].view[k].ae,
				ddat->set[s].desc.doppler.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.doppler.frame[f].view[k].spin[j] = ddat->set[s].desc.doppler.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.doppler.frame[f].view[k].intspin[j];
	}
}
__global__ void realize_spin_deldop_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nviews, int s, int f)
{
	/* nview-threaded kernel */
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (k < nviews) {
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.deldop.frame[f].view[k].t,
				ddat->set[s].desc.deldop.frame[f].t_integrate,
				ddat->set[s].desc.deldop.frame[f].impulse,
				&ddat->set[s].desc.deldop.frame[f].n_integrate,
				s, f, k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.deldop.frame[f].t_integrate,
				ddat->set[s].desc.deldop.frame[f].impulse,
				ddat->set[s].desc.deldop.frame[f].n_integrate,
				ddat->set[s].desc.deldop.frame[f].view[k].intspin,
				ddat->set[s].desc.deldop.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.deldop.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.deldop.frame[f].view[k].intspin,
				ddat->set[s].desc.deldop.frame[f].view[k].ae,
				ddat->set[s].desc.deldop.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.deldop.frame[f].view[k].spin[j] = ddat->set[s].desc.deldop.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.deldop.frame[f].view[k].intspin[j];
	}
}
__global__ void realize_spin_deldop_f_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nframes, int s, int k)
{
	/* nview-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (f < nframes) {
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.deldop.frame[f].view[k].t,
				ddat->set[s].desc.deldop.frame[f].t_integrate,
				ddat->set[s].desc.deldop.frame[f].impulse,
				&ddat->set[s].desc.deldop.frame[f].n_integrate,
				s, f, k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.deldop.frame[f].t_integrate,
				ddat->set[s].desc.deldop.frame[f].impulse,
				ddat->set[s].desc.deldop.frame[f].n_integrate,
				ddat->set[s].desc.deldop.frame[f].view[k].intspin,
				ddat->set[s].desc.deldop.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.deldop.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.deldop.frame[f].view[k].intspin,
				ddat->set[s].desc.deldop.frame[f].view[k].ae,
				ddat->set[s].desc.deldop.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.deldop.frame[f].view[k].spin[j] = ddat->set[s].desc.deldop.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.deldop.frame[f].view[k].intspin[j];
	}
}
__global__ void realize_spin_poset_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nviews, int s, int f)
{
	/* nview-threaded kernel */
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (k < nviews)
	{
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.poset.frame[f].view[k].t,
				ddat->set[s].desc.poset.frame[f].t_integrate,
				ddat->set[s].desc.poset.frame[f].impulse,
				&ddat->set[s].desc.poset.frame[f].n_integrate,s,f,k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.poset.frame[f].t_integrate,
				ddat->set[s].desc.poset.frame[f].impulse,
				ddat->set[s].desc.poset.frame[f].n_integrate,
				ddat->set[s].desc.poset.frame[f].view[k].intspin,
				ddat->set[s].desc.poset.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.poset.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.poset.frame[f].view[k].intspin,
				ddat->set[s].desc.poset.frame[f].view[k].ae,
				ddat->set[s].desc.poset.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.poset.frame[f].view[k].spin[j] = ddat->set[s].desc.poset.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.poset.frame[f].view[k].intspin[j];
	}
}
__global__ void realize_spin_poset_f_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nframes, int s, int k)
{
	/* nview-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	if (f <= nframes)
	{
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.poset.frame[f].view[k].t,
				ddat->set[s].desc.poset.frame[f].t_integrate,
				ddat->set[s].desc.poset.frame[f].impulse,
				&ddat->set[s].desc.poset.frame[f].n_integrate,s,f,k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.poset.frame[f].t_integrate,
				ddat->set[s].desc.poset.frame[f].impulse,
				ddat->set[s].desc.poset.frame[f].n_integrate,
				ddat->set[s].desc.poset.frame[f].view[k].intspin,
				ddat->set[s].desc.poset.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.poset.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.poset.frame[f].view[k].intspin,
				ddat->set[s].desc.poset.frame[f].view[k].ae,
				ddat->set[s].desc.poset.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.poset.frame[f].view[k].spin[j] = ddat->set[s].desc.poset.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.poset.frame[f].view[k].intspin[j];
	}
}
__global__ void realize_spin_lghtcrv_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int s, int i)
{
	/* Single-threaded kernel */
	int j;
	i = i+1;	/* This fixes the offset for rend[i] */
	if (threadIdx.x == 0)
	{
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.lghtcrv.x[i],
				ddat->set[s].desc.lghtcrv.rend[i].t_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].impulse,
				&ddat->set[s].desc.lghtcrv.rend[i].n_integrate,
				s,i,0);	// s = s,  f = i, k = 0

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.lghtcrv.rend[i].t_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].impulse,
				ddat->set[s].desc.lghtcrv.rend[i].n_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].intspin,
				ddat->set[s].desc.lghtcrv.rend[i].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.lghtcrv.rend[i].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.lghtcrv.rend[i].intspin,
				ddat->set[s].desc.lghtcrv.rend[i].ae,
				ddat->set[s].desc.lghtcrv.rend[i].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.lghtcrv.rend[i].spin[j] = ddat->set[s].desc.lghtcrv.rend[i].orbspin[j] +
			ddat->set[s].desc.lghtcrv.rend[i].intspin[j];
	}
}
__global__ void realize_spin_lghtcrv_f_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int s, int size)
{
	/* ncalc-threaded kernel */
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j;
	if (i <= size)
	{
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.lghtcrv.x[i],
				ddat->set[s].desc.lghtcrv.rend[i].t_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].impulse,
				&ddat->set[s].desc.lghtcrv.rend[i].n_integrate,
				s,i,0);	// s = s,  f = i, k = 0

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.lghtcrv.rend[i].t_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].impulse,
				ddat->set[s].desc.lghtcrv.rend[i].n_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].intspin,
				ddat->set[s].desc.lghtcrv.rend[i].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.lghtcrv.rend[i].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.lghtcrv.rend[i].intspin,
				ddat->set[s].desc.lghtcrv.rend[i].ae,
				ddat->set[s].desc.lghtcrv.rend[i].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.lghtcrv.rend[i].spin[j] = ddat->set[s].desc.lghtcrv.rend[i].orbspin[j] +
			ddat->set[s].desc.lghtcrv.rend[i].intspin[j];
	}
}
/* Determine which spin impulses will be encountered in evolving the spin state
 * from initial spin epoch t0 to epoch t of a particular frame or lightcurve
 * point; then create lists of epochs and impulses, starting at t0 and ending
 * at t, with the impulses negated if we're evolving backwards in time.     */

__device__ void dev_realize_impulse(struct spin_t spin, double t,
		double t_integrate[], double impulse[][3], int *n_integrate, int s, int f, int k)
{
	int j, n;
	k = 0;
	t_integrate[k] = spin.t0;
	for (j=0; j<=2; j++)
		impulse[k][j] = 0.0;
	if (t >= spin.t0) {

		/* Integrating forward in time, so add the spin impulses  */
		for (n=0; n<spin.n_impulse; n++) {
			if (spin.t_impulse[n] > spin.t0 && spin.t_impulse[n] <= t) {
				k++;
				t_integrate[k] = spin.t_impulse[n];
				for (j=0; j<=2; j++)
					impulse[k][j] = spin.impulse[n][j].val;
			}
		}
		if (t_integrate[k] < t) {
			k++;
			t_integrate[k] = t;
			for (j=0; j<=2; j++)
				impulse[k][j] = 0.0;
		}
	} else {

		/* Integrating backwards in time, so subtract the spin impulses  */
		for (n=spin.n_impulse-1; n>=0; n--) {
			if (spin.t_impulse[n] < spin.t0 && spin.t_impulse[n] >= t) {
				k++;
				t_integrate[k] = spin.t_impulse[n];
				for (j=0; j<=2; j++)
					impulse[k][j] = -spin.impulse[n][j].val;
			}
		}
		if (t_integrate[k] > t) {
			k++;
			t_integrate[k] = t;
			for (j=0; j<=2; j++)
				impulse[k][j] = 0.0;
		}
	}
	*n_integrate = k + 1;
}
__global__ void update_spin_angle_krnl(struct mod_t *dmod,
		double3 *angle_omega_save)
{
	/* Single-threaded kernel */
	/*	angle_omega_save[0].x,y,z = original anglesave[3]
	 * 	angle_omega_save[1].x,y,z = original omegasave[3]
	 * 		 */
	if(threadIdx.x == 0) {
		dmod->spin.angle[0].val = angle_omega_save[0].x;
		dmod->spin.angle[1].val = angle_omega_save[0].y;
		dmod->spin.angle[2].val = angle_omega_save[0].z;
		dmod->spin.omega[0].val = angle_omega_save[1].x;
		dmod->spin.omega[1].val = angle_omega_save[1].y;
		dmod->spin.omega[2].val = angle_omega_save[1].z;
	}
}
__global__ void update_spin_angle_f_krnl(struct mod_t *dmod,
		float3 *angle_omega_save)
{
	/* Single-threaded kernel */
	/*	angle_omega_save[0].x,y,z = original anglesave[3]
	 * 	angle_omega_save[1].x,y,z = original omegasave[3]
	 * 		 */
	if(threadIdx.x == 0) {
		dmod->spin.angle[0].val = (double)angle_omega_save[0].x;
		dmod->spin.angle[1].val = (double)angle_omega_save[0].y;
		dmod->spin.angle[2].val = (double)angle_omega_save[0].z;
		dmod->spin.omega[0].val = (double)angle_omega_save[1].x;
		dmod->spin.omega[1].val = (double)angle_omega_save[1].y;
		dmod->spin.omega[2].val = (double)angle_omega_save[1].z;
	}
}
__global__ void realize_angleoff_krnl(struct dat_t *ddat)
{
	/* Single-threaded kernel - # of datasets nsets */
	/* Kernel implements the '=' state for each component of the angle offse */
	//int s = blockIdx.x * blockDim.x + threadIdx.x;

	int j, s_angleoff, s;

	if (threadIdx.x == 0) {
		for (j=0; j<=2; j++) {

			/* If a dataset has state '=' for component j of the angle offset, go back-
			 * wards in datafile until we reach a dataset for which component j of the
			 * angle offset has state 'f' or 'c' rather than '='.
			 *         s_angleoff is the number of the dataset we find.   */

			s_angleoff = -1;

			for (s=0; s<ddat->nsets; s++) {
				if (ddat->set[s].angleoff[j].state != '=')
					s_angleoff = s;
				else if (s_angleoff < 0)
					printf("can't use \"=\" state for the first dataset's angle offsets\n");
				else
					ddat->set[s].angleoff[j].val = ddat->set[s_angleoff].angleoff[j].val;
			}
		}
	}
}
__global__ void realize_omegaoff_krnl(struct dat_t *ddat)
{
	/* Multi-threaded kernel - # of datasets nsets */
	/* Implements the '=' state for each component of the spin offset   */
	int s = blockIdx.x * blockDim.x + threadIdx.x;
	int j, s_omegaoff;

	if (s < ddat->nsets) {
		for (j=0; j<=2; j++) {

			/* If a dataset has state = '=' for component j of the spin offset, go
			 * backwards in the datafile until we reach a dataset for which
			 * component j of the spin offset has state 'f' or 'c' rather than '='.
			 *         s_omegaoff is the number of the dataset we find.		 */

			s_omegaoff = -1;

			if (ddat->set[s].omegaoff[j].state != '=')
				s_omegaoff = s;
			else if (s_omegaoff < 0)
				printf("can't use \"=\" state for the first dataset's spin offsets\n");
			else
				ddat->set[s].omegaoff[j].val = ddat->set[s_omegaoff].omegaoff[j].val;
		}
	}
}

__host__ void realize_spin_gpu(
		struct par_t *dpar,
		struct mod_t *dmod,
		struct dat_t *ddat,
		unsigned char *htype,
		int *nframes,
		int *nviews,
		int nsets,
		hipStream_t *rs_stream)
{
	int s, f;
	dim3 nsetsBLK, nsetsTHD, BLK, THD;
	double3 *angle_omega_save;
	THD.x = maxThreadsPerBlock;

	gpuErrchk(hipMalloc((void**)&angle_omega_save, sizeof(double3)*2));

	/* Calculate launch parameters for all kernels going over all vertices */
	nsetsBLK.x = floor((THD.x - 1 + nsets) / THD.x);

	/* Get the three components of the angle and spin offsets for all datasets,
	 * with any "=" states taken into account  */
	realize_angleoff_krnl<<<1,1>>>(ddat);
	checkErrorAfterKernelLaunch("realize_angleoff_krnl (realize_spin_cuda_streams2.cu)");

	realize_omegaoff_krnl<<<nsetsBLK,THD>>>(ddat);
	checkErrorAfterKernelLaunch("realize_omegaoff_krnl, (realize_spin_cuda_streams2.cu");

	/* Note: Maybe turn the dataset loop into cudaStreams later */
	/* Determine the model spin state for each dataset in turn */

	for (s=0; s<nsets; s++) {

		/* Add this dataset's angle offsets to the model Euler angles. Later
		 * we'll add the spin offsets for each frame separately, after updating
		 * the intrinsic spin vector to each epoch. Save the original Euler
		 * angles to be restored later.          */
		/* Launch kernel do add angle offsets to Euler angles.  Three threads total */
		add_offsets_to_euler_krnl<<<1,1>>>(dmod,ddat,angle_omega_save,s);
		checkErrorAfterKernelLaunch("add_offsets_to_euler_streams2_krnl");

		BLK.x = floor((THD.x - 1 + nviews[s]) / THD.x);

		switch (htype[s]) {
		case DOPPLER:
			/* Create lists of epochs and impulses, starting at initial spin
			 * epoch t0 and ending at this view's epoch t, that will be
			 * "encountered" in evolving the spin state from t0 to t, with the
			 * impulses negated if we're evolving backwards in time. These
			 * lists will be used by the inteuler routine to break up evolution
			 * of the spin state) into integrations over several smaller time
			 * intervals, punctuated by spin impulses.                        */
			/* Integrate Euler's equations to get models intrinsic spin vector
			 * at the (light-time corrected) epoch of each view.
			 * dpar->int_method tells inteuler which integration method to use.
			 * If dmod->spin.pa == 1, Euler's equations aren't used (principal-
			 * axis rotator).
			 * Input dmod->spin is initial spin specification given in mod file.
			 * Output is frame[f].view[k].ae, the transformation matrix from
			 * ecliptic to body coordinates at epoch frame[f].view[k].t, and
			 * frame[f].view[k].intspin, the intrinsic spin vector (in body-
			 * fixed coordinates) at this epoch.         */
			/* Loop through every frame and launch a stream kernel with nview
			 * threads  */
			for (f=0; f<nframes[s]; f++)
				realize_spin_dop_krnl<<<BLK,THD,0,rs_stream[f]>>>(dmod,
						ddat, dpar, nviews[s], s, f);
			checkErrorAfterKernelLaunch("realize_spin_dop_streams2_krnl");

			break;
		case DELAY:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * Doppler and delay-Doppler procedures are identical.  */
			/* Deal with spin impulses  */
			/* Get the model's intrinsic spin vector (in body coordinates)
			 * at the (light-time corrected) epoch of each view.            */
			/* Apply dataset's spin offsets (also in body coordinates)
			 * to the intrinsic spin vector of this view.                    */

			for (f=0; f<nframes[s]; f++)
				realize_spin_deldop_krnl<<<BLK,THD,0,rs_stream[f]>>>(
						dmod, ddat, dpar, nviews[s], s, f);
			checkErrorAfterKernelLaunch("realize_spin_deldop_streams2_krnl");

			break;
		case POS:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * Doppler and POS procedures are identical. */
			/* Deal with spin impulses */
			/* Get model's intrinsic spin vector (in body coordinates)
			 * at the (light-time corrected) epoch of each view. */
			/* Apply dataset's spin offsets (also in body coordinates)
			 * to the intrinsic spin vector of this view. */

			for (f=0; f<nframes[s]; f++)
				realize_spin_poset_krnl<<<BLK,THD,0,rs_stream[f]>>>(
						dmod, ddat, dpar, nviews[s], s, f);
			checkErrorAfterKernelLaunch("realize_spin_poset_streams2_krnl");

			break;
		case LGHTCRV:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * procedure for each Doppler frame is identical to the procedure
			 * for each calculated lightcurve point (except that calculated
			 * lightcurve points don't have multiple "views").	 */

			int i, ncalc;
			ncalc = nframes[s];

			/* Deal with spin impulses */
			/* Get model's intrinsic spin vector (in body coordinates)
			 * at (light-time corrected) epoch of lightcurve point.*/
			/* Apply this dataset's spin offsets (also in body coordinates)
				to the intrinsic spin vector of this point. */

			for (i=0; i<ncalc; i++)
				realize_spin_lghtcrv_krnl<<<1,1,0,rs_stream[i]>>>(
						dmod, ddat, dpar, s, i); // f = i, k = 0
			checkErrorAfterKernelLaunch("realize_spin_lghtcrv_streams_krnl");

			break;
		default:
			bailout("realize_spin_cuda_streams: can't handle this type yet\n");
		}

		/* Final kernel launch in realize_spin_cuda */
		update_spin_angle_krnl<<<1,1>>>(dmod, angle_omega_save);
		checkErrorAfterKernelLaunch("update_spin_angle_streams_krnl");

	}
	hipFree(angle_omega_save);
}

__host__ void realize_spin_gpu_f(
		struct par_t *dpar,
		struct mod_t *dmod,
		struct dat_t *ddat,
		unsigned char *htype,
		int *nframes,
		int *nviews,
		int nsets,
		hipStream_t *rs_stream)
{
	/* This version eliminates doubles as much as possible and it changes the
	 * grid sizing. Now there are nframes-threads and nviews-streams.  */
	int s, f, k;
	dim3 nsetsBLK, nsetsTHD, BLK, THD;
	float3 *angle_omega_save;
	THD.x = maxThreadsPerBlock;

	gpuErrchk(hipMalloc((void**)&angle_omega_save, sizeof(float3)*2));

	/* Calculate launch parameters for all kernels going over all vertices */
	nsetsBLK.x = floor((THD.x - 1 + nsets) / THD.x);

	/* Get the three components of the angle and spin offsets for all datasets,
	 * with any "=" states taken into account  */
	realize_angleoff_krnl<<<1,1>>>(ddat);
	checkErrorAfterKernelLaunch("realize_angleoff_krnl (realize_spin_cuda_streams2.cu)");

	realize_omegaoff_krnl<<<nsetsBLK,THD>>>(ddat);
	checkErrorAfterKernelLaunch("realize_omegaoff_krnl, (realize_spin_cuda_streams2.cu");

	/* Note: Maybe turn the dataset loop into cudaStreams later */
	/* Determine the model spin state for each dataset in turn */

	for (s=0; s<nsets; s++) {

		/* Add this dataset's angle offsets to the model Euler angles. Later
		 * we'll add the spin offsets for each frame separately, after updating
		 * the intrinsic spin vector to each epoch. Save the original Euler
		 * angles to be restored later.          */
		/* Launch kernel do add angle offsets to Euler angles.  Three threads total */
		add_offsets_to_euler_f_krnl<<<1,1>>>(dmod,ddat,angle_omega_save,s);
		checkErrorAfterKernelLaunch("add_offsets_to_euler_streams2_F_krnl");

		BLK.x = floor((THD.x - 1 + nframes[s]) / THD.x);

		switch (htype[s]) {
		case DOPPLER:
			/* Create lists of epochs and impulses, starting at initial spin
			 * epoch t0 and ending at this view's epoch t, that will be
			 * "encountered" in evolving the spin state from t0 to t, with the
			 * impulses negated if we're evolving backwards in time. These
			 * lists will be used by the inteuler routine to break up evolution
			 * of the spin state) into integrations over several smaller time
			 * intervals, punctuated by spin impulses.                        */
			/* Integrate Euler's equations to get models intrinsic spin vector
			 * at the (light-time corrected) epoch of each view.
			 * dpar->int_method tells inteuler which integration method to use.
			 * If dmod->spin.pa == 1, Euler's equations aren't used (principal-
			 * axis rotator).
			 * Input dmod->spin is initial spin specification given in mod file.
			 * Output is frame[f].view[k].ae, the transformation matrix from
			 * ecliptic to body coordinates at epoch frame[f].view[k].t, and
			 * frame[f].view[k].intspin, the intrinsic spin vector (in body-
			 * fixed coordinates) at this epoch.         */
			/* Loop through every frame and launch a stream kernel with nview
			 * threads  */
			for (k=0; k<nviews[s]; k++)
				realize_spin_dop_f_krnl<<<BLK,THD,0,rs_stream[k]>>>(dmod,
						ddat, dpar, nframes[s], s, k);
			checkErrorAfterKernelLaunch("realize_spin_dop_streams_f_krnl");

			break;
		case DELAY:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * Doppler and delay-Doppler procedures are identical.  */
			/* Deal with spin impulses  */
			/* Get the model's intrinsic spin vector (in body coordinates)
			 * at the (light-time corrected) epoch of each view.            */
			/* Apply dataset's spin offsets (also in body coordinates)
			 * to the intrinsic spin vector of this view.                    */

			for (k=0; k<nviews[s]; k++)
				realize_spin_deldop_f_krnl<<<BLK,THD,0,rs_stream[k]>>>(
						dmod, ddat, dpar, nframes[s], s, k);
			checkErrorAfterKernelLaunch("realize_spin_deldop_streams_f_krnl");

			break;
		case POS:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * Doppler and POS procedures are identical. */
			/* Deal with spin impulses */
			/* Get model's intrinsic spin vector (in body coordinates)
			 * at the (light-time corrected) epoch of each view. */
			/* Apply dataset's spin offsets (also in body coordinates)
			 * to the intrinsic spin vector of this view. */

			for (k=0; k<nviews[s]; k++)
				realize_spin_poset_f_krnl<<<BLK,THD,0,rs_stream[k]>>>(
						dmod, ddat, dpar, nframes[s], s, k);
			checkErrorAfterKernelLaunch("realize_spin_poset_streams2_krnl");

			break;
		case LGHTCRV:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * procedure for each Doppler frame is identical to the procedure
			 * for each calculated lightcurve point (except that calculated
			 * lightcurve points don't have multiple "views").	 */

			int ncalc;
			ncalc = nframes[s];

			/* Deal with spin impulses */
			/* Get model's intrinsic spin vector (in body coordinates)
			 * at (light-time corrected) epoch of lightcurve point.*/
			/* Apply this dataset's spin offsets (also in body coordinates)
				to the intrinsic spin vector of this point. */
			THD.x = maxThreadsPerBlock;
			BLK.x = floor((THD.x + ncalc)/THD.x);
			realize_spin_lghtcrv_f_krnl<<<BLK,THD>>>(
					dmod, ddat, dpar, s, ncalc); // f = i, k = 0
			checkErrorAfterKernelLaunch("realize_spin_lghtcrv_streams2_f_krnl");

			break;
		default:
			bailout("realize_spin_cuda_streams: can't handle this type yet\n");
		}

		/* Final kernel launch in realize_spin_cuda */
		update_spin_angle_f_krnl<<<1,1>>>(dmod, angle_omega_save);
		checkErrorAfterKernelLaunch("update_spin_angle_streams_f_krnl");

	}
	hipFree(angle_omega_save);
}





