#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                           realize_spin.c

Takes the initial spin state described in mod, computes the spin state at the epoch of
each data frame, and produces the various coordinate transformation matrices needed in
dat.  Also computes the total apparent spin vector at the epoch of each data frame.

Modified 2015 June 3 by CM:
    Implement smearing for the "fit" and "write" actions

Modified 2011 August 15 by CM:
    Determine which spin impulses must be applied to each frame or
        lightcurve point
    Pass the "int_abstol" parameter to the inteuler routine

Modified 2006 June 18 by CM:
    Eliminate range datasets

Modified 2005 January 20 by CM:
    For POS and range datasets, save the intrisic spin vector and total
        (intrinsic plus orbital) spin vector

Modified 2004 March 22 by CM:
    For lightcurve points, save the intrisic spin vector and total
        (intrinsic plus orbital) spin vector

Modified 2004 Feb 5 by CM:
    Implement "=" state for angle and spin offsets by creating
    routines realize_angleoff and realize_omegaoff

Modified 2003 May 4 by CM:
    Apply angle offsets to Doppler datasets, not just delay-Doppler
 *****************************************************************************************/
extern "C" {
#include "../shape/head.h"
}
/* This struct is required to pass data to the pthreaded realize_spin sub-
 * functions. */
typedef struct rspin_thread_t
{
    int thread_no;
	struct par_t *parameter;
    struct mod_t *model;
    struct dat_t *data;
    unsigned char *host_type;
    int *nframes;
    int *nviews;
    int *GPUID;
    int gpuid;
    int nsets;
    hipStream_t *gpu_stream;
} rspin_thd_data;

void *realize_spin_pthread_sub(void *ptr);

__global__ void add_offsets_to_euler_krnl(struct mod_t *dmod,
		struct dat_t *ddat, double3 *angle_omega_save, int s)
{
	/* Single-threaded kernel */
	/*	angle_omega_save[0].x,y,z = original anglesave[3]
	 * 	angle_omega_save[1].x,y,z = original omegasave[3]
	 * 		 */

	if (threadIdx.x == 0) {

		angle_omega_save[0].x = dmod->spin.angle[0].val;
		angle_omega_save[0].y = dmod->spin.angle[1].val;
		angle_omega_save[0].z = dmod->spin.angle[2].val;
		angle_omega_save[1].x = dmod->spin.omega[0].val;
		angle_omega_save[1].y = dmod->spin.omega[1].val;
		angle_omega_save[1].z = dmod->spin.omega[2].val;

		for (int j=0; j<=2; j++)
			dmod->spin.angle[j].val += ddat->set[s].angleoff[j].val;
	}
}
__global__ void realize_spin_dop_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nviews, int s, int nfrm_alloc)
{
	/* nfrm_alloc-threaded kernel */
	int j, k, f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < nfrm_alloc) {
		for (k=0; k<nviews; k++) {
			dev_realize_impulse(dmod->spin,
					ddat->set[s].desc.doppler.frame[f].view[k].t,
					ddat->set[s].desc.doppler.frame[f].t_integrate,
					ddat->set[s].desc.doppler.frame[f].impulse,
					&ddat->set[s].desc.doppler.frame[f].n_integrate,s,f,k);

			dev_inteuler(dmod->spin,
					ddat->set[s].desc.doppler.frame[f].t_integrate,
					ddat->set[s].desc.doppler.frame[f].impulse,
					ddat->set[s].desc.doppler.frame[f].n_integrate,
					ddat->set[s].desc.doppler.frame[f].view[k].intspin,
					ddat->set[s].desc.doppler.frame[f].view[k].ae,
					dmod->spin.pa, dpar->int_method, dpar->int_abstol);

			for (j=0; j<=2; j++)
				ddat->set[s].desc.doppler.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

			dev_cotrans2(ddat->set[s].desc.doppler.frame[f].view[k].intspin,
					ddat->set[s].desc.doppler.frame[f].view[k].ae,
					ddat->set[s].desc.doppler.frame[f].view[k].intspin, -1);

			for (j=0; j<=2; j++)
				ddat->set[s].desc.doppler.frame[f].view[k].spin[j] = ddat->set[s].desc.doppler.frame[f].view[k].orbspin[j] +
				ddat->set[s].desc.doppler.frame[f].view[k].intspin[j];
		}
	}
}
__global__ void realize_spin_deldop_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nviews, int s, int size)
{
	/* nfrm_alloc-threaded kernel */
	int j, k, f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < size) {
		for (k=0; k<nviews; k++) {
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.deldop.frame[f].view[k].t,
				ddat->set[s].desc.deldop.frame[f].t_integrate,
				ddat->set[s].desc.deldop.frame[f].impulse,
				&ddat->set[s].desc.deldop.frame[f].n_integrate,
				s, f, k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.deldop.frame[f].t_integrate,
				ddat->set[s].desc.deldop.frame[f].impulse,
				ddat->set[s].desc.deldop.frame[f].n_integrate,
				ddat->set[s].desc.deldop.frame[f].view[k].intspin,
				ddat->set[s].desc.deldop.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.deldop.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.deldop.frame[f].view[k].intspin,
				ddat->set[s].desc.deldop.frame[f].view[k].ae,
				ddat->set[s].desc.deldop.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.deldop.frame[f].view[k].spin[j] = ddat->set[s].desc.deldop.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.deldop.frame[f].view[k].intspin[j];
	}
	}
}
__global__ void realize_spin_poset_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int nviews, int s, int size)
{
	/* nframes-threaded kernel */
	int j, k, f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < size)
	{
		for (k=0; k<nviews; k++) {
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.poset.frame[f].view[k].t,
				ddat->set[s].desc.poset.frame[f].t_integrate,
				ddat->set[s].desc.poset.frame[f].impulse,
				&ddat->set[s].desc.poset.frame[f].n_integrate,s,f,k);

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.poset.frame[f].t_integrate,
				ddat->set[s].desc.poset.frame[f].impulse,
				ddat->set[s].desc.poset.frame[f].n_integrate,
				ddat->set[s].desc.poset.frame[f].view[k].intspin,
				ddat->set[s].desc.poset.frame[f].view[k].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.poset.frame[f].view[k].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.poset.frame[f].view[k].intspin,
				ddat->set[s].desc.poset.frame[f].view[k].ae,
				ddat->set[s].desc.poset.frame[f].view[k].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.poset.frame[f].view[k].spin[j] = ddat->set[s].desc.poset.frame[f].view[k].orbspin[j] +
			ddat->set[s].desc.poset.frame[f].view[k].intspin[j];
	}
	}
}
__global__ void realize_spin_lghtcrv_krnl(struct mod_t *dmod, struct dat_t *ddat,
		struct par_t *dpar, int s, int size)
{
	/* nframes-threaded kernel (+1 for lightcurve) */
	int j, i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (i < size)
	{
		dev_realize_impulse(dmod->spin,
				ddat->set[s].desc.lghtcrv.x[i],
				ddat->set[s].desc.lghtcrv.rend[i].t_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].impulse,
				&ddat->set[s].desc.lghtcrv.rend[i].n_integrate,
				s,i,0);	// s = s,  f = i, k = 0

		dev_inteuler(dmod->spin,
				ddat->set[s].desc.lghtcrv.rend[i].t_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].impulse,
				ddat->set[s].desc.lghtcrv.rend[i].n_integrate,
				ddat->set[s].desc.lghtcrv.rend[i].intspin,
				ddat->set[s].desc.lghtcrv.rend[i].ae,
				dmod->spin.pa, dpar->int_method, dpar->int_abstol);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.lghtcrv.rend[i].intspin[j] += ddat->set[s].omegaoff[j].val;

		dev_cotrans2(ddat->set[s].desc.lghtcrv.rend[i].intspin,
				ddat->set[s].desc.lghtcrv.rend[i].ae,
				ddat->set[s].desc.lghtcrv.rend[i].intspin, -1);

		for (j=0; j<=2; j++)
			ddat->set[s].desc.lghtcrv.rend[i].spin[j] = ddat->set[s].desc.lghtcrv.rend[i].orbspin[j] +
			ddat->set[s].desc.lghtcrv.rend[i].intspin[j];
	}
}
__device__ void dev_realize_impulse(struct spin_t spin, double t,
		double t_integrate[], double impulse[][3], int *n_integrate, int s, int f, int k)
{
	/* Determine which spin impulses will be encountered in evolving the spin state
	 * from initial spin epoch t0 to epoch t of a particular frame or lightcurve
	 * point; then create lists of epochs and impulses, starting at t0 and ending
	 * at t, with the impulses negated if we're evolving backwards in time.     */
	int j, n;
	k = 0;
	t_integrate[k] = spin.t0;
	for (j=0; j<=2; j++)
		impulse[k][j] = 0.0;
	if (t >= spin.t0) {

		/* Integrating forward in time, so add the spin impulses  */
		for (n=0; n<spin.n_impulse; n++) {
			if (spin.t_impulse[n] > spin.t0 && spin.t_impulse[n] <= t) {
				k++;
				t_integrate[k] = spin.t_impulse[n];
				for (j=0; j<=2; j++)
					impulse[k][j] = spin.impulse[n][j].val;
			}
		}
		if (t_integrate[k] < t) {
			k++;
			t_integrate[k] = t;
			for (j=0; j<=2; j++)
				impulse[k][j] = 0.0;
		}
	} else {

		/* Integrating backwards in time, so subtract the spin impulses  */
		for (n=spin.n_impulse-1; n>=0; n--) {
			if (spin.t_impulse[n] < spin.t0 && spin.t_impulse[n] >= t) {
				k++;
				t_integrate[k] = spin.t_impulse[n];
				for (j=0; j<=2; j++)
					impulse[k][j] = -spin.impulse[n][j].val;
			}
		}
		if (t_integrate[k] > t) {
			k++;
			t_integrate[k] = t;
			for (j=0; j<=2; j++)
				impulse[k][j] = 0.0;
		}
	}
	*n_integrate = k + 1;
}
__global__ void update_spin_angle_krnl(struct mod_t *dmod,
		double3 *angle_omega_save)
{
	/* Single-threaded kernel */
	/*	angle_omega_save[0].x,y,z = original anglesave[3]
	 * 	angle_omega_save[1].x,y,z = original omegasave[3]
	 * 		 */
	if(threadIdx.x == 0) {
		dmod->spin.angle[0].val = angle_omega_save[0].x;
		dmod->spin.angle[1].val = angle_omega_save[0].y;
		dmod->spin.angle[2].val = angle_omega_save[0].z;
		dmod->spin.omega[0].val = angle_omega_save[1].x;
		dmod->spin.omega[1].val = angle_omega_save[1].y;
		dmod->spin.omega[2].val = angle_omega_save[1].z;
	}
}
__global__ void realize_angleoff_krnl(struct dat_t *ddat, int gpuid)
{
	/* Single-threaded kernel - # of datasets nsets */
	/* Kernel implements the '=' state for each component of the angle offse */
	//int s = blockIdx.x * blockDim.x + threadIdx.x;

	int j, s_angleoff, s;

	if (threadIdx.x == 0) {
		for (j=0; j<=2; j++) {

			/* If a dataset has state '=' for component j of the angle offset, go back-
			 * wards in datafile until we reach a dataset for which component j of the
			 * angle offset has state 'f' or 'c' rather than '='.
			 *         s_angleoff is the number of the dataset we find.   */

			s_angleoff = -1;

			for (s=0; s<ddat->nsets; s++) {
				if ((ddat->set[s].inputnode == gpuid)||(gpuid==-1)) {
					if (ddat->set[s].angleoff[j].state != '=')
						s_angleoff = s;
					else if (s_angleoff < 0)
						printf("can't use \"=\" state for the first dataset's angle offsets\n");
					else
						ddat->set[s].angleoff[j].val = ddat->set[s_angleoff].angleoff[j].val;
				}
			}
		}
	}
}
__global__ void realize_omegaoff_krnl(struct dat_t *ddat, int gpuid)
{
	/* Multi-threaded kernel - # of datasets nsets */
	/* Implements the '=' state for each component of the spin offset   */
	int s = blockIdx.x * blockDim.x + threadIdx.x;
	int j, s_omegaoff;

	if (s<ddat->nsets) {
		if ((ddat->set[s].inputnode==gpuid) || (gpuid==-1)) {
			for (j=0; j<=2; j++) {

				/* If a dataset has state = '=' for component j of the spin offset, go
				 * backwards in the datafile until we reach a dataset for which
				 * component j of the spin offset has state 'f' or 'c' rather than '='.
				 *         s_omegaoff is the number of the dataset we find.		 */

				s_omegaoff = -1;

				if (ddat->set[s].omegaoff[j].state != '=')
					s_omegaoff = s;
				else if (s_omegaoff < 0)
					printf("can't use \"=\" state for the first dataset's spin offsets\n");
				else
					ddat->set[s].omegaoff[j].val = ddat->set[s_omegaoff].omegaoff[j].val;
			}
		}
	}
}

__host__ void realize_spin_gpu(
		struct par_t *dpar,
		struct mod_t *dmod,
		struct dat_t *ddat,
		unsigned char *htype,
		int *nframes,
		int *nviews,
		int nsets,
		hipStream_t *rs_stream)
{
	int s;
	dim3 nsetsBLK, nsetsTHD, BLK, THD, BLKfrm, THD64;
	double3 *angle_omega_save;
	THD.x = maxThreadsPerBlock;
	THD64.x = 64;

	gpuErrchk(hipMalloc((void**)&angle_omega_save, sizeof(double3)*2));

	/* Calculate launch parameters for all kernels going over all vertices */
	nsetsBLK.x = floor((THD.x - 1 + nsets) / THD.x);

	/* Get the three components of the angle and spin offsets for all datasets,
	 * with any "=" states taken into account  */
	realize_angleoff_krnl<<<1,1>>>(ddat, -1);
	checkErrorAfterKernelLaunch("realize_angleoff_krnl (realize_spin_cuda_streams2.cu)");

	realize_omegaoff_krnl<<<nsetsBLK,THD>>>(ddat, -1);
	checkErrorAfterKernelLaunch("realize_omegaoff_krnl, (realize_spin_cuda_streams2.cu");

	/* Note: Maybe turn the dataset loop into cudaStreams later */
	/* Determine the model spin state for each dataset in turn */

	for (s=0; s<nsets; s++) {

		/* Add this dataset's angle offsets to the model Euler angles. Later
		 * we'll add the spin offsets for each frame separately, after updating
		 * the intrinsic spin vector to each epoch. Save the original Euler
		 * angles to be restored later.          */
		/* Launch kernel do add angle offsets to Euler angles.  Three threads total */
		add_offsets_to_euler_krnl<<<1,1>>>(dmod,ddat,angle_omega_save,s);
		checkErrorAfterKernelLaunch("add_offsets_to_euler_krnl");

		BLK.x = floor((THD.x - 1 + nviews[s]) / THD.x);
		BLKfrm.x = floor((THD64.x - 1 + nframes[s])/THD64.x);

		switch (htype[s]) {
		case DOPPLER:
			/* Create lists of epochs and impulses, starting at initial spin
			 * epoch t0 and ending at this view's epoch t, that will be
			 * "encountered" in evolving the spin state from t0 to t, with the
			 * impulses negated if we're evolving backwards in time. These
			 * lists will be used by the inteuler routine to break up evolution
			 * of the spin state) into integrations over several smaller time
			 * intervals, punctuated by spin impulses.                        */
			/* Integrate Euler's equations to get models intrinsic spin vector
			 * at the (light-time corrected) epoch of each view.
			 * dpar->int_method tells inteuler which integration method to use.
			 * If dmod->spin.pa == 1, Euler's equations aren't used (principal-
			 * axis rotator).
			 * Input dmod->spin is initial spin specification given in mod file.
			 * Output is frame[f].view[k].ae, the transformation matrix from
			 * ecliptic to body coordinates at epoch frame[f].view[k].t, and
			 * frame[f].view[k].intspin, the intrinsic spin vector (in body-
			 * fixed coordinates) at this epoch.         */
			/* Loop through every frame and launch a stream kernel with nview
			 * threads  */
			realize_spin_dop_krnl<<<BLKfrm,THD64>>>(dmod, ddat, dpar, nviews[s],
					s, nframes[s]);
			checkErrorAfterKernelLaunch("realize_spin_dop_krnl2");

			break;
		case DELAY:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * Doppler and delay-Doppler procedures are identical.  */
			/* Deal with spin impulses  */
			/* Get the model's intrinsic spin vector (in body coordinates)
			 * at the (light-time corrected) epoch of each view.            */
			/* Apply dataset's spin offsets (also in body coordinates)
			 * to the intrinsic spin vector of this view.                    */
			realize_spin_deldop_krnl<<<BLKfrm,THD64>>>(dmod, ddat, dpar,
					nviews[s], s, nframes[s]);
			checkErrorAfterKernelLaunch("realize_spin_deldop_krnl2");

			break;
		case POS:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * Doppler and POS procedures are identical. */
			/* Deal with spin impulses */
			/* Get model's intrinsic spin vector (in body coordinates)
			 * at the (light-time corrected) epoch of each view. */
			/* Apply dataset's spin offsets (also in body coordinates)
			 * to the intrinsic spin vector of this view. */
			realize_spin_poset_krnl<<<BLKfrm,THD64>>>(dmod, ddat, dpar,
					nviews[s], s, nframes[s]);
			checkErrorAfterKernelLaunch("realize_spin_poset_krnl2");

			break;
		case LGHTCRV:
			/* See "case DOPPLER" above for more extensive comments, since the
			 * procedure for each Doppler frame is identical to the procedure
			 * for each calculated lightcurve point (except that calculated
			 * lightcurve points don't have multiple "views").	 */
			/* Deal with spin impulses */
			/* Get model's intrinsic spin vector (in body coordinates)
			 * at (light-time corrected) epoch of lightcurve point.*/
			/* Apply this dataset's spin offsets (also in body coordinates)
				to the intrinsic spin vector of this point. */
			realize_spin_lghtcrv_krnl<<<BLKfrm,THD64>>>(dmod, ddat, dpar, s,
					(nframes[s]+1)); // f = i, k = 0
			checkErrorAfterKernelLaunch("realize_spin_lghtcrv_krnl2");

			break;
		default:
			bailout("realize_spin_gpu2: can't handle this type yet\n");
		}
		/* Final kernel launch in realize_spin_cuda */
		update_spin_angle_krnl<<<1,1>>>(dmod, angle_omega_save);
		checkErrorAfterKernelLaunch("update_spin_angle_krnl");
	}
	hipFree(angle_omega_save);
}

__host__ void realize_spin_pthread(
		struct par_t *dpar0,
		struct par_t *dpar1,
		struct mod_t *dmod0,
		struct mod_t *dmod1,
		struct dat_t *ddat0,
		struct dat_t *ddat1,
		unsigned char *htype,
		int *nframes,
		int *nviews,
		int *GPUID,
		int nsets,
		pthread_t thread1 ,
		pthread_t thread2,
		hipStream_t *gpu0_stream,
		hipStream_t *gpu1_stream)
{
	/* This is the supervisory function handling the pthreaded/dual-GPU realize_spin
	 * sub-functions */
	int s;
	dim3 nsetsBLK, THD;

	THD.x = maxThreadsPerBlock;

	/* Create the data constructs to pass to thread, then fill them up */
	rspin_thd_data data1, data2;

	data1.gpu_stream = gpu0_stream;
	data2.gpu_stream = gpu1_stream;
	data1.thread_no  = 1;
	data2.thread_no  = 2;
	data1.gpuid 	 = GPU0;
	data2.gpuid 	 = GPU1;
	data1.GPUID 	 = data2.GPUID 		= GPUID;
	data1.parameter  = dpar0;
	data2.parameter  = dpar1;
	data1.model 	 = dmod0;
	data2.model 	 = dmod1;
	data1.data		 = ddat0;
	data2.data 		 = ddat1;
	data1.host_type	 = data2.host_type 	= htype;
	data1.nframes 	 = data2.nframes 	= nframes;
	data1.nsets 	 = data2.nsets 		= nsets;
	data1.nviews 	 = data2.nviews		= nviews;

	/* Calculate launch parameters for all kernels going over all vertices */
	nsetsBLK.x = floor((THD.x - 1 + nsets) / THD.x);

//	/* Get the three components of the angle and spin offsets for all datasets,
//	 * with any "=" states taken into account. Since sets are split between two
//	 * GPUs we need to switch devices as needed. Only two small kernels are
//	 * involved and both need to be done before the main pthreaded portion.  */
//	for (s=0; s<nsets; s++) {
//		gpuErrchk(hipSetDevice(GPUID[s]));
//		realize_angleoff_krnl<<<1,1>>>(ddat, GPUID[s]);
//		realize_omegaoff_krnl<<<nsetsBLK,THD>>>(ddat, GPUID[s]);
//	} checkErrorAfterKernelLaunch("realize_angleoff_krnl/realize_omegaoff_krnl");


	/* Temporary code : */
	gpuErrchk(hipSetDevice(GPU0));
	for (s=0; s<nsets; s++) {
		//gpuErrchk(hipSetDevice(GPUID[s]));
		realize_angleoff_krnl<<<1,1>>>(ddat0, GPUID[s]);
		realize_omegaoff_krnl<<<nsetsBLK,THD>>>(ddat0, GPUID[s]);
	} checkErrorAfterKernelLaunch("realize_angleoff_krnl/realize_omegaoff_krnl");
	gpuErrchk(hipSetDevice(GPU1));
	for (s=0; s<nsets; s++) {
		//gpuErrchk(hipSetDevice(GPUID[s]));
		realize_angleoff_krnl<<<1,1>>>(ddat0, GPUID[s]);
		realize_omegaoff_krnl<<<nsetsBLK,THD>>>(ddat1, GPUID[s]);
	} checkErrorAfterKernelLaunch("realize_angleoff_krnl/realize_omegaoff_krnl");
	gpuErrchk(hipSetDevice(GPU0));

	/* From here, launch the pthreaded subfunction */
	pthread_create(&thread1, NULL, realize_spin_pthread_sub,(void*)&data1);
	pthread_create(&thread2, NULL, realize_spin_pthread_sub,(void*)&data2);

	pthread_join(thread1, NULL);
	pthread_join(thread2, NULL);

	gpuErrchk(hipSetDevice(GPU0));
}

void *realize_spin_pthread_sub(void *ptr) {

	double3 *angle_omega_save;
	int s;
	dim3 BLK, BLKfrm, THD, THD64;
	rspin_thd_data *data;
	data = (rspin_thd_data *) ptr;  // type cast to a pointer to thdata

	THD.x = maxThreadsPerBlock; THD64.x = 64;

	gpuErrchk(hipSetDevice(data->gpuid));

	gpuErrchk(hipMalloc((void**)&angle_omega_save, sizeof(double3)*2));

	for (s=0; s<data->nsets; s++) {

		if (data->GPUID[s]==data->gpuid) {
			/* Add this dataset's angle offsets to the model Euler angles. Later
			 * we'll add the spin offsets for each frame separately, after updating
			 * the intrinsic spin vector to each epoch. Save the original Euler
			 * angles to be restored later.          */
			/* Launch kernel do add angle offsets to Euler angles.  Three threads total */
			add_offsets_to_euler_krnl<<<1,1>>>(data->model,data->data,angle_omega_save,s);
			checkErrorAfterKernelLaunch("add_offsets_to_euler_krnl");

			BLK.x = floor((THD.x - 1 + data->nviews[s]) / THD.x);
			BLKfrm.x = floor((THD64.x - 1 + data->nframes[s]) / THD64.x);

			switch (data->host_type[s]) {
			case DOPPLER:
				/* Create lists of epochs and impulses, starting at initial spin
				 * epoch t0 and ending at this view's epoch t, that will be
				 * "encountered" in evolving the spin state from t0 to t, with the
				 * impulses negated if we're evolving backwards in time. These
				 * lists will be used by the inteuler routine to break up evolution
				 * of the spin state) into integrations over several smaller time
				 * intervals, punctuated by spin impulses.                        */
				/* Integrate Euler's equations to get models intrinsic spin vector
				 * at the (light-time corrected) epoch of each view.
				 * dpar->int_method tells inteuler which integration method to use.
				 * If dmod->spin.pa == 1, Euler's equations aren't used (principal-
				 * axis rotator).
				 * Input dmod->spin is initial spin specification given in mod file.
				 * Output is frame[f].view[k].ae, the transformation matrix from
				 * ecliptic to body coordinates at epoch frame[f].view[k].t, and
				 * frame[f].view[k].intspin, the intrinsic spin vector (in body-
				 * fixed coordinates) at this epoch.         */
				/* Loop through every frame and launch a stream kernel with nview
				 * threads  */
				realize_spin_dop_krnl<<<BLKfrm,THD64>>>(data->model, data->data,
						data->parameter, data->nviews[s], s, data->nframes[s]);
				checkErrorAfterKernelLaunch("realize_spin_dop_krnl");

				break;
			case DELAY:
				/* See "case DOPPLER" above for more extensive comments, since the
				 * Doppler and delay-Doppler procedures are identical.  */
				/* Deal with spin impulses  */
				/* Get the model's intrinsic spin vector (in body coordinates)
				 * at the (light-time corrected) epoch of each view.            */
				/* Apply dataset's spin offsets (also in body coordinates)
				 * to the intrinsic spin vector of this view.                    */
				realize_spin_deldop_krnl<<<BLKfrm,THD64>>>(data->model, data->data,
						data->parameter, data->nviews[s], s, data->nframes[s]);
				checkErrorAfterKernelLaunch("realize_spin_deldop_krnl");

				break;
			case POS:
				/* See "case DOPPLER" above for more extensive comments, since the
				 * Doppler and POS procedures are identical. */
				/* Deal with spin impulses */
				/* Get model's intrinsic spin vector (in body coordinates)
				 * at the (light-time corrected) epoch of each view. */
				/* Apply dataset's spin offsets (also in body coordinates)
				 * to the intrinsic spin vector of this view. */
				realize_spin_poset_krnl<<<BLKfrm,THD64>>>(data->model, data->data,
						data->parameter, data->nviews[s], s, data->nframes[s]);
				checkErrorAfterKernelLaunch("realize_spin_poset_krnl");

				break;
			case LGHTCRV:
				/* See "case DOPPLER" above for more extensive comments, since the
				 * procedure for each Doppler frame is identical to the procedure
				 * for each calculated lightcurve point (except that calculated
				 * lightcurve points don't have multiple "views").	 */
				/* Deal with spin impulses */
				/* Get model's intrinsic spin vector (in body coordinates)
				 * at (light-time corrected) epoch of lightcurve point.*/
				/* Apply this dataset's spin offsets (also in body coordinates)
				to the intrinsic spin vector of this point. */
				realize_spin_lghtcrv_krnl<<<BLKfrm,THD64>>>(data->model, data->data,
						data->parameter, s, (data->nframes[s]+1)); // f = i, k = 0
				checkErrorAfterKernelLaunch("realize_spin_lghtcrv_krnl");

				break;
			default:
				bailout("realize_spin_pthread_sub: can't handle this type yet\n");
			}

			/* Final kernel launch in realize_spin_cuda */
			update_spin_angle_krnl<<<1,1>>>(data->model, angle_omega_save);
			checkErrorAfterKernelLaunch("update_spin_angle_krnl");
		}
	}
	hipFree(angle_omega_save);
	pthread_exit(0);
}






