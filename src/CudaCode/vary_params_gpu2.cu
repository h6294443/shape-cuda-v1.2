#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                            vary_params.c

This routine is called by every processing node for every trial value of every floating
parameter during a fit, in order to implement the "vary_radalb" "vary_optalb"
"vary_delcor0" and "vary_dopscale" parameters.  The code, which is essentially lifted from
calc_fits.c, computes up to four means:

a) mean distance towards Earth of the subradar point relative to the COM,
   for delay-Doppler frames whose 0th-order delay correction polynomial coefficient is not
   held constant; this is used to adjust the 0th-order delay correction polynomial
   coefficient if the "vary_delcor0" parameter is turned on.

b) mean "radar" projected area for (delay-)Doppler frames that are treated as absolute
   photometry; this is used to adjust the radar albedo (R) if the "vary_radalb" parameter
   is turned on.

c) mean "optical" unshadowed projected area for calculated lightcurve points that are
   treated as absolute photometry; this is used to adjust the optical albedo (R or w) if
   the "vary_optalb" parameter is turned on.  Note that plane-of-sky datasets are not used
   here, since these frames are always treated as relative photometry.

d) mean cos(subradar latitude) for (delay-)Doppler frames in datasets whose Doppler
   scaling parameter is allowed to float; this is used to adjust those parameters if the
   "vary_dopscale" parameter is turned on.

When a branch node calls this routine, it returns its datasets' summed contributions (NOT
mean contributions) to the four output parameters, deldop_zmax, rad_xsec, opt_brightness,
and cos_subradarlat.

When the root node calls this routine, it first computes its datasets' summed
contributions to these four parameters; then it receives and adds in the contributions
from the branch nodes; and finally it returns the mean (NOT summed) parameters.

Before calling vary_params, the model's size/shape and spin states must be realized
(realize_mod and realize_spin); if albedos are being varied jointly with other parameters,
the photometric state must also be realized (realize_photo); and in either case the
0th-order delay correction polynomial coefficients and the Doppler scaling factors must be
reset to their saved values via the appropriate calls to realize_delcor and
realize_dopscale, respectively.

Modified 2017 March 27 by ME:
	Split off again from previous CUDA code to create a cudaStreams version.
	cudaStreams provide another level of parallelism by executing functions
	inside a stream in that specific order, but other streams are independent
	and can thus perform their own tasks at the same time.  The CUDA runtime
	driver will keep loading the GPU with parallel streamed tasks until
	capacity is reached or the tasks run out.
	A special note on the code structure in this version:  There are a lot of
	if blocks and for loops over frames that may seem weirdly placed or
	inefficient, i.e. multiple for-loops through frames right after one another
	with just one or two lines of code inside each loop.  This is done
	deliberately to launch streamed kernels in parallel.  Alteration to the
	code could break this, resulting in a loss of parallelism and therefore:
	speed.

Modified 2016 November 6 by ME:
	Split off from vary_params to create a version that performs almost exclusively
	on the GPU

Modified 2015 June 10 by CM:
    Implement smearing

Modified 2014 February 12 by CM:
    Add "ilaw" argument to the apply_photo routine

Modified 2012 March 23 by CM:
    Implement Doppler scaling

Modified 2011 September 10 by CM:
    Two small aesthetic changes in the lightcurve section of the code

Modified 2010 June 15 by CM:
    Revise arguments to pos2deldop and pos2doppler routines

Modified 2010 April 12 by CM:
    Include overflow region when computing cross sections
    Added comment about calling realize_delcor before calling vary_params

Modified 2009 March 29 by CM:
    For MPI_Recv calls, mpi_par[0] is no longer equal to the MPI action,
        since the message tag argument already serves that purpose (as of
        2008 April 10) -- so the other mpi_par elements are renumbered
    Add "warn_badradar" argument to pos2deldop and pos2doppler routines

Modified 2008 April 10 by CM:
    Use message tag argument to MPI_Recv to identify the MPI action

Modified 2007 August 18 by CM:
    Rename MPI_TAG to MPI_TAG_1 to avoid name conflict with mpich headers

Modified 2007 August 4 by CM:
    Add orbit_offset and body arguments to posvis routine and remove
        facet argument
    Add orbit_xoff, orbit_yoff, orbit_dopoff, and body arguments to
        pos2deldop and pos2doppler routines
    Add body argument to apply_photo routine

Written 2006 October 1 by CM
 *****************************************************************************************/

extern "C" {
#include "../shape/head.h"
}
__device__ float gpu_deldop_cross_section=0.0, gpu_doppler_cross_section=0.0,
		gpu_sum_rad_xsec=0.0, gpu_sum_cos_subradarlat=0.0, gpu_sum_deldop_zmax,
		gpu_sum_opt_brightness;
__device__ double gpu_deldop_zmax=0.0, gpu_rad_xsec=0.0, gpu_opt_brightness=0.0,
		gpu_cos_subradarlat=0.0;

__global__ void init_krnl(
		struct par_t *dpar,
		struct dat_t *ddat,
		int *compute_zmax,
		int *compute_cosdelta,
		int *compute_brightness,
		unsigned char *dtype,
		int nsets) {
	/* Single-threaded kernel, to be performed by GPU0 */
	int s;
	if (threadIdx.x == 0) {
		/* Initialize __device__ (file scope) variables to zero */
		gpu_deldop_cross_section = 0.0;
		gpu_doppler_cross_section = 0.0;
		gpu_sum_rad_xsec = 0.0;
		gpu_sum_cos_subradarlat = 0.0;
		gpu_sum_deldop_zmax = 0.0;
		gpu_sum_opt_brightness  = 0.0;
		gpu_deldop_zmax = 0.0;
		gpu_rad_xsec = 0.0;
		gpu_opt_brightness = 0.0;
		gpu_cos_subradarlat = 0.0;

		for (s=0; s<nsets; s++) {
			switch(dtype[s]) {
			case DELAY:
				compute_cosdelta[s] = (dpar->vary_dopscale != VARY_NONE
						&& ddat->set[s].desc.deldop.dopscale.state != 'c');
				compute_zmax[s] = (dpar->vary_delcor0 != VARY_NONE
						&& ddat->set[s].desc.deldop.delcor.a[0].state != 'c');
				break;
			case DOPPLER:
				compute_cosdelta[s] = (dpar->vary_dopscale != VARY_NONE &&
						ddat->set[s].desc.doppler.dopscale.state != 'c');
				break;
			case POS:
				break;
			case LGHTCRV:
				compute_brightness[s] = (dpar->vary_optalb != VARY_NONE
						&& ddat->set[s].desc.lghtcrv.cal.state == 'c');
			}
		}
	}
}

__global__ void zmax_final_krnl(float value) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		gpu_sum_deldop_zmax += value;
	}
}

__global__ void xsec_doppler_krnl(struct dat_t *ddat, float frm_xsec,
		int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		switch (ddat->set[s].type) {
		case DELAY:
			gpu_deldop_cross_section = __double2float_rd(ddat->set[s].desc.deldop.frame[f].overflow_xsec);
			gpu_deldop_cross_section += frm_xsec; // fit is the end result of parallel reduction
			gpu_deldop_cross_section *= __double2float_rd(ddat->set[s].desc.deldop.frame[f].cal.val);
			gpu_sum_rad_xsec += gpu_deldop_cross_section *
					__double2float_rd(ddat->set[s].desc.deldop.frame[f].weight);
			break;
		case DOPPLER:
			gpu_doppler_cross_section = __double2float_rd(ddat->set[s].desc.doppler.frame[f].overflow_xsec);
			gpu_doppler_cross_section += frm_xsec;
			gpu_doppler_cross_section *= __double2float_rd(ddat->set[s].desc.doppler.frame[f].cal.val);
			gpu_sum_rad_xsec += gpu_doppler_cross_section *
					__double2float_rd(ddat->set[s].desc.doppler.frame[f].weight);

			break;
		}
	}
}

__global__ void xsec_deldop_krnl(float value) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0)
		gpu_sum_rad_xsec += value;
}

__global__ void cosdelta_krnl(struct dat_t *ddat, int s, int size) {

	/* nfrm_alloc-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < size) {
		int j, view;
		double weight, cos_delta, oa[3][3], to_earth[3];

		switch(ddat->set[s].type){
		case DELAY:
			view = ddat->set[s].desc.deldop.v0;
			/* oa = matrix to transform body-fixed to observer coordinates  */
			/* to_earth = normalized target-to-Earth vector in body-fixed coords  */
			dev_mtrnsps(oa, ddat->set[s].desc.deldop.frame[f].view[view].ae);
			dev_mmmul(oa, ddat->set[s].desc.deldop.frame[f].view[view].oe, oa);
			for (j=0; j<=2; j++)
				to_earth[j] = oa[2][j];
			cos_delta = sqrt(to_earth[0]*to_earth[0] + to_earth[1]*to_earth[1]);
			weight = ddat->set[s].desc.deldop.frame[f].weight;
			gpu_sum_cos_subradarlat += cos_delta*weight;
			break;
		case DOPPLER:
			view = ddat->set[s].desc.doppler.v0;
			/* oa = matrix to transform body-fixed to observer coordinates  */
			/* to_earth = normalized target-to-Earth vector in body-fixed coords  */
			dev_mtrnsps(oa, ddat->set[s].desc.doppler.frame[f].view[view].ae);
			dev_mmmul(oa, ddat->set[s].desc.doppler.frame[f].view[view].oe, oa);
			for (j=0; j<=2; j++)
				to_earth[j] = oa[2][j];
			cos_delta = sqrt(to_earth[0]*to_earth[0] + to_earth[1]*to_earth[1]);
			weight = ddat->set[s].desc.doppler.frame[f].weight;
			gpu_sum_cos_subradarlat += cos_delta*weight;
		}
	}
}

__global__ void finalize_krnl(struct dat_t *ddat) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		if (ddat->sum_deldop_zmax_weights > 0.0)
			gpu_deldop_zmax = gpu_sum_deldop_zmax / ddat->sum_deldop_zmax_weights;
		else
			gpu_deldop_zmax = 0.0;
		if (ddat->sum_rad_xsec_weights > 0.0) {
			gpu_rad_xsec = gpu_sum_rad_xsec / ddat->sum_rad_xsec_weights;			}
		else
			gpu_rad_xsec = 0.0;
		if (ddat->sum_opt_brightness_weights > 0.0)
			gpu_opt_brightness = gpu_sum_opt_brightness / ddat->sum_opt_brightness_weights;
		else
			gpu_opt_brightness = 0.0;
		if (ddat->sum_cos_subradarlat_weights > 0.0)
			gpu_cos_subradarlat = gpu_sum_cos_subradarlat / ddat->sum_cos_subradarlat_weights;
		else
			gpu_cos_subradarlat = 0.0;
	}
}

__global__ void delay_params_krnl(struct par_t *dpar, struct dat_t *ddat,
		struct pos_t **pos, struct deldopfrm_t **frame, int *compute_xsec,
		int *posn, int *ndel, int *ndop, int s, int size) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < size) {
		compute_xsec[f] = (dpar->vary_radalb != VARY_NONE
				&& ddat->set[s].desc.deldop.frame[f].cal.state == 'c');

		pos[f] = &ddat->set[s].desc.deldop.frame[f].pos;
		posn[f] = pos[f]->n;
		ndel[f] = ddat->set[s].desc.deldop.frame[f].ndel;
		ndop[f] = ddat->set[s].desc.deldop.frame[f].ndop;
		frame[f] = &ddat->set[s].desc.deldop.frame[f];
	}
}

__global__ void dop_params_krnl(struct par_t *dpar, struct dat_t *ddat,
		struct pos_t **pos, struct dopfrm_t **frame, int *compute_xsec,
		int *posn, int *ndop, int s, int size) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < size) {
		compute_xsec[f] = (dpar->vary_radalb != VARY_NONE &&
				ddat->set[s].desc.doppler.frame[f].cal.state == 'c');
		pos[f] = &ddat->set[s].desc.doppler.frame[f].pos;
		posn[f] = pos[f]->n;
		ndop[f] = ddat->set[s].desc.doppler.frame[f].ndop;
		frame[f] = &ddat->set[s].desc.doppler.frame[f];
	}
}

__global__ void lghtcrv_params_krnl(struct par_t *dpar, struct dat_t *ddat,
		struct pos_t **pos, int *posn, int *bistatic, int s, int size) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (f <= size) {
		pos[f] = &ddat->set[s].desc.lghtcrv.rend[f].pos;
		posn[f] = pos[f]->n;
		bistatic[f] = pos[f]->bistatic;
	}
}

__global__ void set_ae_oe_krnl(struct dat_t *ddat,
		struct pos_t **pos,
		unsigned char type,
		int s,
		int size) {

	/* nfrm_alloc-threaded kernel*/
	int indx, i, j, f = blockIdx.x * blockDim.x + threadIdx.x;

	if (f < size) {
		switch (type) {
		case DELAY:
			indx = ddat->set[s].desc.deldop.v0;
			for (i=0; i<3; i++) {
				for (j=0; j<3; j++) {
					pos[f]->ae[i][j] = ddat->set[s].desc.deldop.frame[f].view[indx].ae[i][j];
					pos[f]->oe[i][j] = ddat->set[s].desc.deldop.frame[f].view[indx].oe[i][j];
					if (i==1 && j==1)
						pos[f]->bistatic = 0;
				}
			}
			break;
		case DOPPLER:
			indx = ddat->set[s].desc.doppler.v0;
			for (i=0; i<3; i++) {
				for (j=0; j<3; j++) {
					pos[f]->ae[i][j] =	ddat->set[s].desc.doppler.frame[f].view[indx].ae[i][j];
					pos[f]->oe[i][j] =	ddat->set[s].desc.doppler.frame[f].view[indx].oe[i][j];
					if (i==1 && j==1)
						pos[f]->bistatic = 0;
				}
			}
			break;
		case LGHTCRV:
			f++; /* Lightcurve offset */
			for (i=0; i<3; i++) {
				for (j=0; j<3; j++) {
					pos[f]->ae[i][j] = ddat->set[s].desc.lghtcrv.rend[f].ae[i][j];
					pos[f]->oe[i][j] = ddat->set[s].desc.lghtcrv.rend[f].oe[i][j];
					pos[f]->se[i][j] = ddat->set[s].desc.lghtcrv.rend[f].se[i][j];
					if (i==1 && j==1)
						pos[f]->bistatic = 1;
				}
			}
			break;
		}
	}
}

__global__ void get_xylim_krnl(struct par_t *dpar, struct pos_t **pos,
		int4 *xylim, int nframes) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x;
	if (f < nframes) {
		xylim[f].w = pos[f]->xlim[0];
		xylim[f].x = pos[f]->xlim[1];
		xylim[f].y = pos[f]->ylim[0];
		xylim[f].z = pos[f]->ylim[1];
	}
}

__host__ void vary_params_gpu2(
		struct par_t *dpar,
		struct mod_t *dmod,
		struct dat_t *ddat,
		int action,
		double *deldop_zmax,
		double *rad_xsec,
		double *opt_brightness,
		double *cos_subradarlat,
		int *hnframes,
		int *hlc_n,
		int *nviews,
		struct vertices_t **verts,
		unsigned char *htype,
		unsigned char *dtype,
		int nf,
		int nsets,
		hipStream_t *vp_stream,
		int max_frames)
{
	/* This third iteration uses streams that are passed as argument.
	 * It also does not calculate/copy the various parameters but accepts
	 * them as arguments. Many doubles are floats or CUDA internal types like
	 * float3 or int4.
	 * Inputs:
	 * int action 				- dpar->action
	 * double *deldop_zmax 		- &deldop_zmax_save
	 * double *rad_xsec 		- &rad_xsec_save
	 * double *optbrightness	- &opt_brightness_save
	 * double *cos_subradarlat	- &cos_subradarlat_save
	 * int nsets 				- ddat->nsets
	 */

	int c=0, f, s, *compute_brightness, *compute_zmax, *bistatic, bistatic_all,
		*compute_cosdelta, *compute_xsec, n, ncalc, nx, lghtcrv_bistatic,
		nfrm_alloc, nfrm_alloc_max, nThreads, *posn, *ndel, *ndop, compute_zmax_flag;
	nfrm_alloc_max = max_frames + 1;
	int hcomp_xsec[nfrm_alloc_max], npxls[nfrm_alloc_max], ddsize[nfrm_alloc_max],
		hndop[nfrm_alloc_max], hndel[nfrm_alloc_max], lc_xspan[nfrm_alloc_max],
		*outbndarr,	hposn[nfrm_alloc_max], hbistatic[nfrm_alloc_max],
		nThreadspx1[nfrm_alloc_max], hcomp_cosdelta[nsets], hcomp_zmax[nsets+1],
		hcomp_brightness[nsets+1];
	int2 span[nfrm_alloc_max];
	int4 *xylim, hxylim[nfrm_alloc_max];
	float zmax, *pixels_per_km, xsec[nfrm_alloc_max];
	float3 orbit_offset;
	double *u;
	double3 *so;
	struct pos_t **pos;
	struct dopfrm_t **dframe;
	struct deldopfrm_t **ddframe;

	hipEvent_t start1, stop1;
	float milliseconds;

	dim3 BLKfrm, pxBLK,THD,BLKncalc,THD9,THD64, BLKpx1, BLK[nfrm_alloc_max], ddBLK[nfrm_alloc_max];
	THD.x = maxThreadsPerBlock;	THD9.x = 9; THD64.x = 64;

	/* Initialize */
	orbit_offset.x = orbit_offset.y = orbit_offset.z = 0.0;

	/* Some arrays are allocated for the maximum number of frames in
	 * any one set.  That way they are allocated once and deallocated once.
	 * They are re-used for each loop.	 */
	gpuErrchk(hipMalloc((void**)&compute_brightness, sizeof(int) * (nsets+1)));
	gpuErrchk(hipMalloc((void**)&compute_zmax, sizeof(int) * (nsets+1)));
	gpuErrchk(hipMalloc((void**)&compute_cosdelta, sizeof(int) * (nsets+1)));
	gpuErrchk(hipMalloc((void**)&pos, 		   sizeof(struct pos_t*) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&posn, 	   sizeof(int) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&ndel, 	   sizeof(int) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&ndop, 	   sizeof(int) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&outbndarr,   sizeof(int) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&compute_xsec,sizeof(int) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&xylim, 	   sizeof(int4)* nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&so, sizeof(double3)*(nfrm_alloc_max*3)));
	gpuErrchk(hipMalloc((void**)&pixels_per_km, sizeof(int)*nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&u, sizeof(double) * nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&dframe, sizeof(struct dopfrm_t*)*nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&ddframe, sizeof(struct deldopfrm_t*)*nfrm_alloc_max));
	gpuErrchk(hipMalloc((void**)&bistatic, sizeof(int)*nfrm_alloc_max));

	/* Initialize the device file-scope variables */
	init_krnl<<<1,1>>>(dpar, ddat, compute_zmax, compute_cosdelta,
			compute_brightness, dtype,nsets);
	checkErrorAfterKernelLaunch("vpst_init_krnl3");

	/* Process each dataset in turn */
	for (s=0; s<nsets; s++) {
		/* Get the allocation right as the indices for lghtcrv start with 1
		 * instead of 0 everywhere else. To not run into problems at the end
		 * or start of the array, we allocate one more than strictly needed */
		if (htype[s]==LGHTCRV)			nfrm_alloc = hnframes[s] + 1;
		else							nfrm_alloc = hnframes[s];

		/* Set up initial kernel launch parameter */
		BLK[0].x = floor((THD.x - 1 + nfrm_alloc) / THD.x);
		BLKfrm = floor((THD64.x - 1 + nfrm_alloc) / THD64.x);

		switch (htype[s]) {
		case DELAY:
			/* Get computation flags */
			gpuErrchk(hipMemcpy(&hcomp_cosdelta, compute_cosdelta,
					sizeof(int)*(nsets+1), hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(&hcomp_zmax, compute_zmax,
					sizeof(int)*(nsets+1), hipMemcpyDeviceToHost));

			/* Launch nframes-threaded kernel to get all relevant parameters */
			delay_params_krnl<<<BLK[0],THD>>>(dpar, ddat, pos, ddframe,
					compute_xsec, posn, ndel, ndop, s, nfrm_alloc);
			checkErrorAfterKernelLaunch("vpst_delay_params_krnl");
			gpuErrchk(hipMemcpy(&hposn, posn, sizeof(int)*nfrm_alloc,
					hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(&hcomp_xsec, compute_xsec,
					sizeof(int)*nfrm_alloc, hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(&hndop, ndop, sizeof(int)*nfrm_alloc,
					hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(&hndel, ndel, sizeof(int)*nfrm_alloc,
					hipMemcpyDeviceToHost));

			/* Create streams and calculate parameters*/
			for (f=0; f<nfrm_alloc; f++) {
				npxls[f] = (2*hposn[f] + 1)*(2*hposn[f] + 1);
				BLK[f].x = floor((THD.x - 1 + npxls[f]) / THD.x);
				ddsize[f]= hndel[f] * hndop[f];
				ddBLK[f] = floor((THD.x -1 + ddsize[f]) / THD.x);
			}

			/* Assign ae and oe matrices (all frames at once) */
			if (hcomp_zmax[s])
				set_ae_oe_krnl<<<BLKfrm,THD64,0,vp_stream[0]>>>(ddat, pos, htype[s], s, nfrm_alloc);
			checkErrorAfterKernelLaunch("set_ae_oe_krnl in vary_params_gpu2");

			/* Launch posclr_streams_krnl to initialize POS view */
			for (f=0; f<nfrm_alloc; f++) {
				/* Start the if block for computing zmax and/or cross-section */
				if (hcomp_zmax[s] || hcomp_xsec[f]) {
					posclr_krnl<<<BLK[f],THD, 0, vp_stream[f]>>>(pos,posn,f,0);
				}
			} checkErrorAfterKernelLaunch("posclr_krnl (Delay-Doppler in vary_params_gpu2)");

			/* Determine which POS pixels cover the target, and get distance
			 * toward Earth of each POS pixel. Pass the frame streams, too. */
			posvis_gpu2(dpar, dmod, ddat, pos, verts, orbit_offset,
					hposn, outbndarr, s, nfrm_alloc, 0, nf, 0, c, htype[s],
					vp_stream);

			for (f=0; f<nfrm_alloc; f++) {
				if (hcomp_zmax[s] || hcomp_xsec[f]) {
					/* Zero out the fit delay-Doppler image and call pos2deldop
					 * to create the fit image by mapping power from the plane
					 * of sky to delay-Doppler space.    				  */
					clrvect_krnl<<<ddBLK[f],THD, 0, vp_stream[f]>>>(ddat,
							ddsize[f], s, f);
				}/* End frames loop again to call pos2deldop streams version */
			} checkErrorAfterKernelLaunch("clrvect_streams_krnl");

			/* Call the CUDA pos2deldop function */
			pos2deldop_gpu2(dpar, dmod, ddat, pos, ddframe, xylim, ndel, ndop,
					0.0, 0.0, 0.0, 0, s, nfrm_alloc, 0, outbndarr, vp_stream);

			/* Calculate zmax for all frames (assumption: all pos in this set
			 * have the same pixel dimensions) */
			if (hcomp_zmax[s]) {
				zmax = compute_zmax_gpu(ddat, pos, nfrm_alloc, npxls[0], s, vp_stream);
				zmax_final_krnl<<<1,1>>>(zmax);
				checkErrorAfterKernelLaunch("zmax_finalize_streams2_krnl");
			}

			if (TIMING) {
				/* Create the timer events */
				hipEventCreate(&start1);
				hipEventCreate(&stop1);
				hipEventRecord(start1);
			}
			/* Calculate radar cross section for each frame in set */
			xsec[0] = compute_deldop_xsec_gpu(ddat, hnframes[s], ddsize[0], s, vp_stream);
			xsec_deldop_krnl<<<1,1>>>(xsec[0]);
			checkErrorAfterKernelLaunch("compute_xsec_final_streams2_krnl");

			if (TIMING) {
				hipEventRecord(stop1);
				hipEventSynchronize(stop1);
				milliseconds = 0;
				hipEventElapsedTime(&milliseconds, start1, stop1);
				printf("Deldop xsec_streams: %3.3f ms with %i frames.\n", milliseconds, hnframes[s]);
				hipEventDestroy(start1);
				hipEventDestroy(stop1);
			}

			if (hcomp_cosdelta[s])
				cosdelta_krnl<<<BLKfrm,THD64>>>(ddat, s, f);
			checkErrorAfterKernelLaunch("compute_cosdelta_streams_krnl");

			break;
		case DOPPLER:
			/* Get computation flags */
			gpuErrchk(hipMemcpy(&hcomp_cosdelta, compute_cosdelta,
					sizeof(int)*(nsets+1), hipMemcpyDeviceToHost));

			/* Launch nframes-threaded kernel to get all relevant parameters */
			dop_params_krnl<<<BLK[0],THD>>>(dpar, ddat, pos, dframe,
					compute_xsec, posn, ndop, s, nfrm_alloc);
			checkErrorAfterKernelLaunch("vpst_dop_params_krnl");
			gpuErrchk(hipMemcpy(&hposn, posn, sizeof(int)*nfrm_alloc,
					hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(&hcomp_xsec, compute_xsec,
					sizeof(int)*nfrm_alloc, hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(&hndop, ndop, sizeof(int)*nfrm_alloc,
					hipMemcpyDeviceToHost));

			/* Calculate launch parameters and create streams */
			for (f=0; f<nfrm_alloc; f++) {
				npxls[f] = (2*hposn[f] + 1)*(2*hposn[f] + 1);
				BLK[f].x = floor((THD.x - 1 + npxls[f]) / THD.x);
				ddBLK[f] = floor((THD.x -1 + hndop[f]) / THD.x);
			}

			/* Assign ae and oe matrices (all frames at once) */
			set_ae_oe_krnl<<<BLKfrm,THD64,0,vp_stream[0]>>>(ddat, pos, htype[s], s, nfrm_alloc);
			checkErrorAfterKernelLaunch("set_ae_oe_krnl in vary_params_gpu2");

			/* Launch posclr_streams_krnl to initialize POS view */
			for (f=0; f<nfrm_alloc; f++) {
				/* Start the if block for computing zmax and/or cross-section */
				if (hcomp_xsec[f]) {
					posclr_krnl<<<BLK[f],THD, 0, vp_stream[f]>>>(pos,posn,f,0);
				}
			} checkErrorAfterKernelLaunch("posclr_krnl (Doppler in vary_params_gpu2)");

			/* Determine which POS pixels cover the target, and get distance
			 * toward Earth of each POS pixel. Pass the frame streams, too. */
			posvis_gpu2(dpar, dmod, ddat, pos, verts, orbit_offset,
					hposn, outbndarr, s, nfrm_alloc, 0, nf, 0, c, htype[s],
					vp_stream);

			for (f=0; f<nfrm_alloc; f++) {
				if (hcomp_xsec[f]) {
					/* Zero out the fit delay-Doppler image and call pos2deldop
					 * to create the fit image by mapping power from the plane
					 * of sky to delay-Doppler space.    				      */
					clrvect_krnl<<<ddBLK[f],THD, 0, vp_stream[f]>>>(ddat,
							hndop[f], s, f);
					/* End frames loop again to call pos2deldop streams version */
				}
			} checkErrorAfterKernelLaunch("clrvect_streams_krnl");

			pos2doppler_gpu2(dpar, dmod, ddat, pos, dframe, xylim, 0.0,
					0.0, 0.0, ndop, 0, s, hnframes[s], 0, outbndarr, vp_stream);

			/* Calculate the Doppler cross-section if applicable */
			for (f=0; f<nfrm_alloc; f++) {
				if (hcomp_xsec[f]) {
					/* Compute cross section */
					xsec[f]=0.0;
					xsec[f] = compute_doppler_xsec(ddat, hndop[f], s, f);
				}
			}
			/* Finalize the xsec calculations and calculate cosdelta if specified */
			for (f=0; f<nfrm_alloc; f++) {
				if (hcomp_xsec[f])
					xsec_doppler_krnl<<<1,1,0,vp_stream[f]>>>(ddat, xsec[f], s, f);
			}
			if (compute_cosdelta)
				cosdelta_krnl<<<BLKfrm,THD64,0,vp_stream[0]>>>(ddat, s, nfrm_alloc);

			break;
		case POS:
			break;
		case LGHTCRV:
			/* Figure out the compute_brightness flag first */
			gpuErrchk(hipMemcpy(&hcomp_brightness, compute_brightness,
					sizeof(int)*(nsets+1), hipMemcpyDeviceToHost));

			if (hcomp_brightness[s]) {
				/* Launch nframes-threaded kernel to get all relevant parameters */
				lghtcrv_params_krnl<<<BLK[0],THD>>>(dpar, ddat, pos,
						posn, bistatic, s, nfrm_alloc);
				checkErrorAfterKernelLaunch("vpst_lghtcrv_params_krnl");
				gpuErrchk(hipMemcpy(&hposn, posn, sizeof(int)*nfrm_alloc,
						hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpy(&hbistatic, bistatic, sizeof(int)*nfrm_alloc,
						hipMemcpyDeviceToHost));

				/* Calculate launch parameters and create streams */
				for (f=1; f<nfrm_alloc; f++) {
					lc_xspan[f] = 2*posn[f] + 1;
					npxls[f] = (2*posn[f]+1)*(2*posn[f]+1);
					BLK[f].x = floor((THD.x - 1 + npxls[f]) / THD.x);
				}

				/* Assign ae and oe matrices (all frames at once) */
				set_ae_oe_krnl<<<BLKfrm,THD64,0,vp_stream[0]>>>(ddat, pos, htype[s], s, nfrm_alloc);
				checkErrorAfterKernelLaunch("set_ae_oe_krnl in vary_params_gpu2");

				/* Launch posclr_streams_krnl to initialize POS view */
				for (f=1; f<nfrm_alloc; f++) {
					/* Start the if block for computing zmax and/or cross-section */
					if (hcomp_xsec[f]) {
						posclr_krnl<<<BLK[f],THD, 0, vp_stream[f]>>>(pos,posn,f,0);
					}
				} checkErrorAfterKernelLaunch("posclr_krnl (Doppler in vary_params_gpu2)");

				/* Determine which POS pixels cover the target */
				posvis_gpu2(dpar, dmod, ddat, pos, verts, orbit_offset,
						hposn, outbndarr, s, hnframes[s], 0, nf, 0, c, htype[s],
						vp_stream);

				/* Now view the model from the source (sun) and get the facet
				 * number and distance toward the source of each pixel in this
				 * projected view; use this information to determine which POS
				 * pixels are shadowed */
				for (f=1; f<nfrm_alloc; f++)
					if (hbistatic[f])	bistatic_all = 1;

				if (bistatic_all)
					posvis_gpu2(dpar, dmod, ddat, pos, verts,
							orbit_offset, hposn, outbndarr, s, hnframes[s], 1,
							nf, 0, c, htype[s],	vp_stream);

				if (bistatic_all) {
					posmask_init_krnl2<<<BLKfrm,THD64>>>(pos, so, pixels_per_km, nfrm_alloc);
					checkErrorAfterKernelLaunch("posmask_init_krnl in vary_params_gpu2");

					for (f=1; f<nfrm_alloc; f++) {
						/* Now call posmask kernel for this stream, then loop
						 * to next stream and repeat 	 */
						posmask_krnl<<<BLK[f],THD,0,vp_stream[f-1]>>>(
								dpar, pos, so, pixels_per_km, posn, npxls[f],
								lc_xspan[f], f);
					} checkErrorAfterKernelLaunch("posmask_krnl in vary_params_gpu2");
				}

				BLKpx1.x = floor((THD.x - 1 + hnframes[s])/THD.x);
				get_xylim_krnl<<<BLKpx1,THD>>>(dpar, pos, xylim, hnframes[s]);
				checkErrorAfterKernelLaunch("vpst_get_xylim_krnl");
				gpuErrchk(hipMemcpy(&hxylim, xylim, sizeof(int4)*(hnframes[s]+1),
						hipMemcpyDeviceToHost));

				/* Calculate launch parameters for all frames */
				for (f=1; f<=hnframes[s]; f++) {
					span[f].x = hxylim[f].x - hxylim[f].w + 1;
					span[f].y = hxylim[f].z - hxylim[f].y + 1;
					nThreadspx1[f] = span[f].x * span[f].y;
					BLK[f].x = floor ((THD.x -1 + nThreadspx1[f]) / THD.x);
				}

				/* Compute model brightness for this lightcurve point */
				/* lghtcrv->y[ncalc]: calculated points for interpolation,
				 * ncalc-points total 					 */
				apply_photo_cuda_streams_f(dmod, ddat, pos, xylim, span, BLK, nThreadspx1,
							0, s, hnframes[s], npxls, vp_stream);

				/* Now that we have calculated the model lightcurve brightnesses
				 * y at each of the epochs x, we use cubic spline interpolation
				 * (Numerical Recipes routines spline and splint) to get model
				 * lightcurve brightness fit[i] at each OBSERVATION epoch t[i],
				 * with i=1,2,...,n.  This will allow us (in routine chi2) to
				 * compare model to data (fit[i] to obs[i]) to get chi squared.
				 * Note that vector y2 contains the second derivatives of the
				 * interpolating function at the calculation epochs x. */

				/* First make a pointer for u and hipMalloc device memory for it */
				gpuErrchk(hipMemset(u, 0, nfrm_alloc_max*sizeof(double)));

				lghtcrv_spline_streams_test_krnl<<<BLKncalc,THD>>>(ddat, s, 2.0e30,
						2.0e30, u, hnframes[s]);
				checkErrorAfterKernelLaunch("lghtcrv_spline_streams_krnl");

				/* Change launch parameters from ncalc threads to n threads */
				//BLKncalc.x = floor((THD.x - 1 + hlc_n[s]) / THD.x);
				lghtcrv_splint_streams3_test_krnl<<<1,1>>>(ddat, s, hlc_n[s], hnframes[s]);
				checkErrorAfterKernelLaunch("lghtcrv_splint_streams_krnl");
				/* Cleanup */

			}
			break;
		default:
			bailout("vary_params.c: can't handle this dataset type yet\n");
		}
	}

	/* Calculate the zmax, radar cross-section, optical brightness, and cosine
	 * subradar latitude */
	double dd_zmax, rd_xsec, opt_brtns, cs_sb_rdr_lat;
	finalize_krnl<<<1,1>>>(ddat);
	checkErrorAfterKernelLaunch("vps_set_four_parameters, line ");
	gpuErrchk(hipMemcpyFromSymbol(&dd_zmax, HIP_SYMBOL(gpu_deldop_zmax),
			sizeof(double), 0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&rd_xsec, HIP_SYMBOL(gpu_rad_xsec),
			sizeof(double), 0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&opt_brtns, HIP_SYMBOL(gpu_opt_brightness),
			sizeof(double), 0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&cs_sb_rdr_lat, HIP_SYMBOL(gpu_cos_subradarlat),
			sizeof(double), 0, hipMemcpyDeviceToHost));

	*deldop_zmax = dd_zmax;
	*rad_xsec = rd_xsec;
	*opt_brightness = opt_brtns;
	*cos_subradarlat = cs_sb_rdr_lat;

	hipFree(u);
	hipFree(so);
	hipFree(pixels_per_km);
	hipFree(pos);
	hipFree(posn);
	hipFree(ndel);
	hipFree(ndop);
	hipFree(outbndarr);
	hipFree(compute_xsec);
	hipFree(xylim);
	hipFree(compute_brightness);
	hipFree(compute_zmax);
	hipFree(compute_cosdelta);
	hipFree(dframe);
	hipFree(ddframe);
}
