#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                            apply_photo.c

For each plane-of-sky pixel, compute the model's scattered optical power per unit
projected (POS) area per unit solid angle per unit incident flux, and then sum these
values over the entire POS.  (The POS pixel area is multiplied in elsewhere.)

The expressions given here differ from the bidirectional reflectance functions defined by,
say, Hapke 1993: bidirectional reflectance includes an extra factor of
cos(scattering angle), since it is defined per unit surface area rather than per unit
projected area.

Modified 2014 February 12 by CM:
    Implement multiple optical scatering laws

Modified 2011 September 2 by CM:
    Add the "harmlambert" and "inholambert" optical scattering laws

Modified 2007 August 4 by CM:
    Add body parameter for use with the "orbit" action: it denotes which
        orbiting body's optical power contributions are being computed
        on this call to the routine
    Don't zero out blank-sky and shadowed POS pixels in the sky rendering
        (the pos->b matrix): do it instead in the calling routine by
        having it call the posclr routine.  This way apply_photo can be
        called twice for the "orbit" action, once for each orbiting body.
    Add comp matrix for POS frames

Modified 2006 October 1 by CM:
    Add "intensity_factor" parameter: account for POS pixel area,
        1 AU Sun-target distance, and solar apparent magnitude here
        rather than after calling the routine

Modified 2006 September 1 by CM and MCN:
    For inhomogeneous laws, add check that facet number pos->f[i][j]
        is nonnegative

Modified 2005 September 7 by CM:
    Implement the "harmlommel" "harmhapke" and "harmkaas" optical
        scattering laws

Modified 2005 August 8 by CM:
    Implement the "inhokaas" optical scattering law
    Add some (cosi > 0) checks
    Move "sum == 0" check to the end

Modified 2005 July 4 by CM:
    Changed structure name for the INHOLOMMEL optical scattering law

Modified 2005 March 1 by CM:
    Add NOLAW case

Modified 2005 January 25 by CM:
    Eliminate unused variables

Modified 2004 April 29 by CM:
    Modify Kaasalainen scattering law to use "wt" as the relative
        weighting factor (0 = pure Lommel-Seeliger, 1 = pure Lambert)
        rather than "c" (which ranged from 0 to infinity)

Modified 2004 March 25 by CM:
    hapke routine now takes phase rather than cos(phase) as argument

Modified 2004 February 29 by CM:
    Added comments
    Added Kaasalainen "Lommel-Seeliger + Lambert" scattering law
    Eliminated "type" argument, since this routine was only being
       used to handle optical scattering.  (Radar scattering is
       instead handled by the "radlaw" routine.)
    Added "phase" argument (solar phase angle) so that we can compute
       the phase just once per calculated lightcurve point (in read_dat)
       rather than computing it every time we call apply_photo
*****************************************************************************************/
extern "C" {
#include "../shape/head.h"
}

#define TINY 1.0e-40

__device__ int ap_ilaw;

__global__ void ap_init_krnl32(struct dat_t *ddat, struct mod_t *dmod,
        struct pos_t **pos, int set, int nframes, unsigned char *type,
        float *dsum, float *intensity_factor, float *phase) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x + 1;
	double temp;
	if (f <= nframes) {
		ap_ilaw = ddat->set[set].desc.lghtcrv.ioptlaw;
		type[0] = dmod->photo.opttype[ap_ilaw];
		type[1] = 0;
		dsum[f] = 0.0;
		temp = pos[f]->km_per_pixel/AU;
		temp *= temp;
		intensity_factor[f] = __double2float_rn(temp);
		phase[f] = __double2float_rn(ddat->set[set].desc.lghtcrv.solar_phase[f]);
	}
}
__global__ void ap_init_krnl64(struct dat_t *ddat, struct mod_t *dmod,
		struct pos_t **pos, int set, int nframes, unsigned char *type,
        double *dsum, double *intensity_factor,	double *phase) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x + 1;
	double temp;
	if (f <= nframes) {
		ap_ilaw = ddat->set[set].desc.lghtcrv.ioptlaw;
		type[0] = dmod->photo.opttype[ap_ilaw];
		type[1] = 0;
		dsum[f] = 0.0;
		temp = pos[f]->km_per_pixel/AU;
		intensity_factor[f] = temp*temp;
		phase[f] = ddat->set[set].desc.lghtcrv.solar_phase[f];
	}
}
__global__ void ap_lambertlaw_krnl32(struct mod_t *dmod, struct pos_t **pos,
		float *intensity_factor, int4 *xylim, int nThreads, int body,
		int2 span, int f) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[f].w;
	int j = offset / span.x + xylim[f].y;
	float scale;

	if (offset < nThreads) {
		scale = __double2float_rn(dmod->photo.optical[ap_ilaw].R.R.val/PIE);

		if (pos[f]->cose_s[offset] > 0.0 && pos[f]->cosi_s[offset] > 0.0
				&& pos[f]->body[i][j] == body) {
			pos[f]->b[i][j] = intensity_factor[f] * scale * pos[f]->cosi_s[offset];
		}
	}
}
__global__ void ap_lambertlaw_krnl64(struct mod_t *dmod, struct pos_t **pos,
		double *intensity_factor, int4 *xylim, int nThreads, int body,
		int2 span, int f) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[f].w;
	int j = offset / span.x + xylim[f].y;
	double scale;

	if (offset < nThreads) {
		scale = dmod->photo.optical[ap_ilaw].R.R.val/PIE;

		if (pos[f]->cose[i][j] > 0.0 && pos[f]->cosi[i][j] > 0.0
				&& pos[f]->body[i][j] == body) {
			pos[f]->b[i][j] = intensity_factor[f] * scale * pos[f]->cosi[i][j];
		}
	}
}
__global__ void ap_harmlambert_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,	float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float scale;

	if (offset < nThreads) {

		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val)/PIE;
			pos[frm]->b[i][j] = intensity_factor[frm] * scale *
					pos[frm]->cosi_s[offset];
		}
	}
}
__global__ void ap_harmlambert_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span, double *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int c, f ,offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double scale;

	if (offset < nThreads) {

		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val/PIE;
			pos[frm]->b[i][j] = intensity_factor[frm] * scale *	pos[frm]->cosi[i][j];
		}
	}
}
__global__ void ap_inholambert_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span, float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int c ,f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float scale;

	if (offset < nThreads) {

		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val)/PIE;
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi_s[offset];
		}
	}
}
__global__ void ap_inholambert_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        double *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int c ,f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double scale;

	if (offset < nThreads) {

		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val/PIE;
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi[i][j];
		}
	}
}
__global__ void ap_lommel_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		float *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float scale;

	if (offset < nThreads) {
		scale = __double2float_rn(dmod->photo.optical[ap_ilaw].R.R.val)/(4*PIE);
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
				&& pos[frm]->body[i][j] == body) {
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi_s[offset]
			  / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
		}
	}
}
__global__ void ap_lommel_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        double *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double scale;

	if (offset < nThreads) {
		scale = dmod->photo.optical[ap_ilaw].R.R.val/(4*PIE);
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
				&& pos[frm]->body[i][j] == body) {
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi[i][j]
			  / (pos[frm]->cosi[i][j] + pos[frm]->cose[i][j]);
		}
	}
}
__global__ void ap_harmlommel_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        float *intensity_factor, int frm) {

	/* Multi-threaded kernel */
	int c ,f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float scale;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val)/(4*PIE);
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi_s[offset]
			   / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
		}
	}
}
__global__ void ap_harmlommel_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        double *intensity_factor, int frm) {

	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double scale;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val/(4*PIE);
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi[i][j]
			   / (pos[frm]->cosi[i][j] + pos[frm]->cose[i][j]);
		}
	}
}
__global__ void ap_inholommel_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        float *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float scale;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val)/(4*PIE);
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi_s[offset]
			   / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
		}
	}
}
__global__ void ap_inholommel_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        double *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double scale;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val/(4*PIE);
			pos[frm]->b[i][j] = intensity_factor[frm] * scale * pos[frm]->cosi[i][j]
			   / (pos[frm]->cosi[i][j] + pos[frm]->cose[i][j]);
		}
	}
}
__global__ void ap_geometrical_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        float *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b[i][j] = intensity_factor[frm] * dmod->photo.optical[ap_ilaw].R.R.val;
		}
	}
}
__global__ void ap_geometrical_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        double *intensity_factor, int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b[i][j] = intensity_factor[frm] * dmod->photo.optical[ap_ilaw].R.R.val;
		}
	}
}
__global__ void ap_hapke_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        float *intensity_factor, float *phase, int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b[i][j] = intensity_factor[frm]
					* dev_hapke_f(pos[frm]->cosi_s[offset],
							pos[frm]->cose_s[offset],
							phase[frm],
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.w.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.h.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.B0.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.g.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.theta.val));

		}
	}
}
__global__ void ap_hapke_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		double *intensity_factor, double *phase, int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b[i][j] = intensity_factor[frm]
					* dev_hapke(pos[frm]->cosi[i][j],
							pos[frm]->cose[i][j],
							phase[frm],
							dmod->photo.optical[ap_ilaw].hapke.w.val,
							dmod->photo.optical[ap_ilaw].hapke.h.val,
							dmod->photo.optical[ap_ilaw].hapke.B0.val,
							dmod->photo.optical[ap_ilaw].hapke.g.val,
							dmod->photo.optical[ap_ilaw].hapke.theta.val);

		}
	}
}
__global__ void ap_harmhapke_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
        float *intensity_factor, float *phase, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
	     && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			pos[frm]->b[i][j] = intensity_factor[frm]
			            *dev_hapke_f(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset], phase[frm],
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].w.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].h.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].B0.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].g.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].theta.val));
		}
	}
}
__global__ void ap_harmhapke_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		double *intensity_factor, double *phase, int frm) {
	/* Multi-threaded kernel */
	int c ,f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
	     && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			pos[frm]->b[i][j] = intensity_factor[frm]
			            *dev_hapke(pos[frm]->cosi[i][j], pos[frm]->cose[i][j],
			            phase[frm],
			            dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].w.val,
			            dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].h.val,
			            dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].B0.val,
			            dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].g.val,
			            dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].theta.val);
		}
	}
}
__global__ void ap_inhohapke_krnl32(struct mod_t *dmod,	struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		float *intensity_factor, float *phase, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			pos[frm]->b[i][j] = intensity_factor[frm]
					* dev_hapke_f(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset],
							phase[frm],
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].w.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].h.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].B0.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].g.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].theta.val));
		}
	}
}
__global__ void ap_inhohapke_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		double *intensity_factor, double *phase, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			pos[frm]->b[i][j] = intensity_factor[frm]
					* dev_hapke(pos[frm]->cosi[i][j], pos[frm]->cose[i][j],	phase[frm],
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].w.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].h.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].B0.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].g.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].theta.val);
		}
	}
}
__global__ void ap_kaas_init_krnl32(struct mod_t *dmod,
		float *phasefunc, float *phase, float *scale_lommsee,
		float *scale_lambert, int nframes) {
	/* nframes-threaded kernel */
	int frm = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (frm <= nframes) {
		phasefunc[frm] = dmod->photo.optical[ap_ilaw].kaas.A0.val
				* exp( -phase[frm] / dmod->photo.optical[ap_ilaw].kaas.D.val)
		+ dmod->photo.optical[ap_ilaw].kaas.k.val * phase[frm] + 1;

		scale_lommsee[frm] = (1 - dmod->photo.optical[ap_ilaw].kaas.wt.val)
				* phasefunc[frm] * dmod->photo.optical[ap_ilaw].kaas.R.val/(4*PIE);
		scale_lambert[frm] = dmod->photo.optical[ap_ilaw].kaas.wt.val
				* phasefunc[frm] * dmod->photo.optical[ap_ilaw].kaas.R.val/PIE;
	}
}
__global__ void ap_kaas_init_krnl64(struct mod_t *dmod, double *phasefunc,
        double *phase, double *scale_lommsee, double *scale_lambert, int nframes) {
	/* nframes-threaded kernel */
	int frm = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (frm <= nframes) {
		phasefunc[frm] = dmod->photo.optical[ap_ilaw].kaas.A0.val
				* exp( -phase[frm] / dmod->photo.optical[ap_ilaw].kaas.D.val)
		+ dmod->photo.optical[ap_ilaw].kaas.k.val * phase[frm] + 1;

		scale_lommsee[frm] = (1 - dmod->photo.optical[ap_ilaw].kaas.wt.val)
				* phasefunc[frm] * dmod->photo.optical[ap_ilaw].kaas.R.val/(4*PIE);
		scale_lambert[frm] = dmod->photo.optical[ap_ilaw].kaas.wt.val
				* phasefunc[frm] * dmod->photo.optical[ap_ilaw].kaas.R.val/PIE;
	}
}
__global__ void ap_kaas_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		float *intensity_factor, float *phase, float *phasefunc,
		float *scale_lommsee, float *scale_lambert, int frm, int nframes) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int n = pos[frm]->n;
	int pxa = (j+n)*(2*n+1) + (i+n);

	if (offset < nThreads) {
		if (pos[frm]->cose_s[pxa] > 0.0 && pos[frm]->cosi_s[pxa] > 0.0
				&& pos[frm]->body[i][j] == body) {

			pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi_s[pxa]
			   *(scale_lommsee[frm] / (pos[frm]->cosi_s[pxa] + pos[frm]->cose_s[pxa])
			 + scale_lambert[frm]);
		}
	}
}
__global__ void ap_kaas_krnl48(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,	double *intensity_factor,
		double *phase, double *phasefunc, double *scale_lommsee, double
		*scale_lambert, int frm, int nframes, int s) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int n = pos[frm]->n;
	int pxa = (j+n)*(2*n+1) + (i+n);

	if (offset < nThreads) {
		if (pos[frm]->cose_s[pxa] > 0.0 && pos[frm]->cosi_s[pxa] > 0.0
				&& pos[frm]->body[i][j] == body) {

			pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi_s[pxa]
			*(scale_lommsee[frm] / (pos[frm]->cosi_s[pxa] + pos[frm]->cose_s[pxa])
			 + scale_lambert[frm]);
		}
	}
}
__global__ void ap_kaas_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body,	int4 *xylim, int2 span,
        double *intensity_factor, double *phase,	double *phasefunc,
        double *scale_lommsee, double *scale_lambert, int frm, int nframes) {
			/* Multi-threaded kernel */
			int offset = blockIdx.x * blockDim.x + threadIdx.x;
			int i = offset % span.x + xylim[frm].w;
			int j = offset / span.x + xylim[frm].y;

            if (offset < nThreads) {
				if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
						&& pos[frm]->body[i][j] == body) {

					pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi[i][j]
			       *(scale_lommsee[frm] / (pos[frm]->cosi[i][j] + pos[frm]->cose[i][j])
			  		  + scale_lambert[frm]);
				}
			}
}
__global__ void ap_harmkaas_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span,
		float *intensity_factor, float *phase,	int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float phasefunc, scale_lommsee, scale_lambert;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefunc = __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].A0.val)
			* exp( -phase[frm] / __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].D.val))
			+ __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].k.val) * phase[frm] + 1;

			scale_lommsee = (1 - __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val))
		    * phasefunc * __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val) / (4*PIE);
			scale_lambert = __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val)
			* phasefunc * __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val) / PIE;
			 pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi_s[offset] * (scale_lommsee /
					( pos[frm]->cosi_s[offset] +  pos[frm]->cose_s[offset]) + scale_lambert);
		}
	}
}
__global__ void ap_harmkaas_krnl64(struct mod_t *dmod, struct pos_t **pos,
        int nThreads, int body, int4 *xylim, int2 span,
		double *intensity_factor, double *phase, int frm) {
	/* Multi-threaded kernel */
	int c ,f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double phasefunc, scale_lommsee, scale_lambert;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefunc = dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].A0.val
			* exp( -phase[frm] / dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].D.val)
			+ dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].k.val * phase[frm] + 1;

			scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val)
		    * phasefunc * dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val / (4*PIE);
			scale_lambert = dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val
			* phasefunc * dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val / PIE;
			 pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi[i][j] * (scale_lommsee /
					( pos[frm]->cosi[i][j] +  pos[frm]->cose[i][j]) + scale_lambert);
		}
	}
}
__global__ void ap_inhokaas_krnl32(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span, float *intensity_factor,
		float *phase, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float phasefunc, scale_lommsee, scale_lambert;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefunc = __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].A0.val)
			* exp( -phase[frm] / __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].D.val))
			+ __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].k.val) * phase[frm] + 1;
			scale_lommsee = (1 - __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val))
		    * phasefunc * __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val) / (4*PIE);
			scale_lambert = __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val)
			* phasefunc * __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val) / PIE;
			pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi_s[offset] * (scale_lommsee /
					(pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]) + scale_lambert);
		}
	}
}
__global__ void ap_inhokaas_krnl64(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int4 *xylim, int2 span, double *intensity_factor,
		double *phase, int frm) {
	/* Multi-threaded kernel */
	int c, f, offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double phasefunc, scale_lommsee, scale_lambert;

	if (offset < nThreads) {
		if (pos[frm]->cose[i][j] > 0.0 && pos[frm]->cosi[i][j] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefunc = dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].A0.val
			* exp( -phase[frm] / dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].D.val)
			+ dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].k.val * phase[frm] + 1;
			scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val)
		    * phasefunc * dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val / (4*PIE);
			scale_lambert = dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val
			* phasefunc * dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val / PIE;
			pos[frm]->b[i][j] = intensity_factor[frm] * pos[frm]->cosi[i][j] * (scale_lommsee /
					(pos[frm]->cosi[i][j] + pos[frm]->cose[i][j]) + scale_lambert);
		}
	}
}

__host__ void apply_photo_gpu32(struct mod_t *dmod,	struct dat_t *ddat,
		struct pos_t **pos, int4 *xylim, int2 *span, dim3 *BLKpx, int *nThreads,
		int body, int set, int nframes, int *nThreadspx, int maxthds,
		int4 maxxylim, hipStream_t *ap_stream)
{
	/* The following function launches kernels to calculate the per-pixel brightness
	 * of the model asteroid.  None of kernels examine ALL of the pixels in any
	 * one pos; rather only the area defined by pos->xlim and pos->ylim.  In this
	 * context, the input argument span contains the x and y direction spans for
	 * each frame's pos.  BLKpx defines the number of blocks needed for those
	 * kernels.  nThreadspx defines the number of pixels calculated for each
	 * frame's pos.  nThreads defines the full number of pixels per pos.	 */

	unsigned char *type, *htype;
	int f;
	float *dsum;
	double *hsum, *sum;
	float *intensity_factor, *phase, *phasefunc, *scale_lommsee, *scale_lambert;
	dim3 BLK, THD;

	gpuErrchk(hipMalloc((void**)&type, sizeof(unsigned char) * 2));
	gpuErrchk(hipMalloc((void**)&sum, sizeof(double) * (nframes+1)));
	gpuErrchk(hipMalloc((void**)&dsum, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&intensity_factor, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&phase, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&scale_lommsee, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&scale_lambert, sizeof(float)*(nframes+1)));
	htype = (unsigned char *) malloc(2*sizeof(unsigned char));
	hsum = (double *) malloc((nframes+1)*sizeof(double));

	/* Launch single-thread kernel to assign pos address and get type */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nframes) / THD.x);
	ap_init_krnl32<<<BLK,THD>>>(ddat, dmod, pos, set, nframes, type, dsum,
			intensity_factor, phase);
	checkErrorAfterKernelLaunch("ap_init_krnl32");
	gpuErrchk(hipMemcpy(htype, type, sizeof(unsigned char) *2,
			hipMemcpyDeviceToHost));

	switch (htype[0]) {
	case LAMBERTLAW:
		/* Launch Lambert Law kernel */
		for (f=1; f<=nframes; f++)
			ap_lambertlaw_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,pos,
					intensity_factor, xylim, nThreadspx[f], body, span[f], f);
		checkErrorAfterKernelLaunch("ap_lambertlaw_krnl32");
		break;
	case HARMLAMBERT:
		/* Launch the HarmLambert kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlambert_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(
					dmod, pos, nThreadspx[f], body, xylim, span[f],
					intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlambert_krnl32");
		break;
	case INHOLAMBERT:
		/* Launch the Inhomogeneous Lambert kernel */
		for (f=1; f<=nframes; f++)
			ap_inholambert_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreadspx[f], body, xylim, span[f], intensity_factor,f);
		checkErrorAfterKernelLaunch("ap_inholambert_krnl32");
		break;
	case LOMMEL:
		/* Launch the Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_lommel_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreadspx[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_lommel_krnl32");
		break;
	case HARMLOMMEL:
		/* Launch the HarmLommel kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlommel_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreadspx[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlommel_krnl32");
		break;
	case INHOLOMMEL:
		/* Launch the Inhomogeneous Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_inholommel_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreadspx[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_inholommel_krnl32");
		break;
	case GEOMETRICAL:
		/* Launch the Geometrical law kernel */
		for (f=1; f<=nframes; f++)
			ap_geometrical_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_geometrical_krnl32");
		break;
	case HAPKE:
		/* Launch the Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_hapke_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreadspx[f], body, xylim, span[f], intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_hapke_krnl32");
		break;
	case HARMHAPKE:
		/* Launch the HarmHapke kernel */
		for (f=1; f<=nframes; f++)
			ap_harmhapke_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreadspx[f], body, xylim, span[f],	intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_harmhapke_krnl32");
		break;
	case INHOHAPKE:
		/* Launch the Inhomogeneous Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_inhohapke_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreadspx[f], body, xylim, span[f], intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_inhohapke_krnl32");
		break;
	case KAASALAINEN:
		/* Launch single-thread kernel to init Kaas */
		gpuErrchk(hipMalloc((void**)&phasefunc, sizeof(float)*(nframes+1)));
		ap_kaas_init_krnl32<<<BLK,THD>>>(dmod, phasefunc, phase,
				scale_lommsee, scale_lambert, nframes);
		checkErrorAfterKernelLaunch("ap_kaas_init_krnl32");

		/* Launch the main Kaasalainen kernel */
		for (f=1; f<=nframes; f++){
			ap_kaas_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreadspx[f], body, xylim, span[f], intensity_factor,
					phase, phasefunc, scale_lommsee, scale_lambert, f, nframes);
		}
		checkErrorAfterKernelLaunch("ap_kaas_krnl32");
		hipFree(phasefunc);
		break;
	case HARMKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_harmkaas_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreadspx[f], body, xylim, span[f], intensity_factor, phase, f);
		checkErrorAfterKernelLaunch("ap_harmkaas_krnl32");
		break;
	case INHOKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_inhokaas_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,	pos,
                    nThreadspx[f], body, xylim, span[f], intensity_factor, phase, f);
		checkErrorAfterKernelLaunch("ap_inhokaas_krnl32");
		break;
	case NOLAW:
		bailout("apply_photo_gpu32.c: can't set optical scattering law = \"none\" when optical data are used\n");
		break;
	default:
		bailout("apply_photo_gpu32.c: can't handle that optical scattering law yet\n");
	}

	/* Call a streamed parallel reduction which calculates the sums of pos->b
	 * for all frames in a dataset (up to 4 simultaneously)	 */
	sum_brightness_gpu32(ddat, pos, nframes, maxthds, set, maxxylim,ap_stream);

	hipFree(dsum);
	hipFree(sum);
	hipFree(type);
	hipFree(intensity_factor);
	hipFree(phase);
	hipFree(scale_lommsee);
	hipFree(scale_lambert);
	free(htype);
	free(hsum);
}

__host__ void apply_photo_gpu48(struct mod_t *dmod,	struct dat_t *ddat,
		struct pos_t **pos, int4 *xylim, int2 *span, dim3 *BLKpx,
		int body, int set, int nframes, int maxthds,
		int4 maxxylim, hipStream_t *ap_stream)
{
	/* The following function launches kernels to calculate the per-pixel brightness
	 * of the model asteroid.  None of kernels examine ALL of the pixels in any
	 * one pos; rather only the area defined by pos->xlim and pos->ylim.  In this
	 * context, the input argument span contains the x and y direction spans for
	 * each frame's pos.  BLKpx defines the number of blocks needed for those
	 * kernels.  nThreadspx defines the number of pixels calculated for each
	 * frame's pos.  nThreads defines the full number of pixels per pos.	 */

	unsigned char *type, *htype;
	int f;
	double *hsum, *dsum, *sum, *intensity_factor, *phase, *phasefunc,
		*scale_lommsee, *scale_lambert;
	dim3 BLK, THD;

	gpuErrchk(hipMalloc((void**)&type, sizeof(unsigned char) * 2));
	gpuErrchk(hipMalloc((void**)&sum, sizeof(double) * (nframes+1)));
	gpuErrchk(hipMalloc((void**)&dsum, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&intensity_factor, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&phase, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&scale_lommsee, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&scale_lambert, sizeof(double)*(nframes+1)));
	htype = (unsigned char *) malloc(2*sizeof(unsigned char));
	hsum = (double *) malloc((nframes+1)*sizeof(double));

	/* Launch single-thread kernel to assign pos address and get type */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nframes) / THD.x);
	ap_init_krnl64<<<BLK,THD>>>(ddat, dmod, pos, set, nframes, type, dsum,
			intensity_factor, phase);
	checkErrorAfterKernelLaunch("ap_init_krnl32");
	gpuErrchk(hipMemcpy(htype, type, sizeof(unsigned char) *2,
			hipMemcpyDeviceToHost));

	switch (htype[0]) {
//	case LAMBERTLAW:
//		/* Launch Lambert Law kernel */
//		for (f=1; f<=nframes; f++)
//			ap_lambertlaw_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,pos,
//					intensity_factor, xylim, nThreadspx[f], body, span[f], f);
//		checkErrorAfterKernelLaunch("ap_lambertlaw_krnl32");
//		break;
//	case HARMLAMBERT:
//		/* Launch the HarmLambert kernel */
//		for (f=1; f<=nframes; f++)
//			ap_harmlambert_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(
//					dmod, pos, nThreadspx[f], body, xylim, span[f],
//					intensity_factor, f);
//		checkErrorAfterKernelLaunch("ap_harmlambert_krnl32");
//		break;
//	case INHOLAMBERT:
//		/* Launch the Inhomogeneous Lambert kernel */
//		for (f=1; f<=nframes; f++)
//			ap_inholambert_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
//					pos, nThreadspx[f], body, xylim, span[f], intensity_factor,f);
//		checkErrorAfterKernelLaunch("ap_inholambert_krnl32");
//		break;
//	case LOMMEL:
//		/* Launch the Lommel kernel */
//		for (f=1; f<=nframes; f++)
//			ap_lommel_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
//					nThreadspx[f], body, xylim, span[f], intensity_factor, f);
//		checkErrorAfterKernelLaunch("ap_lommel_krnl32");
//		break;
//	case HARMLOMMEL:
//		/* Launch the HarmLommel kernel */
//		for (f=1; f<=nframes; f++)
//			ap_harmlommel_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
//					pos, nThreadspx[f], body, xylim, span[f], intensity_factor, f);
//		checkErrorAfterKernelLaunch("ap_harmlommel_krnl32");
//		break;
//	case INHOLOMMEL:
//		/* Launch the Inhomogeneous Lommel kernel */
//		for (f=1; f<=nframes; f++)
//			ap_inholommel_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
//					pos, nThreadspx[f], body, xylim, span[f], intensity_factor, f);
//		checkErrorAfterKernelLaunch("ap_inholommel_krnl32");
//		break;
//	case GEOMETRICAL:
//		/* Launch the Geometrical law kernel */
//		for (f=1; f<=nframes; f++)
//			ap_geometrical_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
//					pos, nThreads[f], body, xylim, span[f], intensity_factor, f);
//		checkErrorAfterKernelLaunch("ap_geometrical_krnl32");
//		break;
//	case HAPKE:
//		/* Launch the Hapke kernel */
//		for (f=1; f<=nframes; f++)
//			ap_hapke_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
//					nThreadspx[f], body, xylim, span[f], intensity_factor,
//					phase, f);
//		checkErrorAfterKernelLaunch("ap_hapke_krnl32");
//		break;
//	case HARMHAPKE:
//		/* Launch the HarmHapke kernel */
//		for (f=1; f<=nframes; f++)
//			ap_harmhapke_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
//					pos, nThreadspx[f], body, xylim, span[f],	intensity_factor,
//					phase, f);
//		checkErrorAfterKernelLaunch("ap_harmhapke_krnl32");
//		break;
//	case INHOHAPKE:
//		/* Launch the Inhomogeneous Hapke kernel */
//		for (f=1; f<=nframes; f++)
//			ap_inhohapke_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
//					pos, nThreadspx[f], body, xylim, span[f], intensity_factor,
//					phase, f);
//		checkErrorAfterKernelLaunch("ap_inhohapke_krnl32");
//		break;
	case KAASALAINEN:
		/* Launch single-thread kernel to init Kaas */
		gpuErrchk(hipMalloc((void**)&phasefunc, sizeof(double)*(nframes+1)));
		ap_kaas_init_krnl64<<<BLK,THD, 0, ap_stream[0]>>>(dmod, phasefunc, phase,
				scale_lommsee, scale_lambert, nframes);
		checkErrorAfterKernelLaunch("ap_kaas_init_krnl64");

		/* Launch the main Kaasalainen kernel */
		for (f=1; f<=nframes; f++){
			ap_kaas_krnl48<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					maxthds, body, xylim, span[f], intensity_factor,
					phase, phasefunc, scale_lommsee, scale_lambert, f, nframes, set);
		}
		checkErrorAfterKernelLaunch("ap_kaas_krnl48");
		hipFree(phasefunc);
		break;
//	case HARMKAAS:
//		/* Launch the HarmKaas kernel */
//		for (f=1; f<=nframes; f++)
//			ap_harmkaas_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
//					nThreadspx[f], body, xylim, span[f], intensity_factor, phase, f);
//		checkErrorAfterKernelLaunch("ap_harmkaas_krnl32");
//		break;
//	case INHOKAAS:
//		/* Launch the HarmKaas kernel */
//		for (f=1; f<=nframes; f++)
//			ap_inhokaas_krnl32<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,	pos,
//                    nThreadspx[f], body, xylim, span[f], intensity_factor, phase, f);
//		checkErrorAfterKernelLaunch("ap_inhokaas_krnl32");
//		break;
//	case NOLAW:
//		bailout("apply_photo_gpu32.c: can't set optical scattering law = \"none\" when optical data are used\n");
//		break;
//	default:
//		bailout("apply_photo_gpu32.c: can't handle that optical scattering law yet\n");
	}

	/* Call a streamed parallel reduction which calculates the sums of pos->b
	 * for all frames in a dataset (up to 4 simultaneously)	 */
	sum_brightness_gpu32(ddat, pos, nframes, maxthds, set, maxxylim,ap_stream);

	hipFree(dsum);
	hipFree(sum);
	hipFree(type);
	hipFree(intensity_factor);
	hipFree(phase);
	hipFree(scale_lommsee);
	hipFree(scale_lambert);
	free(htype);
	free(hsum);
}
__host__ void apply_photo_gpu64(
		struct mod_t *dmod,
		struct dat_t *ddat,
		struct pos_t **pos,
		int4 *xylim,
		int2 *span,
		dim3 *BLKpx,
		int *nThreads,
		int body,
		int set,
		int nframes,
		int *nThreadspx,
		int maxthds,
		int4 maxxylim,
		hipStream_t *ap_stream)
{
	unsigned char *type, *htype;
	int f;
	double *dsum;
	double *hsum, *sum;
	double *intensity_factor, *phase, *phasefunc, *scale_lommsee, *scale_lambert;
	dim3 BLK, THD;

	gpuErrchk(hipMalloc((void**)&type, sizeof(unsigned char) * 2));
	gpuErrchk(hipMalloc((void**)&sum, sizeof(double) * (nframes+1)));
	gpuErrchk(hipMalloc((void**)&dsum, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&intensity_factor, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&phase, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&scale_lommsee, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&scale_lambert, sizeof(double)*(nframes+1)));
	htype = (unsigned char *) malloc(2*sizeof(unsigned char));
	hsum = (double *) malloc((nframes+1)*sizeof(double));

	/* Launch single-thread kernel to assign pos address and get type */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nframes) / THD.x);
	ap_init_krnl64<<<BLK,THD>>>(ddat, dmod, pos, set, nframes, type, dsum,
			intensity_factor, phase);
	checkErrorAfterKernelLaunch("ap_init_krnl64");
	gpuErrchk(hipMemcpy(htype, type, sizeof(unsigned char) *2,
			hipMemcpyDeviceToHost));

	switch (htype[0]) {
	case LAMBERTLAW:
		/* Launch Lambert Law kernel */
		for (f=1; f<=nframes; f++)
			ap_lambertlaw_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,pos,
					intensity_factor, xylim, nThreads[f], body, span[f], f);
		checkErrorAfterKernelLaunch("ap_lambertlaw_krnl64");
		break;
	case HARMLAMBERT:
		/* Launch the HarmLambert kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlambert_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(
					dmod, pos, nThreads[f], body, xylim, span[f],
					intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlambert_krnl64");
		break;
	case INHOLAMBERT:
		/* Launch the Inhomogeneous Lambert kernel */
		for (f=1; f<=nframes; f++)
			ap_inholambert_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor,f);
		checkErrorAfterKernelLaunch("ap_inholambert_krnl64");
		break;
	case LOMMEL:
		/* Launch the Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_lommel_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_lommel_krnl64");
		break;
	case HARMLOMMEL:
		/* Launch the HarmLommel kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlommel_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlommel_krnl64");
		break;
	case INHOLOMMEL:
		/* Launch the Inhomogeneous Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_inholommel_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_inholommel_krnl64");
		break;
	case GEOMETRICAL:
		/* Launch the Geometrical law kernel */
		for (f=1; f<=nframes; f++)
			ap_geometrical_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_geometrical_krnl64");
		break;
	case HAPKE:
		/* Launch the Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_hapke_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_hapke_krnl64");
		break;
	case HARMHAPKE:
		/* Launch the HarmHapke kernel */
		for (f=1; f<=nframes; f++)
			ap_harmhapke_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f],	intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_harmhapke_krnl64");
		break;
	case INHOHAPKE:
		/* Launch the Inhomogeneous Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_inhohapke_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_inhohapke_krnl64");
		break;
	case KAASALAINEN:
		/* Launch single-thread kernel to init Kaas */
		gpuErrchk(hipMalloc((void**)&phasefunc, sizeof(double)*(nframes+1)));
		ap_kaas_init_krnl64<<<BLK,THD>>>(dmod, phasefunc, phase,
				scale_lommsee, scale_lambert, nframes);
		checkErrorAfterKernelLaunch("ap_kaas_init_krnl64");

		/* Launch the main Kaasalainen kernel */
		for (f=1; f<=nframes; f++){
			ap_kaas_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreadspx[f], body, xylim, span[f], intensity_factor,
					phase, phasefunc, scale_lommsee, scale_lambert, f, nframes);
		}
		checkErrorAfterKernelLaunch("ap_kaas_krnl64");
		hipFree(phasefunc);
		break;
	case HARMKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_harmkaas_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], intensity_factor, phase, f);
		checkErrorAfterKernelLaunch("ap_harmkaas_krnl64");
		break;
	case INHOKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_inhokaas_krnl64<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], intensity_factor,
					phase, f);
		checkErrorAfterKernelLaunch("ap_inhokaas_krnl64");
		break;
	case NOLAW:
		bailout("apply_photo_gpu64.c: can't set optical scattering law = \"none\" when optical data are used\n");
		break;
	default:
		bailout("apply_photo_gpu64.c: can't handle that optical scattering law yet\n");
	}

	/* Call a streamed parallel reduction which calculates the sums of pos->b
	 * for all frames in a dataset (up to 4 simultaneously)	 */
	sum_brightness_gpu64(ddat, pos, nframes, maxthds, 1, set, maxthds,
			maxxylim, ap_stream);

	hipFree(dsum);
	hipFree(sum);
	hipFree(type);
	hipFree(intensity_factor);
	hipFree(phase);
	hipFree(scale_lommsee);
	hipFree(scale_lambert);
	free(htype);
	free(hsum);
}
#undef TINY
