#include "hip/hip_runtime.h"

extern "C" {
#include "../shape/head.h"
#include <limits.h>
}

__device__ static float atomicMinf(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}
__device__ static float atomicMin64(double* address, double val)
{
	unsigned long long* address_as_i = (unsigned long long*) address;
	unsigned long long old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__double_as_longlong(::fminf(val, __longlong_as_double(assumed))));
	} while (assumed != old);
	return __longlong_as_double(old);
}
__device__ static float atomicMaxf(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}
__device__ static float atomicMax64(double* address, double val)
{
	unsigned long long* address_as_i = (unsigned long long*) address;
	unsigned long long old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__double_as_longlong(::fmaxf(val, __longlong_as_double(assumed))));
	} while (assumed != old);
	return __longlong_as_double(old);
}
__device__ void dev_POSrect_gpu32(
		struct pos_t **pos,
		int src,
		float imin_dbl,
		float imax_dbl,
		float jmin_dbl,
		float jmax_dbl,
		float4 *ijminmax_overall,
		int frm)	{
	int n, imin, imax, jmin, jmax;
	n = pos[frm]->n;

	/* Update the POS region that contains the target without
	 * regard to whether or not it extends beyond the POS frame */
	atomicMinf(&ijminmax_overall[frm].w, imin_dbl);
	atomicMaxf(&ijminmax_overall[frm].x, imax_dbl);
	atomicMinf(&ijminmax_overall[frm].y, jmin_dbl);
	atomicMaxf(&ijminmax_overall[frm].z, jmax_dbl);

	/*  Update the subset of the POS frame that contains the target  */
	imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
	imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
	jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
	jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

	/* Make sure it's smaller than n */
	imin = MAX(imin,-n);
	imax = MIN(imax, n);
	jmin = MAX(jmin,-n);
	jmax = MIN(jmax, n);

	if (src) {
		atomicMin(&pos[frm]->xlim2[0], imin);
		atomicMax(&pos[frm]->xlim2[1], imax);
		atomicMin(&pos[frm]->ylim2[0], jmin);
		atomicMax(&pos[frm]->ylim2[1], jmax);
	} else {
		atomicMin(&pos[frm]->xlim[0], imin);
		atomicMax(&pos[frm]->xlim[1], imax);
		atomicMin(&pos[frm]->ylim[0], jmin);
		atomicMax(&pos[frm]->ylim[1], jmax);
	}
}
__device__ void dev_POSrect_gpu64(
		struct pos_t **pos,
		int src,
		double imin_dbl,
		double imax_dbl,
		double jmin_dbl,
		double jmax_dbl,
		double4 *ijminmax_overall,
		int frm)	{
	int n, imin, imax, jmin, jmax;
	n = pos[frm]->n;

	/* Update the POS region that contains the target without
	 * regard to whether or not it extends beyond the POS frame */
	atomicMin64(&ijminmax_overall[frm].w, imin_dbl);
	atomicMax64(&ijminmax_overall[frm].x, imax_dbl);
	atomicMin64(&ijminmax_overall[frm].y, jmin_dbl);
	atomicMax64(&ijminmax_overall[frm].z, jmax_dbl);

	/*  Update the subset of the POS frame that contains the target  */
	imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
	imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
	jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
	jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

	/* Make sure it's smaller than n */
	imin = MAX(imin,-n);
	imax = MIN(imax, n);
	jmin = MAX(jmin,-n);
	jmax = MIN(jmax, n);

	if (src) {
		atomicMin(&pos[frm]->xlim2[0], imin);
		atomicMax(&pos[frm]->xlim2[1], imax);
		atomicMin(&pos[frm]->ylim2[0], jmin);
		atomicMax(&pos[frm]->ylim2[1], jmax);
	} else {
		atomicMin(&pos[frm]->xlim[0], imin);
		atomicMax(&pos[frm]->xlim[1], imax);
		atomicMin(&pos[frm]->ylim[0], jmin);
		atomicMax(&pos[frm]->ylim[1], jmax);
	}
}
