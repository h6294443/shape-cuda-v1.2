#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                                bestfit.c

Iterate over all floating parameters, at each step adjusting just one parameter x in order
to minimize objective(x), the objective function (reduced chi-square plus penalties).
Continue until the fractional reduction in objective(x) due to a full pass through the
parameter list is less than term_prec.  Return the final value of the objective function.
__________________________________________________________________________________________
Modified 2016 July 7 by Matt Engels:
	Adapted for use in shape-cuda.
------------------------------------------------------------------------------------------
Modified 2014 February 19 by CM:
    Allow for multiple optical scattering laws when setting the "vary_hapke" flag

Modified 2013 July 14 by CM:
    Implement the "term_maxiter" parameter

Modified 2012 July 5 by MCN and CM:
    Use the gethostname function rather than the HOST environment variable to get root's
        hostname
    List root's PID in addition to the hostname
    List the PID for each branch node, not just the hostname

Modified 2012 June 13 by CM:
    Implement "objfunc_start" parameter

Modified 2012 March 23 by CM:
    Implement Doppler scaling -- more particularly, simultaneous adjustment of shape/spin
        parameters and Doppler scale factors via the "vary_dopscale" parameter

Modified 2010 April 12 by CM:
    Bug fix: When fitting a size, shape, or spin parameter with the
         "vary_delcor0" parameter being used, call realize_delcor to reset
         the 0th-order delay correction polynomial coefficients to their
         saved values before calling vary_params.  (For infinitely fine
         model resolution and delay-Doppler resolution this wouldn't
         matter but in practice it does.)

Modified 2009 November 15 by CM:
    Fix printf statement with too many arguments

Modified 2009 July 5 by CM:
    Add "npar_update" parameter rather than hard-wiring an update (rewrite
        mod and obs files and display reduced chi2 and penalty functions)
        every 20th parameter adjustment

Modified 2009 April 3 by CM:
    If the model has illegal properties (e.g., negative ellipsoid diameters)
        then, for each type of problem, multiply the objective function not
        only by the "bad_objfactor" parameter but also by an additional
        factor that increases as the problem gets worse.  The
        "baddiam_logfactor" "badphoto_logfactor" "posbnd_logfactor"
        "badposet_logfactor" and "badradar_logfactor" parameters are the
        logarithms of the additional factors for the five possible problem
        types; the calc_fits routine computes the logarithms rather than the
        factors themselves so as to avoid floating-point overflow.
    Revise MPI_CALC so that root receives the "posbnd_logfactor" parameter
        from each branch node rather than the "posbnd" parameter:
        posbnd_logfactor > 0.0 if the model extends beyond the POS frame
        for any of the branch node's datasets.  If root sees that this
        value is > 0.0, it will set its "posbnd" flag and will increase the
        objective function accordingly.
    Revise MPI_CALC so that root receives the "badposet_logfactor"
        parameter from each branch node: badposet_logfactor > 0.0 if the
        model extends beyond the fit frame for any of the branch node's
        plane-of-sky datasets.  If root sees that this value is > 0.0, it
        will set its "badposet" flag and will increase the objective
        function accordingly.
    Revise MPI_CALC so that root receives the "badradar_logfactor"
        parameter from each branch node: badradar_logfactor > 0.0 if the
        model is too wide in delay-Doppler space for the program to
        construct some or all (delay-)Doppler fit frames.  If root sees
        that this value is > 0.0, it will set its "badradar" flag and will
        increase the objective function accordingly.
    For MPI_Recv calls, mpi_par[0] is no longer equal to the MPI action,
        since the message tag argument already serves that purpose (as of
        2008 April 10) -- so the other mpi_par elements are renumbered

Modified 2008 August 10 by CM:
    Never terminate the fit at the end of a partial iteration -- that is,
        after the first iteration of a fit where first_fitpar > 0

Modified 2008 July 11 by CM:
    Display the hostname even for single-processor fits

Modified 2008 April 10 by CM:
    For parallel-processing fits, display the hostname for each node
    Use message tag argument to MPI_Recv to identify the MPI action

Modified 2007 August 29 by CM:
    Implement the "avoid_badpos" parameter: if this parameter is turned on
        and the model extends beyond the POS frame and it is time to fit a
        size parameter, start by shrinking that size parameter until the
        model fits within the POS frame
    Implement the "bad_objfactor" parameter in routine objective: multiply
        the objective function by this factor for illegal photometric
        parameters, for tiny or negative ellipsoid diameters, and for
        models that extend beyond the plane-of-sky frame.  (Previously
        this factor was fixed at 2.0.)
    Rename MPI_TAG to MPI_TAG_1 to avoid name conflict with mpich headers

Modified 2007 August 16 by CM:
    Implement the "term_badmodel" parameter: If this parameter is turned on
        and, at the end of any fit iteration, the model ever extends beyond
        the POS frame OR has any illegal photometric parameters OR has any
        tiny or negative ellipsoid diameters, the fit is terminated.

Modified 2007 August 10 by CM:
    Eliminate unused variables

Modified 2006 December 20 by CM:
    Revise MPI_CALC so that root receives the "posbnd" parameter from each
        branch node, so that the objective function can be doubled if the
        model extends beyond the plane-of-sky frame for any datasets
    If the model extends beyond the plane-of-sky frame for any trial value
        of a parameter, evaluate the model for the best-fit parameter value
        to check whether or not it extends beyond the POS frame

Modified 2006 October 1 by CM:
    Add two new arguments to realize_delcor
    Add three new arguments to realize_photo
    Implement "vary_delcor0" "vary_radalb" and "vary_optalb" parameters
    Implement SIZEPAR parameters via the "newsize" variable

Modified 2005 June 27 by CM:
    Renamed "round" function to "iround" to avoid conflict

Modified 2005 March 17 by CM:
    For parallel processing, check that root is receiving the responses
        to the correct broadcast
    Root no longer needs to compute degrees of freedom or to receive
        dof values from branch nodes: Now they are computed in read_dat
    Degrees of freedom can now be floating-point rather than integer

Modified 2005 February 28 by CM:
    Add screen warnings if objective function has been doubled due to
        (a) tiny or negative ellipsoid diameters
        (b) illegal photometric parameters
        (c) model extending beyond the model POS frame
    Initialize the three parameters (baddiam, badphoto, posbnd) that
        flag these three problems in other routines (realize_mod,
        realize_photo, calc_fits) rather than in objective(x), so that
        these three parameters can be used for actions other than "fit"
    Rename DATAPAR to be DELCORPAR
    Add XYOFFPAR and implement the new realize_xyoff routine

Modified 2005 February 22 by CM:
    Move branch nodes' signoff statements from shape.c to here, so that
        they can appear in order

Modified 2005 February 13 by CM:
    Rename objective function "f(x)" to be "objective(x)"
    Only broadcast to branch nodes if there are any branch nodes
        (i.e., if mpi_nproc > 1)
    Broadcast the new MPI_DUMMYPAR signal to branch nodes before evaluating
        objective(0.0), the objective function for the existing model;
        this tells each branch node to point hotparam to a dummy variable
        rather than to a model parameter, so that the dummy variable will
        be set to 0.0 and the model will be unchanged.
    Broadcast the new MPI_CALFACT signal to branch nodes to get updated
        calibration factors before rewriting the obs file
    Root now realizes the model after setting a parameter to its best value
    Make sure that root and branch nodes update the model (i.e., that they
        call the calc_fits and chi2 routines) before rewriting the mod and
        obs files and before calling routine show_deldoplim
    Avoid unnecessary model realizations for root by allowing newshape,
        newspin, newphoto, and newdelcor to be 0, not always 1 as before
    Move MPI_DONE broadcast to here from shape.c

Modified 2005 January 25 by CM:
    Eliminated unused variable

Modified 2005 January 10 by CM:
    When fitting using parallel processing, ping all of the branch nodes
        and inform the user that they're active

Modified 2004 October 29 by CM:
    Add "first_fitpar" parameter so that a fit can be started (or resumed)
        at some parameter (counting from 0) other than the first parameter

Modified 2004 October 10 by CM:
    Fix chi-square display at start of each iteration and at the
        end of the fit by calling realize_mod, realize_spin, realize_photo,
        realize_delcor, and calc_fits before calling chi2

Modified 2004 August 13 by CM:
    Call modified minimum search routine brent_abs rather than brent
        so that absolute fitting tolerances can be specified

Modified 2004 May 21 by CM:
    Display the final values of the individual penalty functions

Modified 2004 April 3 by CM:
    Add the "list_breakdown" argument to routine chi2 so that we can
        display the chi2 breakdown by data type (Doppler, delay-Doppler,
        POS, lightcurves) at the start of each fit iteration and at
        the end of the fit

Modified 2004 February 26 by CM:
    realize_photo now takes two arguments rather than one

Modified 2003 April 26 by CM:
    Added "show_deldoplim" call at the end of each fit iteration,
        to check for overly tight data vignetting

Modified 2003 April 23 by CM:
    Implemented '=' state for delay correction polynomial coefficients
        via the "realize_delcor" routine

Modified 2003 April 17 by CM:
    Added "baddiam" parameter to function f so that the objective
        function is doubled if an ellipsoid component has a tiny or
        negative diameter

Modified 2003 April 2 by CM:
    In function f (which computes reduced-chi-squared-plus-penalties),
        moved call to "penalties" from before spar->showstate is set
        to after.
    Values of reduced chi-squared and of the various penalties are
        printed to the screen after every 20th parameter adjustment.
        To be precise, they're printed at the very first call to f when
        adjusting parameter 21, 41, 61, etc.  This call is made within
        function bestfit by minimum-bracketing function mnbrak;
        it corresponds to the *unadjusted* value of parameter 21 (or 41
        or ...), which is what we want.
    Until now, the individual penalty values were being printed on the
 *second* call to f, also made by mnbrak but with parameter 21
        incremented by the relevant initial step size (e.g., length_step).
        Hence these printed values were irrelevant and misleadingly large.
        Moving the call to "penalties" later in the code fixes the problem.
 *****************************************************************************************/
extern "C" {
#include "../shape/head.h"
}
//static __device__ double *hotparam;
static __device__ int dmax_frames;
static struct par_t *spar, *sdev_par, *sdev_par1;
static struct mod_t *smod, *smod1, *sdev_mod, *sdev_mod1;
static struct dat_t *sdat, *sdev_dat, *sdev_dat1;

static int newsize, newshape, newspin, newphoto, newdelcor, newdopscale, newxyoff,
showvals=0, vary_delcor0_size, vary_delcor0_shapespin, vary_dopscale_spin,
vary_dopscale_sizeshape, vary_alb_size, vary_alb_shapespin, vary_hapke,
call_vary_params, check_posbnd, check_badposet, check_badradar;
static double deldop_zmax, deldop_zmax_save, cos_subradarlat, cos_subradarlat_save,
rad_xsec, rad_xsec_save, opt_brightness, opt_brightness_save, baddiam_factor,
badphoto_factor, posbnd_factor, badposet_factor, badradar_factor,
baddopscale_factor;
static unsigned char type;
static double hotparamval;

__host__ double objective_gpu(double x, struct vertices_t **verts,
		unsigned char *htype, unsigned char *dtype, int *nframes, int *nviews,
		int *lc_n, int nsets, int nf, hipStream_t *bf_stream);
__host__ double objective_pthreads(double x, struct vertices_t **verts0, struct
		vertices_t **verts1, unsigned char *htype, unsigned char *dtype0, unsigned char *dtype1, int
		*nframes, int *nviews, int *lc_n, int *GPUID, int nsets, int nf, int
		max_frames, pthread_t thread1,  pthread_t thread2, hipStream_t
		*gpu0_stream, hipStream_t *gpu1_stream);

__device__ double bf_hotparamval, bf_dummyval=0.0, *hotparam;
__device__ int bf_partype;

__global__ void bf_get_flags_krnl(struct par_t *dpar, unsigned char *flags) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		flags[0] = dpar->baddiam;
		flags[1] = dpar->badphoto;
		flags[2] = dpar->posbnd;
		flags[3] = dpar->badposet;
		flags[4] = dpar->badradar;
		flags[5] = dpar->baddopscale;
	}
}
__global__ void ocs_get_flags_krnl(struct par_t *dpar, unsigned char *flags,
		double *dlogfactors) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		flags[0] = dpar->baddiam;
		flags[1] = dpar->badphoto;
		flags[2] = dpar->posbnd;
		flags[3] = dpar->badposet;
		flags[4] = dpar->badradar;
		flags[5] = dpar->baddopscale;

		dlogfactors[0] = dpar->bad_objfactor;
		dlogfactors[1] = dpar->baddiam_logfactor;
		dlogfactors[2] = dpar->badphoto_logfactor;
		dlogfactors[3] = dpar->posbnd_logfactor;
		dlogfactors[4] = dpar->badposet_logfactor;
		dlogfactors[5] = dpar->badradar_logfactor;
		dlogfactors[6] = dpar->baddopscale_logfactor;
	}
}
__global__ void bf_set_hotparam_initial_krnl() {
	/* Single-threaded kernel */
	if (threadIdx.x == 0)
		hotparam = &bf_dummyval;
}
__global__ void bf_set_hotparam_pntr_krnl(double **fpntr,
		int *fpartype, int p) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		hotparam = fpntr[p];	/* This is pointing at a device variable */
		bf_partype = fpartype[p];  /* parameter type */
	}
}
__global__ void bf_get_hotparam_val_krnl() {
	/* Single threaded kernel */
	if (threadIdx.x == 0)
		bf_hotparamval = *hotparam;
}
__global__ void bf_mult_hotparam_val_krnl(double factor) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0)
		*hotparam *= factor;
}
__global__ void bf_set_hotparam_val_krnl(double newvalue) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		*hotparam = newvalue;
		bf_hotparamval = newvalue;
	}
}
__global__ void set_verts_shortcut_krnl(struct mod_t *dmod,
		struct vertices_t **verts, int max_frames) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		verts[0] = &dmod->shape.comp[0].real;
		dmax_frames = max_frames;
	}
}

__host__ double bestfit_gpu(struct par_t *dpar, struct mod_t *dmod,
		struct dat_t *ddat, struct par_t *par, struct mod_t *mod,
		struct dat_t *dat)
{
	char hostname[MAXLEN], dofstring[MAXLEN];
	int i, iter=0, p, cntr, first_fitpar, partype, keep_iterating=1, ilaw, nf, term_maxiter;
	long pid_long;
	pid_t pid;
	double beginerr, enderr, ax, bx, cx, obja, objb, objc, xmin, final_chi2,
		final_redchi2, dummyval2, dummyval3, dummyval4, delta_delcor0,
		dopscale_factor, radalb_factor, optalb_factor, *hfparstep, *hfpartol,
		*hfparabstol, objfunc_start, term_prec;
	unsigned char *flags, *hflags, *htype, *dtype, action, avoid_badpos, term_badmodel;
	int nsets, *nframes, *lc_n, *nviews, nfpar, *hfpartype, npar_update, max_frames=0,
			max_streams=0;
	struct vertices_t **verts;
	dim3 THD, BLK;

	gpuErrchk(hipSetDevice(GPU0));
	/* This section collects parameters used for CUDA kernel launches throughout
	 * the program.  The cudaStreams created here are used/re-used for the
	 * lifetime of one program run */
	nsets = dat->nsets;
	nfpar = par->nfpar;
	nf = mod->shape.comp[0].real.nf;
	action = par->action;
	npar_update = par->npar_update;
	avoid_badpos = par->avoid_badpos;
	objfunc_start = par->objfunc_start;
	term_prec = par->term_prec;
	term_badmodel = par->term_badmodel;
	type = mod->shape.comp[0].type;
	htype 	= (unsigned char *) malloc(nsets*sizeof(unsigned char));
	nframes = (int *) malloc(nsets*sizeof(int));
	lc_n	= (int *) malloc(nsets*sizeof(int));
	nviews 	= (int *) malloc(nsets*sizeof(int));
	gpuErrchk(hipMalloc((void**)&dtype, sizeof(unsigned char)*nsets));
	gpuErrchk(hipMalloc((void**)&verts, sizeof(struct vertices_t*)*2));

	for (int s=0; s<nsets; s++) {
		htype[s] = dat->set[s].type;
		switch (htype[s]) {
		case DELAY:
			nframes[s] = dat->set[s].desc.deldop.nframes;
			nviews[s]  = dat->set[s].desc.deldop.nviews;
			lc_n[s]    = 0;
			break;
		case DOPPLER:
			nframes[s] = dat->set[s].desc.doppler.nframes;
			nviews[s]  = dat->set[s].desc.doppler.nviews;
			lc_n[s]    = 0;
			break;
		case POS:
			nframes[s] = dat->set[s].desc.poset.nframes;
			nviews[s]  = dat->set[s].desc.poset.nviews;
			lc_n[s]    = 0;
			break;
		case LGHTCRV:
			nframes[s] = dat->set[s].desc.lghtcrv.ncalc;
			nviews[s]  = dat->set[s].desc.lghtcrv.nviews;
			lc_n[s]    = dat->set[s].desc.lghtcrv.n;
			break;
		}
		if (nframes[s]>max_frames)	max_frames = nframes[s];
	}
	gpuErrchk(hipMemcpy(dtype, htype, sizeof(unsigned char)*nsets,
			hipMemcpyHostToDevice));

	/* The following check is necessary to ensure the dVdIdCOM reduction has
	 * enough streams to operate properly	 */
	if (max_frames < 13) max_streams = 13;
	else	max_streams = max_frames;

	/* Create streams for gpu0 (the only gpu in single-GPU mode) */
	hipStream_t bf_stream[max_streams];
	for (int f=0; f<max_streams; f++)
		gpuErrchk(hipStreamCreate(&bf_stream[f]));
	//hipStream_t bf1_stream[max_streams];

	/*..........................End section..................................*/

	/* Get the hostname of host machine and the PID */
	(void) gethostname(hostname, MAXLEN-1);
	pid = getpid();
	pid_long = (long) pid;  /* Assumes pid_t fits in a long */
	printf("#\n# CUDA fit (pid %ld on %s)\n", pid_long, hostname);
	fflush(stdout);

	/* Allocate memory for pointers, steps, and tolerances on bothhost and
	 * device. fpntr remains a hipMallocManaged allocation because it is a
	 * double pointer.  */
	gpuErrchk(hipMalloc((void**)&sdev_par, sizeof(struct par_t)));
	gpuErrchk(hipMemcpy(sdev_par, &par, sizeof(struct par_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&sdev_mod, sizeof(struct mod_t)));
	gpuErrchk(hipMemcpy(sdev_mod, &mod, sizeof(struct mod_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&sdev_dat, sizeof(struct dat_t)));
	gpuErrchk(hipMemcpy(sdev_dat, &dat, sizeof(struct dat_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&flags, sizeof(unsigned char) * 7));
	gpuErrchk(hipMalloc((void**)&fparstep,   sizeof(double)  * nfpar));
	gpuErrchk(hipMalloc((void**)&fpartol,    sizeof(double)  * nfpar));
	gpuErrchk(hipMalloc((void**)&fparabstol, sizeof(double)  * nfpar));
	gpuErrchk(hipMalloc((void**)&fpartype,   sizeof(int) 	  * nfpar));
	cudaCalloc1((void**)&fpntr,	  sizeof(double*), nfpar);
	hfparstep 	 = (double *) malloc(nfpar*sizeof(double));
	hfpartol	 = (double *) malloc(nfpar*sizeof(double));
	hfparabstol  = (double *) malloc(nfpar*sizeof(double));
	hfpartype 	 = (int *) 	  malloc(nfpar*sizeof(int));
	hflags 		 = (unsigned char *) malloc(7*sizeof(unsigned char));

	for (i=0; i<nfpar; i++)
		gpuErrchk(hipMalloc((void**)&fpntr[i], sizeof(double) * 1));

	/* Set vertices shortcut and also set max_frames (the maximum number of
	 * frames for any one set) to device so that objective_gpu
	 * can retrieve it later */
	//gpuErrchk(hipDeviceSynchronize());

	set_verts_shortcut_krnl<<<1,1>>>(dmod, verts, max_frames);
	checkErrorAfterKernelLaunch("set_verts_shortcut_krnl");

	/* Initialize static global pointers used by objective(x) below
      to be compatible with "Numerical Recipes in C" routines       */
	spar = par;			smod = mod;			sdat = dat;
	sdev_par = dpar;	sdev_mod = dmod;	sdev_dat = ddat;

	/*  Initialize static global parameters  */
	newsize = newshape = newspin = newphoto = newdelcor = newdopscale = newxyoff = 1;
	deldop_zmax = deldop_zmax_save = 0.0;
	cos_subradarlat = cos_subradarlat_save = 0.0;
	rad_xsec = rad_xsec_save = 0.0;
	opt_brightness = opt_brightness_save = 0.0;
	vary_delcor0_size = (par->vary_delcor0 != VARY_NONE);
	vary_delcor0_shapespin = (par->vary_delcor0 == VARY_ALL);
	vary_dopscale_spin = (par->vary_dopscale != VARY_NONE);
	vary_dopscale_sizeshape = (par->vary_dopscale == VARY_ALL);
	vary_alb_size = (par->vary_radalb != VARY_NONE || par->vary_optalb != VARY_NONE);
	vary_alb_shapespin = (par->vary_radalb == VARY_ALL || par->vary_optalb == VARY_ALL);
	vary_hapke = 0;
	if (par->vary_optalb != VARY_NONE)
		for (ilaw=0; ilaw<mod->photo.noptlaws; ilaw++)
			if (mod->photo.opttype[ilaw] == HAPKE || mod->photo.opttype[ilaw] == HARMHAPKE
					|| mod->photo.opttype[ilaw] == INHOHAPKE)
				vary_hapke = 1;
	call_vary_params = (par->vary_delcor0 != VARY_NONE || par->vary_dopscale != VARY_NONE
			|| par->vary_radalb != VARY_NONE
			|| par->vary_optalb != VARY_NONE);

	/*  Initialize local parameters  */
	delta_delcor0 = 0.0;
	dopscale_factor = radalb_factor = optalb_factor = 1.0;

	/* The following call sets up the parameter lists allocated above and copy
	 * the device contents to host copies */
	mkparlist_gpu(dpar, dmod,	ddat, fparstep, fpartol, fparabstol, fpartype,
			fpntr, nfpar, nsets);
	gpuErrchk(hipMemcpy(hfparstep, fparstep, sizeof(double)*nfpar, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hfpartol, fpartol, sizeof(double)*nfpar, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hfparabstol, fparabstol, sizeof(double)*nfpar, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hfpartype,	fpartype, sizeof(int)*nfpar, hipMemcpyDeviceToHost));

	/* Compute deldop_zmax_save, cos_subradarlat_save, rad_xsec_save, and
	 * opt_brightness_save for the initial model  */
	//call_vary_params=1;
	if (call_vary_params)
	{
		realize_mod_gpu(dpar, dmod, type, nf, bf_stream);

		realize_spin_gpu(dpar, dmod, ddat, htype, nframes, nviews,
				nsets, bf_stream);

		realize_photo_gpu(dpar, dmod, 1.0, 1.0, 0, nf);  /* set R_save to R */

		vary_params_gpu(dpar, dmod, ddat, action, &deldop_zmax_save,
				&rad_xsec_save, &opt_brightness_save, &cos_subradarlat_save,
				nframes, lc_n, nviews, verts, htype, dtype, nf, nsets,
				bf_stream, max_frames);
	}
	printf("rad_xsec: %f\n", rad_xsec_save);
	printf("deldop_zmax: %f\n", (float)deldop_zmax_save);

	/* Point hotparam to a dummy variable (dummyval) rather than to a model pa-
	 * rameter; then call objective(0.0) to set dummy variable = 0.0, realize
	 * the initial model, calculate the fits, return initial model's objective
	 * function as enderr.                          */
	bf_set_hotparam_initial_krnl<<<1,1>>>();
	checkErrorAfterKernelLaunch("bf_set_hotparam_initial_krnl");

	enderr = objective_gpu(0.0, verts, htype, dtype, nframes,
				nviews, lc_n, nsets, nf, bf_stream);

	printf("#\n# searching for best fit ...\n");
	printf("%4d %8.6f to begin", 0, enderr);

	/* Launch single-thread kernel to retrieve flags in dev_par */
	/*		flags[0] = dpar->baddiam;
			flags[1] = dpar->badphoto;
			flags[2] = dpar->posbnd;
			flags[3] = dpar->badposet;
			flags[4] = dpar->badradar;
			flags[5] = dpar->baddopscale;*/

	bf_get_flags_krnl<<<1,1>>>(dpar, flags);
	checkErrorAfterKernelLaunch("bf_get_flags_krnl");
	gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*7,
			hipMemcpyDeviceToHost));

	/* Now act on the flags just retrieved from dev_par */
	if (hflags[0])		printf("  (BAD DIAMS)");
	if (hflags[1])		printf("  (BAD PHOTO)");
	if (hflags[2])		printf("  (BAD POS)");
	if (hflags[3])		printf("  (BAD POSET)");
	if (hflags[4])		printf("  (BAD RADAR)");
	if (hflags[5])		printf("  (BAD DOPSCALE)");		printf("\n");
	fflush(stdout);

	/* Display the region within each delay-Doppler or Doppler frame that, ac-
	 * cording to initial model, has nonzero power. A warning is displayed if
	 * any region extends beyond the data limits: the vignetting is too tight,
	 * or else some model parameter (such as a delay correction polynomial co-
	 * efficient) is seriously in error.   */
	show_deldoplim_gpu(ddat, htype, nsets, nframes, max_frames);

	/* Set the starting fit parameter for the first iteration only  */
	first_fitpar = par->first_fitpar;
	term_maxiter = par->term_maxiter;
	if (first_fitpar < 0 || first_fitpar >= nfpar) {
		printf("ERROR: need 0 <= first_fitpar < nparams (%d)\n", nfpar);
		bailout("bestfit.c\n");
	}

	/* Iteratively adjust model; for each iteration, step through all free pa-
	 * rameters, adjusting one parameter at a time so as to minimize the objec-
	 * tive function at each step. Stop when fractional decrease in the objec-
	 * tive function from one iteration to the next is less than term_prec.   */

	do {
		showvals = 1;        /* show reduced chi-square and penalties at beginning */
		beginerr = enderr;
		printf("# iteration %d %f", ++iter, beginerr);

		/* Launch single-thread kernel to retrieve flags in dev_par */
		bf_get_flags_krnl<<<1,1>>>(dpar, flags);
		checkErrorAfterKernelLaunch("bf_get_flags_krnl");
		gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*7,
				hipMemcpyDeviceToHost));

		/* Now act on the flags just retrieved from dev_par */
		if (hflags[0])		printf("  (BAD DIAMS)");
		if (hflags[1])		printf("  (BAD PHOTO)");
		if (hflags[2])		printf("  (BAD POS)");
		if (hflags[3])		printf("  (BAD POSET)");
		if (hflags[4])		printf("  (BAD RADAR)");
		if (hflags[5])		printf("  (BAD DOPSCALE)");		printf("\n");
		fflush(stdout);

		/* Show breakdown of chi-square by data type    */
		chi2_gpu(dpar, ddat, htype, dtype, nframes, lc_n, 1,
				nsets, bf_stream, max_frames);

		/*  Loop through the free parameters  */
		cntr = first_fitpar % npar_update;
		//p = first_fitpar = 1;
		for (p=first_fitpar; p<nfpar; p++) {

//		p = first_fitpar;
			/*  Adjust only parameter p on this try  */
			bf_set_hotparam_pntr_krnl<<<1,1>>>(fpntr, fpartype, p);
			checkErrorAfterKernelLaunch("bf_set_hotparam_pntr_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&partype, HIP_SYMBOL(bf_partype), sizeof(int),
					0, hipMemcpyDeviceToHost));

			newsize = newshape = newspin = newphoto = newdelcor = newdopscale
					= newxyoff = 0;
			if 		(partype == SIZEPAR)		newsize	 	= 1;
			else if (partype == SHAPEPAR)		newshape 	= 1;
			else if (partype == SPINPAR)		newspin 	= 1;
			else if (partype == PHOTOPAR)		newphoto 	= 1;
			else if (partype == DELCORPAR)		newdelcor 	= 1;
			else if (partype == DOPSCALEPAR)	newdopscale	= 1;
			else if (partype == XYOFFPAR)		newxyoff 	= 1;

			/* If this is a size parameter AND model extends beyond POS frame
			 * AND the "avoid_badpos" parameter is turned on, shrink model by
			 * 5% at a time until it fits within the POS frame.
			 * We must start with the redundant model evaluation for the un-
			 * changed value of the size parameter, in case the first call to
			 * objective displays reduced chi-square and the penalty functions.  */
			if (avoid_badpos && partype == SIZEPAR) {
				bf_get_flags_krnl<<<1,1>>>(dpar, flags);
				checkErrorAfterKernelLaunch("bf_get_flags_krnl");
				gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*7,
						hipMemcpyDeviceToHost));

				/* Get value of (*hotparam) */
				bf_get_hotparam_val_krnl<<<1,1>>>();
				checkErrorAfterKernelLaunch("bf_get_hotparam_val_krnl");
				gpuErrchk(hipMemcpyFromSymbol(&hotparamval, HIP_SYMBOL(bf_hotparamval),
						sizeof(double),	0, hipMemcpyDeviceToHost));

				while (hflags[2]) {
					objective_gpu(hotparamval, verts, htype, dtype,
							nframes, nviews, lc_n, nsets, nf, bf_stream);

					bf_get_flags_krnl<<<1,1>>>(dpar, flags);
					checkErrorAfterKernelLaunch("bf_get_flags_krnl");
					gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*7,
							hipMemcpyDeviceToHost));

					if (hflags[2]) {
						/* Set the value pointed to by hotparam to 0.95 of its
						 * previous value */
						bf_mult_hotparam_val_krnl<<<1,1>>>(0.95);
						checkErrorAfterKernelLaunch("bf_mult_hotparam_val_krnl");
					}
				}
			}

			/* Get value of (*hotparam) so that mnbrak can use it*/
			bf_get_hotparam_val_krnl<<<1,1>>>();
			checkErrorAfterKernelLaunch("bf_get_hotparam_val_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&hotparamval, HIP_SYMBOL(bf_hotparamval),
					sizeof(double),	0, hipMemcpyDeviceToHost));

			/* Use Numerical Recipes routine mnbrak to bracket a minimum in the
			 * objective function (reduced chi-square plus penalties) objec-
			 * tive(x), where x is the value of parameter p.  As initial trial
			 * parameter values, use ax (unadjusted value) and bx, that value
			 * incremented by the appropriate step size (length_step,spin_step,
			 * etc.). mnbrak returns 3 parameter values, with bx between ax
			 * and cx; note that ax and bx are changed from their input values.
			 * It also returns the 3 corresponding objective(x) values, where
			 * objb is less than obja and objc.  Hence there is at least one
			 * local minimum (but not necessarily *any* global minimum)
			 * somewhere between ax and cx.          */
			ax = hotparamval;
			bx = ax + hfparstep[p]; /* par usage us fine here */

			mnbrak_gpu(&ax, &bx, &cx, &obja, &objb, &objc,
					objective_gpu, verts, htype, dtype, nframes,
					nviews, lc_n, nsets, nf, bf_stream);

			/* Before homing in on local minimum, initialize flags that will
			 * tell us if model extended beyond POS frame (sky rendering) for
			 * any trial parameter value(s), if it extended beyond any POS ima-
			 * ges, and if it was too wide in delay-Doppler space         */
			check_posbnd = 0;
			check_badposet = 0;
			check_badradar = 0;

			/* Now use Numerical Recipes function brent to find local minimum -
			 * that is, to find xmin, the best value of x, to within the
			 * *fractional* tolerance specified for parameter p (length_tol,
			 * spin_tol, etc.). brent's return value is the minimized objective
			 * function, objective(xmin). If more than one local minimum bet-
			 * ween ax and cx, brent might not find the best one. brent_abs is
			 * a modified version of brent that has an absolute fitting tole-
			 * rance as one of its arguments, in addition to the existing
			 * fractional tolerance.                                      */
			enderr = brent_abs_gpu(ax, bx, cx, objective_gpu, hfpartol[p],
					hfparabstol[p], &xmin, verts, htype, dtype, nframes, nviews, lc_n,
					nsets, nf, bf_stream);

			/* Realize whichever part(s) of the model has changed.
			 *
			 * The code here is somewhat opaque because more than one part of
			 * the model may have changed - if the "vary_delcor0" "vary_radalb"
			 * and/or "vary_optalb" parameter is being used to permit joint pa-
			 * rameter adjustments. Before calling the vary_params routine, the
			 * size/shape and spin states must be realized (realize_mod and
			 * realize_spin); if albedos are being varied jointly with other
			 * parameters, the photometric state must also be realized
			 * (realize_photo); and in either case the 0th-order delay correc-
			 * tion polynomial coefficients must be reset to their saved
			 * values via the appropriate call to realize_delcor.          */
			/* Set the value pointed to by hotparam to 0.95 of its
			 * previous value (*hotparam) = xmin; */
			bf_set_hotparam_val_krnl<<<1,1>>>(xmin);
			checkErrorAfterKernelLaunch("bf_set_hotparam_val_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&hotparamval, HIP_SYMBOL(bf_hotparamval),
					sizeof(double),	0, hipMemcpyDeviceToHost));

			if (newsize || newshape)
				realize_mod_gpu(dpar, dmod, type, nf, bf_stream);
			if (newspin) {
				realize_spin_gpu(dpar, dmod, ddat, htype, nframes,
						nviews, nsets, bf_stream);
			}
			if ((newsize && vary_alb_size) || ((newshape ||
					newspin) && vary_alb_shapespin))
				realize_photo_gpu(dpar, dmod, 1.0, 1.0, 1, nf);  /* set R to R_save */
			if ((newsize && vary_delcor0_size) || ((newshape || newspin)
					&& vary_delcor0_shapespin)) {
				realize_delcor_gpu(ddat, 0.0, 1, nsets, nframes);  /* set delcor0 to delcor0_save */
			}
			if ((newspin && vary_dopscale_spin) || ((newsize || newshape)
					&& vary_dopscale_sizeshape))
				realize_dopscale_gpu(dpar, ddat, 1.0, 1, nsets, dtype);  /* set dopscale to dopscale_save */
			if (call_vary_params) {
				/* Call vary_params to get the adjustments to 0th-order delay
				 * correction polynomial coefficients, to Doppler scaling fac-
				 * tors, and to radar and optical albedos                  */

				vary_params_gpu(dpar,dmod,ddat,11,&deldop_zmax,
						&rad_xsec, &opt_brightness, &cos_subradarlat,
						nframes, lc_n, nviews, verts, htype, dtype, nf, nsets,
						bf_stream, max_frames);

				delta_delcor0 = (deldop_zmax - deldop_zmax_save)*KM2US;
				if (cos_subradarlat != 0.0)
					dopscale_factor = cos_subradarlat_save/cos_subradarlat;
				if (rad_xsec != 0.0)
					radalb_factor = rad_xsec_save/rad_xsec;
				if (opt_brightness != 0.0)
					optalb_factor = opt_brightness_save/opt_brightness;
			}
			if ((newsize && vary_alb_size) || ((newshape || newspin) &&
					vary_alb_shapespin)) {
				realize_photo_gpu(dpar, dmod, radalb_factor, optalb_factor, 2, nf);  /* reset R, then R_save */

				/* Must update opt_brightness_save for Hapke optical scattering
				 * law, since single-scattering albedo w isn't just an overall
				 * scaling factor  */
				if (vary_hapke) {
					vary_params_gpu(dpar,dmod,ddat,12,&dummyval2,
							&dummyval3,&opt_brightness,&dummyval4,
							nframes, lc_n, nviews, verts, htype, dtype, nf, nsets,
							bf_stream, max_frames);
				}
			} else if (newphoto) {
				rad_xsec_save = rad_xsec;
				opt_brightness_save = opt_brightness;
				realize_photo_gpu(dpar, dmod, 1.0, 1.0, 0, nf);  /* set R_save to R */
			}
			if ((newsize && vary_delcor0_size) || ((newshape || newspin) &&
					vary_delcor0_shapespin)) {
				deldop_zmax_save = deldop_zmax;
				realize_delcor_gpu(ddat, delta_delcor0, 2, nsets, nframes);  /* reset delcor0, then delcor0_save */
			} else if (newdelcor)
				realize_delcor_gpu(ddat, 0.0, 0, nsets, nframes);  /* set delcor0_save to delcor0 */

			if ((newspin && vary_dopscale_spin) || ((newsize || newshape) &&
					vary_dopscale_sizeshape)) {
				cos_subradarlat_save = cos_subradarlat;
				realize_dopscale_gpu(dpar, ddat, dopscale_factor, 2, nsets, dtype);  /* reset dopscale, then dopscale_save */
			} else if (newdopscale) {
				realize_dopscale_gpu(dpar, ddat, 1.0, 0, nsets, dtype);  /* set dopscale_save to dopscale */
			}
			if (newxyoff)
				realize_xyoff_gpu(ddat, nsets, dtype);

			/* If the model extended beyond POS frame (sky rendering) for any
			 * trial parameter value(s), if it extended beyond any plane-of-
			 * sky fit frames, or if it was too wide in delay-Doppler space,
			 * evaluate model for best-fit parameter value to check if these
			 * problems persist - that is, to update "posbnd" "badposet" and
			 * "badradar" parameters for updated model.
			 * (This needn't be done for "baddiam" "badphoto" flags: if we've
			 * just finished adjusting an ellipsoid dimension or photometric
			 * parameter, realize_mod or realize_photo was called in code block
			 * above in order to realize the changed portion of model, and that
			 * call updated corresponding flag. Also we needn't worry about the
			 * "baddopscale" flag, since realize_dopscale was called above if
			 * Doppler scaling factors were changed.) The call to objective
			 * (*hotparam) first sets *hotparam (the parameter that we just
			 * adjusted) equal to itself (i.e., no change) and then calls
			 * calc_fits to evaluate the model for all datasets.          */
			if (check_posbnd || check_badposet || check_badradar)
				objective_gpu(hotparamval, verts, htype, dtype,
						nframes, nviews, lc_n, nsets, nf, bf_stream);

			/* Launch single-thread kernel to retrieve flags in dev_par */
			bf_get_flags_krnl<<<1,1>>>(dpar, flags);
			checkErrorAfterKernelLaunch("bf_get_flags_krnl");
			gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*7,
					hipMemcpyDeviceToHost));
			/* Display the objective function after each parameter adjustment.  */
			printf("%4d %8.6f %d", p, enderr, iround(par->fpartype[p]));
			if (hflags[0])		printf("  (BAD DIAMS)");
			if (hflags[1])		printf("  (BAD PHOTO)");
			if (hflags[2])		printf("  (BAD POS)");
			if (hflags[3])		printf("  (BAD POSET)");
			if (hflags[4])		printf("  (BAD RADAR)");
			if (hflags[5])		printf("  (BAD DOPSCALE)");
			printf("\n");
			fflush(stdout);

			/* Display reduced chi-square and individual penalty values after
			 * every 20th parameter adjustment. Setting showvals to 1 here
			 * means that these things will be displayed next time objective(x)
			 * is evaluated - at start of NEXT parameter adjustment.  Specifi-
			 * cally, they will be displayed when routine mnbrak evaluates
			 * objective(x) for *unadjusted* parameter value ax (see comment
			 * above).
			 * Also rewrite model and obs files after every 20th parameter
			 * adjustment. Most of obs file doesn't change, but some floating
			 * parameters (i.e. delay correction polynomial coefficients) do.  */
			if (++cntr >= npar_update) {
				cntr = 0;
				showvals = 1;
				calc_fits_gpu(dpar, dmod, ddat, verts, nviews,
						nframes, lc_n, htype, nsets, nf, bf_stream, max_frames);
				chi2_gpu(dpar, ddat, htype, dtype, nframes,
						lc_n, 0, nsets, bf_stream, max_frames);

//				write_mod( dpar, dmod);
//				write_dat( dpar, ddat);
			}
		}  // End fitpar loop

		/* End of this iteration: Write model and data to disk, and display the
		 * region within each delay-Doppler or Doppler frame for which model
		 * power is nonzero.                                               */
		if (cntr != 0) {
			calc_fits_gpu(dpar, dmod, ddat, verts, nviews,
					nframes, lc_n, htype, nsets, nf, bf_stream, max_frames);
			chi2_gpu(dpar, ddat, htype, dtype, nframes,
					lc_n, 0, nsets, bf_stream, max_frames);

//			write_mod( dpar, dmod);
//			write_dat( dpar, ddat);
		}
		show_deldoplim_gpu(ddat, htype, nsets, nframes, max_frames);

		/* Check if we should start a new iteration  */
		if (iter == term_maxiter) {
			/* Just completed last iteration permitted by "term_maxiter" para-
			 * meter, so stop iterating; note that since iter is 1-based, this
			 * test is always false if "term_maxiter" = 0 (its default value)  */
			keep_iterating = 0;

		} else if (first_fitpar > 0) {
			/* Just completed partial iteration (possible for iteration 1): if
			 * "objfunc_start" parameter was given, check if fractional decrea-
			 * se in objective function *relative to objfunc_start* during the
			 * just-completed iteration was larger than term_prec, thus
			 * justifying a new iteration; if it wasn't specified, definitely
			 * proceed to a new iteration.                            */
			if (objfunc_start > 0.0)
				keep_iterating = ((objfunc_start - enderr)/enderr >= term_prec);
			else
				keep_iterating = 1;
			first_fitpar = 0;     /* for all iterations after the first iteration */

		} else if (term_badmodel && (hflags[0] || hflags[1] || hflags[2] ||
				hflags[3] || hflags[4] || hflags[5]) ) {

			/* Just completed a full iteration, stop iterating because "term_
			 * badmodel" parameter is turned on and model has a fatal flaw: it
			 * extends beyond POS frame OR it one or more illegal photometric
			 * parameters OR it has one or more tiny or negative ellipsoid dia-
			 * meters OR it has plane-of-sky fit frames too small to "contain"
			 * model OR it is too wide in delay-Doppler space for (delay-)
			 * Doppler fit frames to be correctly constructed OR it has out-of-
			 * range values for one or more Doppler scaling factors    */
			keep_iterating = 0;

		} else {
			/* Just completed a full iteration and the model has no fatal flaws
			 * (or else the "term_badmodel" parameter is turned off): keep
			 * iterating if fractional decrease objective function during the
			 * just-completed iteration was greater than term_prec         */
			keep_iterating = ((beginerr - enderr)/enderr >= term_prec);
		}

	} while (keep_iterating);

		/* Show final values of reduced chi-square, individual penalty functions,
		 * and the objective function  */
		final_chi2 = chi2_gpu(dpar, ddat, htype, dtype, nframes,
				lc_n, 1, nsets, bf_stream, max_frames);

		final_redchi2 = final_chi2/dat->dof;
		printf("# search completed\n");

		/* Launch single-thread kernel to get these final flags from dev->par:
		 * pen.n, baddiam, badphoto, posbnd, badposet, badradar, baddopscale */
		/* Launch single-thread kernel to retrieve flags in dev_par */
		bf_get_flags_krnl<<<1,1>>>(dpar, flags);
		checkErrorAfterKernelLaunch("bf_get_flags_krnl");
		gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*7,
				hipMemcpyDeviceToHost));

		if (par->pen.n > 0 || hflags[0] || hflags[1] || hflags[2]	|| hflags[3] ||
				hflags[4] || hflags[5]) {
			printf("#\n");
			printf("# %15s %e\n", "reduced chi2", final_redchi2);
			if (par->pen.n > 0) {
				par->showstate = 1;
			penalties_gpu(dpar, dmod, ddat);
			par->showstate = 0;
		}
		if (hflags[0])
			printf("# objective func multiplied by %.1f: illegal ellipsoid diameters\n",
					baddiam_factor);
		if (hflags[1])
			printf("# objective func multiplied by %.1f: illegal photometric parameters\n",
					badphoto_factor);
		if (hflags[2])
			printf("# objective func multiplied by %.1f: model extends beyond POS frame\n",
					posbnd_factor);
		if (hflags[3])
			printf("# objective func multiplied by %.1f: "
					"model extends beyond plane-of-sky fit image\n",
					badposet_factor);
		if (hflags[4])
			printf("# objective func multiplied by %.1f: "
					"model is too wide in delay-Doppler space to construct fit image\n",
					badradar_factor);
		if (hflags[5])
			printf("# objective func multiplied by %.1f: illegal Doppler scaling factors\n",
					baddopscale_factor);
		printf("# ----------------------------\n");
		printf("# %15s %e\n", "objective func", enderr);
		printf("#\n");
	}
	intifpossible( dofstring, MAXLEN, dat->dof, SMALLVAL, "%f");
	printf("# final chi2 = %e for %s dof (reduced chi2 = %f)\n",
			final_chi2, dofstring, final_redchi2);
	printf("#\n");
	printf("\nIterations total: %i\n", iter);
	printf("GPU fit enderr: %g\n", enderr);
	fflush(stdout);

	/* Destroy the streams */
	hipSetDevice(GPU0);
	for (int f=0; f<max_frames; f++)
		hipStreamDestroy(bf_stream[f]);



	free(hflags);
	free(htype);
	free(nframes);
	free(lc_n);
	free(nviews);
	free(hfparstep);
	free(hfpartol);
	free(hfparabstol);
	free(hfpartype);
	hipFree(sdev_par);
	hipFree(sdev_mod);
	hipFree(sdev_dat);
	hipFree(fparstep);
	hipFree(fpartol);
	hipFree(fparabstol);
	hipFree(fpartype);
	hipFree(fpntr);
	hipFree(flags);
	hipFree(dtype);
	hipFree(verts);
	hipDeviceReset();
	//hipProfilerStop();
	return enderr;
}

__host__ double bestfit_gpu_pthreads(struct par_t *dpar, struct par_t *dpar1,
		struct mod_t *dmod, struct mod_t *dmod1, struct dat_t *ddat, struct
		dat_t *ddat1, struct par_t *par, struct par_t *par1, struct mod_t *mod,
		struct mod_t *mod1,	struct dat_t *dat, struct dat_t *dat1, pthread_t
		thread1, pthread_t thread2)
{
	char hostname[MAXLEN], dofstring[MAXLEN];
	int i, iter=0, p, cntr, first_fitpar, partype, keep_iterating=1, ilaw, nf, term_maxiter;
	long pid_long;
	pid_t pid;
	double beginerr, enderr, ax, bx, cx, obja, objb, objc, xmin, final_chi2,
	final_redchi2, dummyval2, dummyval3, dummyval4, delta_delcor0,
	dopscale_factor, radalb_factor, optalb_factor, *hfparstep, *hfpartol,
	*hfparabstol, objfunc_start, term_prec;
	unsigned char *flags, *hflags, *htype, *dtype0, *dtype1, action, avoid_badpos, term_badmodel;
	int nsets, *nframes, *lc_n, *nviews, nfpar, *hfpartype, npar_update,
	max_frames=0, max_streams=0, *GPUID;
	struct vertices_t **verts0, **verts1;	/* One for each GPU */
	dim3 THD, BLK;

	gpuErrchk(hipSetDevice(GPU0));
	/* This section collects parameters used for CUDA kernel launches throughout
	 * the program.  The cudaStreams created here are used/re-used for the
	 * lifetime of one program run */
	nsets = dat->nsets;
	nfpar = par->nfpar;
	nf = mod->shape.comp[0].real.nf;
	action = par->action;
	npar_update = par->npar_update;
	avoid_badpos = par->avoid_badpos;
	objfunc_start = par->objfunc_start;
	term_prec = par->term_prec;
	term_badmodel = par->term_badmodel;
	type = mod->shape.comp[0].type;
	htype 	= (unsigned char *) malloc(nsets*sizeof(unsigned char));
	nframes = (int *) malloc(nsets*sizeof(int));
	lc_n	= (int *) malloc(nsets*sizeof(int));
	nviews 	= (int *) malloc(nsets*sizeof(int));
	GPUID 	= (int *) malloc(nsets*sizeof(int));
	gpuErrchk(hipMalloc((void**)&dtype0, sizeof(unsigned char)*nsets));
	gpuErrchk(hipMalloc((void**)&verts0, sizeof(struct vertices_t*)*2));
	gpuErrchk(hipSetDevice(GPU1));
	gpuErrchk(hipMalloc((void**)&dtype1, sizeof(unsigned char)*nsets));
	gpuErrchk(hipMalloc((void**)&verts1, sizeof(struct vertices_t*)*2));
	gpuErrchk(hipSetDevice(GPU0));

	for (int s=0; s<nsets; s++) {
		htype[s] = dat->set[s].type;
		switch (htype[s]) {
		case DELAY:
			nframes[s]	= dat->set[s].desc.deldop.nframes;
			nviews[s]	= dat->set[s].desc.deldop.nviews;
			GPUID[s]	= dat->set[s].inputnode;
			lc_n[s]		= 0;
			break;
		case DOPPLER:
			nframes[s] = dat->set[s].desc.doppler.nframes;
			nviews[s]  = dat->set[s].desc.doppler.nviews;
			GPUID[s]	= dat->set[s].inputnode;
			lc_n[s]    = 0;
			break;
		case POS:
			nframes[s] = dat->set[s].desc.poset.nframes;
			nviews[s]  = dat->set[s].desc.poset.nviews;
			GPUID[s]	= dat->set[s].inputnode;
			lc_n[s]    = 0;
			break;
		case LGHTCRV:
			nframes[s] = dat->set[s].desc.lghtcrv.ncalc;
			nviews[s]  = dat->set[s].desc.lghtcrv.nviews;
			GPUID[s]	= dat->set[s].inputnode;
			lc_n[s]    = dat->set[s].desc.lghtcrv.n;
			break;
		}
		if (nframes[s]>max_frames)	max_frames = nframes[s];
	}
	gpuErrchk(hipMemcpy(dtype0, htype, sizeof(unsigned char)*nsets,
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dtype1, htype, sizeof(unsigned char)*nsets,
			hipMemcpyHostToDevice));

	/* The following check is necessary to ensure the dVdIdCOM reduction has
	 * enough streams to operate properly	 */
	if (max_frames < 13) max_streams = 13;
	else	max_streams = max_frames;

	/* Create streams for gpu0 and gpu1 */
	gpuErrchk(hipSetDevice(GPU0));
	hipStream_t gpu0_stream[max_streams];
	for (int f=0; f<max_streams; f++)
		gpuErrchk(hipStreamCreate(&gpu0_stream[f]));
	gpuErrchk(hipSetDevice(GPU1));
	hipStream_t gpu1_stream[max_streams];
	for (int f=0; f<max_streams; f++)
		gpuErrchk(hipStreamCreate(&gpu1_stream[f]));
	gpuErrchk(hipSetDevice(GPU0));

	/*..........................End section..................................*/

	/* Get the hostname of host machine and the PID */
	(void) gethostname(hostname, MAXLEN-1);
	pid = getpid();
	pid_long = (long) pid;  /* Assumes pid_t fits in a long */
	printf("#\n# CUDA fit (pid %ld on %s)\n", pid_long, hostname);
	fflush(stdout);

	/* Allocate memory for pointers, steps, and tolerances on both host and
	 * device. fpntr remains a hipMallocManaged allocation because it is a
	 * double pointer.  */
	gpuErrchk(hipMalloc((void**)&sdev_par, sizeof(struct par_t)));
	gpuErrchk(hipMemcpy(sdev_par, &par, sizeof(struct par_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&sdev_mod, sizeof(struct mod_t)));
	gpuErrchk(hipMemcpy(sdev_mod, &mod, sizeof(struct mod_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&sdev_dat, sizeof(struct dat_t)));
	gpuErrchk(hipMemcpy(sdev_dat, &dat, sizeof(struct dat_t), hipMemcpyHostToDevice));

	gpuErrchk(hipSetDevice(GPU1));
	gpuErrchk(hipMalloc((void**)&sdev_par1, sizeof(struct par_t)));
	gpuErrchk(hipMemcpy(sdev_par1, &par1, sizeof(struct par_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&sdev_mod1, sizeof(struct mod_t)));
	gpuErrchk(hipMemcpy(sdev_mod1, &mod1, sizeof(struct mod_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&sdev_dat1, sizeof(struct dat_t)));
	gpuErrchk(hipMemcpy(sdev_dat1, &dat1, sizeof(struct dat_t), hipMemcpyHostToDevice));
	gpuErrchk(hipSetDevice(GPU0));

	gpuErrchk(hipMalloc((void**)&flags, sizeof(unsigned char) * 6));
	gpuErrchk(hipMalloc((void**)&fparstep,   sizeof(double)  * nfpar));
	gpuErrchk(hipMalloc((void**)&fpartol,    sizeof(double)  * nfpar));
	gpuErrchk(hipMalloc((void**)&fparabstol, sizeof(double)  * nfpar));
	gpuErrchk(hipMalloc((void**)&fpartype,   sizeof(int) 	  * nfpar));
	cudaCalloc1((void**)&fpntr,	  sizeof(double*), nfpar);
	hfparstep 	 = (double *) malloc(nfpar*sizeof(double));
	hfpartol	 = (double *) malloc(nfpar*sizeof(double));
	hfparabstol  = (double *) malloc(nfpar*sizeof(double));
	hfpartype 	 = (int *) 	  malloc(nfpar*sizeof(int));
	hflags 		 = (unsigned char *) malloc(6*sizeof(unsigned char));

	for (i=0; i<nfpar; i++)
		gpuErrchk(hipMalloc((void**)&fpntr[i], sizeof(double) * 1));

	/* Set vertices shortcut and also set max_frames (the maximum number of
	 * frames for any one set) to device so that objective_gpu
	 * can retrieve it later. Have to set verts0 for gpu0 and verts1 for gpu1 */
	gpuErrchk(hipSetDevice(GPU0));
	set_verts_shortcut_krnl<<<1,1>>>(dmod, verts0, max_frames);
	checkErrorAfterKernelLaunch("set_verts_shortcut_krnl");
	gpuErrchk(hipSetDevice(GPU1));
	set_verts_shortcut_krnl<<<1,1>>>(dmod1, verts1, max_frames);
	checkErrorAfterKernelLaunch("set_verts_shortcut_krnl");
	gpuErrchk(hipSetDevice(GPU0));

	/* Initialize static global pointers used by objective(x) below
      to be compatible with "Numerical Recipes in C" routines       */
	spar = par;			smod = mod;			sdat = dat;
	sdev_par = dpar;	sdev_mod = dmod;	sdev_dat = ddat;
	sdev_par1 = dpar1;	sdev_mod1 = dmod1;	sdev_dat1 = ddat1;

	/*  Initialize static global parameters  */
	newsize = newshape = newspin = newphoto = newdelcor = newdopscale = newxyoff = 1;
	deldop_zmax = deldop_zmax_save = 0.0;
	cos_subradarlat = cos_subradarlat_save = 0.0;
	rad_xsec = rad_xsec_save = 0.0;
	opt_brightness = opt_brightness_save = 0.0;
	vary_delcor0_size = (par->vary_delcor0 != VARY_NONE);
	vary_delcor0_shapespin = (par->vary_delcor0 == VARY_ALL);
	vary_dopscale_spin = (par->vary_dopscale != VARY_NONE);
	vary_dopscale_sizeshape = (par->vary_dopscale == VARY_ALL);
	vary_alb_size = (par->vary_radalb != VARY_NONE || par->vary_optalb != VARY_NONE);
	vary_alb_shapespin = (par->vary_radalb == VARY_ALL || par->vary_optalb == VARY_ALL);
	vary_hapke = 0;
	if (par->vary_optalb != VARY_NONE)
		for (ilaw=0; ilaw<mod->photo.noptlaws; ilaw++)
			if (mod->photo.opttype[ilaw] == HAPKE || mod->photo.opttype[ilaw] == HARMHAPKE
					|| mod->photo.opttype[ilaw] == INHOHAPKE)
				vary_hapke = 1;
	call_vary_params = (par->vary_delcor0 != VARY_NONE || par->vary_dopscale != VARY_NONE
			|| par->vary_radalb != VARY_NONE
			|| par->vary_optalb != VARY_NONE);

	/*  Initialize local parameters  */
	delta_delcor0 = 0.0;
	dopscale_factor = radalb_factor = optalb_factor = 1.0;

	/* The following call sets up the parameter lists allocated above and copy
	 * the device contents to host copies */
	mkparlist_gpu(dpar, dmod,	ddat, fparstep, fpartol, fparabstol, fpartype,
			fpntr, nfpar, nsets);
	gpuErrchk(hipMemcpy(hfparstep, fparstep, sizeof(double)*nfpar, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hfpartol, fpartol, sizeof(double)*nfpar, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hfparabstol, fparabstol, sizeof(double)*nfpar, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hfpartype,	fpartype, sizeof(int)*nfpar, hipMemcpyDeviceToHost));

	/* Compute deldop_zmax_save, cos_subradarlat_save, rad_xsec_save, and
	 * opt_brightness_save for the initial model  */
	if (call_vary_params) {

		realize_mod_pthread(dpar, dpar1, dmod, dmod1, type, nf, thread1,
				thread2, gpu0_stream, gpu1_stream);

		realize_photo_pthread(dpar, dpar1, dmod, dmod1, 1.0, 1.0, 0, nf,
				thread1, thread2);

		realize_spin_pthread(dpar, dpar1, dmod, dmod1, ddat, ddat1, htype,
				nframes, nviews, GPUID, nsets, thread1, thread2, gpu0_stream,
				gpu1_stream);

		vary_params_pthreads(dpar, dpar1, dmod, dmod1, ddat, ddat1, action,
				&deldop_zmax_save, &rad_xsec_save, &opt_brightness_save,
				&cos_subradarlat_save, nframes, lc_n, nviews, GPUID, verts0,
				verts1, htype, dtype0, dtype1, nf, nsets, max_frames, thread1,
				thread2, gpu0_stream, gpu1_stream);
	}
	printf("rad_xsec: %f\n", rad_xsec_save);
	printf("deldop_zmax: %f\n", (float)deldop_zmax_save);
	gpuErrchk(hipSetDevice(GPU0));

	/* Point hotparam to a dummy variable (dummyval) rather than to a model pa-
	 * rameter; then call objective(0.0) to set dummy variable = 0.0, realize
	 * the initial model, calculate the fits, return initial model's objective
	 * function as enderr.                          */
	bf_set_hotparam_initial_krnl<<<1,1>>>();
	checkErrorAfterKernelLaunch("bf_set_hotparam_initial_krnl");

	enderr = objective_pthreads(0.0, verts0, verts1, htype, dtype0, dtype1, nframes,
			nviews, lc_n, GPUID, nsets, nf, max_frames, thread1, thread2,
			gpu0_stream, gpu1_stream);

	printf("#\n# searching for best fit ...\n");
	printf("%4d %8.6f to begin", 0, enderr);

	/* Launch single-thread kernel to retrieve flags in dev_par
			flags[0] = dpar->baddiam;
			flags[1] = dpar->badphoto;
			flags[2] = dpar->posbnd;
			flags[3] = dpar->badposet;
			flags[4] = dpar->badradar;
			flags[5] = dpar->baddopscale;*/

	bf_get_flags_krnl<<<1,1>>>(dpar, flags);
	checkErrorAfterKernelLaunch("bf_get_flags_krnl");
	gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*6,
			hipMemcpyDeviceToHost));

	/* Now act on the flags just retrieved from dev_par */
	if (hflags[0])		printf("  (BAD DIAMS)");
	if (hflags[1])		printf("  (BAD PHOTO)");
	if (hflags[2])		printf("  (BAD POS)");
	if (hflags[3])		printf("  (BAD POSET)");
	if (hflags[4])		printf("  (BAD RADAR)");
	if (hflags[5])		printf("  (BAD DOPSCALE)");		printf("\n");
	fflush(stdout);

	/* Display the region within each delay-Doppler or Doppler frame that, ac-
	 * cording to initial model, has nonzero power. A warning is displayed if
	 * any region extends beyond the data limits: the vignetting is too tight,
	 * or else some model parameter (such as a delay correction polynomial co-
	 * efficient) is seriously in error.   */
	//show_deldoplim_gpu(ddat, htype, nsets, nframes, max_frames);
	show_deldoplim_pthread(ddat, ddat1, htype, nsets, nframes, max_frames, GPUID);
	/* Set the starting fit parameter for the first iteration only  */
	first_fitpar = par->first_fitpar;
	term_maxiter = par->term_maxiter;
	if (first_fitpar < 0 || first_fitpar >= nfpar) {
		printf("ERROR: need 0 <= first_fitpar < nparams (%d)\n", nfpar);
		bailout("bestfit.c\n");
	}

	/* Iteratively adjust model; for each iteration, step through all free pa-
	 * rameters, adjusting one parameter at a time so as to minimize the objec-
	 * tive function at each step. Stop when fractional decrease in the objec-
	 * tive function from one iteration to the next is less than term_prec.   */

	do {
		showvals = 1;        /* show reduced chi-square and penalties at beginning */
		beginerr = enderr;
		printf("# iteration %d %f", ++iter, beginerr);

		/* Launch single-thread kernel to retrieve flags in dev_par */
		bf_get_flags_krnl<<<1,1>>>(dpar, flags);
		checkErrorAfterKernelLaunch("bf_get_flags_krnl");
		gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*6,
				hipMemcpyDeviceToHost));

		/* Now act on the flags just retrieved from dev_par */
		if (hflags[0])		printf("  (BAD DIAMS)");
		if (hflags[1])		printf("  (BAD PHOTO)");
		if (hflags[2])		printf("  (BAD POS)");
		if (hflags[3])		printf("  (BAD POSET)");
		if (hflags[4])		printf("  (BAD RADAR)");
		if (hflags[5])		printf("  (BAD DOPSCALE)");		printf("\n");
		fflush(stdout);

		/* Show breakdown of chi-square by data type    */
		chi2_pthreads(dpar,	dpar1, ddat, ddat1, htype, dtype0, dtype1, nframes,
				lc_n, GPUID, 1, nsets, max_frames, thread1, thread2,
				gpu0_stream, gpu1_stream);

		/*  Loop through the free parameters  */
		cntr = first_fitpar % npar_update;
		//p = first_fitpar = 1;
		for (p=first_fitpar; p<nfpar; p++) {
			/*  Adjust only parameter p on this try  */
			bf_set_hotparam_pntr_krnl<<<1,1>>>(fpntr, fpartype, p);
			checkErrorAfterKernelLaunch("bf_set_hotparam_pntr_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&partype, HIP_SYMBOL(bf_partype), sizeof(int),
					0, hipMemcpyDeviceToHost));

			newsize = newshape = newspin = newphoto = newdelcor = newdopscale
					= newxyoff = 0;
			if 		(partype == SIZEPAR)		newsize	 	= 1;
			else if (partype == SHAPEPAR)		newshape 	= 1;
			else if (partype == SPINPAR)		newspin 	= 1;
			else if (partype == PHOTOPAR)		newphoto 	= 1;
			else if (partype == DELCORPAR)		newdelcor 	= 1;
			else if (partype == DOPSCALEPAR)	newdopscale	= 1;
			else if (partype == XYOFFPAR)		newxyoff 	= 1;

			/* If this is a size parameter AND model extends beyond POS frame
			 * AND the "avoid_badpos" parameter is turned on, shrink model by
			 * 5% at a time until it fits within the POS frame.
			 * We must start with the redundant model evaluation for the un-
			 * changed value of the size parameter, in case the first call to
			 * objective displays reduced chi-square and the penalty functions.  */
			if (avoid_badpos && partype == SIZEPAR) {
				bf_get_flags_krnl<<<1,1>>>(dpar, flags);
				checkErrorAfterKernelLaunch("bf_get_flags_krnl");
				gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*6,
						hipMemcpyDeviceToHost));

				/* Get value of (*hotparam) */
				bf_get_hotparam_val_krnl<<<1,1>>>();
				checkErrorAfterKernelLaunch("bf_get_hotparam_val_krnl");
				gpuErrchk(hipMemcpyFromSymbol(&hotparamval, HIP_SYMBOL(bf_hotparamval),
						sizeof(double),	0, hipMemcpyDeviceToHost));

				while (hflags[2]) {
					objective_pthreads(hotparamval, verts0, verts1, htype,
							dtype0, dtype1, nframes, nviews, lc_n, GPUID, nsets,
							nf,	max_frames, thread1, thread2, gpu0_stream,
							gpu1_stream);

					bf_get_flags_krnl<<<1,1>>>(dpar, flags);
					checkErrorAfterKernelLaunch("bf_get_flags_krnl");
					gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*6,
							hipMemcpyDeviceToHost));

					if (hflags[2]) {
						/* Set the value pointed to by hotparam to 0.95 of its
						 * previous value */
						bf_mult_hotparam_val_krnl<<<1,1>>>(0.95);
						checkErrorAfterKernelLaunch("bf_mult_hotparam_val_krnl");
					}
				}
			}

			/* Get value of (*hotparam) so that mnbrak can use it*/
			bf_get_hotparam_val_krnl<<<1,1>>>();
			checkErrorAfterKernelLaunch("bf_get_hotparam_val_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&hotparamval, HIP_SYMBOL(bf_hotparamval),
					sizeof(double),	0, hipMemcpyDeviceToHost));

			/* Use Numerical Recipes routine mnbrak to bracket a minimum in the
			 * objective function (reduced chi-square plus penalties) objec-
			 * tive(x), where x is the value of parameter p.  As initial trial
			 * parameter values, use ax (unadjusted value) and bx, that value
			 * incremented by the appropriate step size (length_step,spin_step,
			 * etc.). mnbrak returns 3 parameter values, with bx between ax
			 * and cx; note that ax and bx are changed from their input values.
			 * It also returns the 3 corresponding objective(x) values, where
			 * objb is less than obja and objc.  Hence there is at least one
			 * local minimum (but not necessarily *any* global minimum)
			 * somewhere between ax and cx.          */
			ax = hotparamval;
			bx = ax + hfparstep[p]; /* par usage us fine here */

			mnbrak_pthreads(&ax, &bx, &cx, &obja, &objb, &objc,
					objective_pthreads,	verts0,	verts1, htype, dtype0,	dtype1,
					nframes, nviews, lc_n, GPUID, nsets, nf, max_frames,
					thread1, thread2, gpu0_stream, gpu1_stream);

			/* Before homing in on local minimum, initialize flags that will
			 * tell us if model extended beyond POS frame (sky rendering) for
			 * any trial parameter value(s), if it extended beyond any POS ima-
			 * ges, and if it was too wide in delay-Doppler space         */
			check_posbnd = 0;
			check_badposet = 0;
			check_badradar = 0;

			/* Now use Numerical Recipes function brent to find local minimum -
			 * that is, to find xmin, the best value of x, to within the
			 * *fractional* tolerance specified for parameter p (length_tol,
			 * spin_tol, etc.). brent's return value is the minimized objective
			 * function, objective(xmin). If more than one local minimum bet-
			 * ween ax and cx, brent might not find the best one. brent_abs is
			 * a modified version of brent that has an absolute fitting tole-
			 * rance as one of its arguments, in addition to the existing
			 * fractional tolerance.                                      */

			enderr = brent_abs_pthreads(ax, bx, cx,
					objective_pthreads, hfpartol[p], hfparabstol[p], &xmin,
					verts0, verts1, htype, dtype0, dtype1, nframes, nviews,
					lc_n, GPUID, nsets, nf, max_frames, thread1, thread2,
					gpu0_stream, gpu1_stream);

			/* Realize whichever part(s) of the model has changed.
			 *
			 * The code here is somewhat opaque because more than one part of
			 * the model may have changed - if the "vary_delcor0" "vary_radalb"
			 * and/or "vary_optalb" parameter is being used to permit joint pa-
			 * rameter adjustments. Before calling the vary_params routine, the
			 * size/shape and spin states must be realized (realize_mod and
			 * realize_spin); if albedos are being varied jointly with other
			 * parameters, the photometric state must also be realized
			 * (realize_photo); and in either case the 0th-order delay correc-
			 * tion polynomial coefficients must be reset to their saved
			 * values via the appropriate call to realize_delcor.          */
			/* Set the value pointed to by hotparam to 0.95 of its
			 * previous value (*hotparam) = xmin; */
			bf_set_hotparam_val_krnl<<<1,1>>>(xmin);
			checkErrorAfterKernelLaunch("bf_set_hotparam_val_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&hotparamval, HIP_SYMBOL(bf_hotparamval),
					sizeof(double),	0, hipMemcpyDeviceToHost));

			if (newsize || newshape)
				realize_mod_pthread(dpar, dpar1, dmod, dmod1, type, nf, thread1,
					thread2, gpu0_stream, gpu1_stream);
			if (newspin)
				realize_spin_pthread(dpar, dpar1, dmod, dmod1, ddat, ddat1,
						htype, nframes, GPUID, nviews, nsets, thread1, thread2,
						gpu0_stream, gpu1_stream);

			if ((newsize && vary_alb_size) || ((newshape ||
					newspin) && vary_alb_shapespin))
				realize_photo_pthread(dpar, dpar1, dmod, dmod1, 1.0, 1.0, 1, nf,
						thread1, thread2);
			if ((newsize && vary_delcor0_size) || ((newshape || newspin)
					&& vary_delcor0_shapespin)) {
				realize_delcor_pthreads(ddat, ddat1, 0.0, 1, nsets, nframes,
						GPUID, htype, thread1, thread2);
				/* set delcor0 to delcor0_save */
			}
			if ((newspin && vary_dopscale_spin) || ((newsize || newshape)
					&& vary_dopscale_sizeshape))
				realize_dopscale_pthreads(dpar, dpar1, ddat, ddat1, 1.0, 1,
						nsets, dtype0, dtype1, GPUID);/* set dopscale to dopscale_save */

			if (call_vary_params) {
				/* Call vary_params to get the adjustments to 0th-order delay
				 * correction polynomial coefficients, to Doppler scaling fac-
				 * tors, and to radar and optical albedos                  */
				vary_params_pthreads(dpar, dpar1, dmod, dmod1, ddat, ddat1, 11,
						&deldop_zmax, &rad_xsec, &opt_brightness,
						&cos_subradarlat, nframes, lc_n, nviews, GPUID, verts0,
						verts1, htype, dtype0, dtype1, nf, nsets, max_frames,
						thread1, thread2, gpu0_stream, gpu1_stream);

				delta_delcor0 = (deldop_zmax - deldop_zmax_save)*KM2US;
				if (cos_subradarlat != 0.0)
					dopscale_factor = cos_subradarlat_save/cos_subradarlat;
				if (rad_xsec != 0.0)
					radalb_factor = rad_xsec_save/rad_xsec;
				if (opt_brightness != 0.0)
					optalb_factor = opt_brightness_save/opt_brightness;
			}
			if ((newsize && vary_alb_size) || ((newshape || newspin) &&
					vary_alb_shapespin)) {
				realize_photo_pthread(dpar, dpar1, dmod, dmod1, radalb_factor,
						optalb_factor, 2, nf, thread1, thread2);

				/* Must update opt_brightness_save for Hapke optical scattering
				 * law, since single-scattering albedo w isn't just an overall
				 * scaling factor  */
				if (vary_hapke) {
					vary_params_pthreads(dpar, dpar1, dmod, dmod1, ddat, ddat1,
							12, &dummyval2,	&dummyval3, &opt_brightness,
							&dummyval4, nframes, lc_n, nviews, GPUID, verts0,
							verts1, htype, dtype0, dtype1, nf, nsets, max_frames,
							thread1, thread2, gpu0_stream, gpu1_stream);
				}
			} else if (newphoto) {
				rad_xsec_save = rad_xsec;
				opt_brightness_save = opt_brightness;
				realize_photo_pthread(dpar, dpar1, dmod, dmod1, 1.0, 1.0, 0, nf,
						thread1, thread2);	/* set R_save to R */
			}
			if ((newsize && vary_delcor0_size) || ((newshape || newspin) &&
					vary_delcor0_shapespin)) {
				deldop_zmax_save = deldop_zmax;
				/* reset delcor0, then delcor0_save */
				realize_delcor_pthreads(ddat, ddat1, delta_delcor0, 2, nsets,
						nframes, GPUID, htype, thread1, thread2);
			} else if (newdelcor)
				realize_delcor_pthreads(ddat, ddat1, 0.0, 0, nsets, nframes,
						GPUID, htype, thread1, thread2);
			/* set delcor0_save to delcor0 */

			if ((newspin && vary_dopscale_spin) || ((newsize || newshape) &&
					vary_dopscale_sizeshape)) {
				cos_subradarlat_save = cos_subradarlat;
				/* reset dopscale, then dopscale_save */
				realize_dopscale_pthreads(dpar, dpar1, ddat, ddat1,
						dopscale_factor, 2, nsets, dtype0, dtype1, GPUID);
			} else if (newdopscale) {
				/* set dopscale_save to dopscale */
				realize_dopscale_pthreads(dpar, dpar1, ddat, ddat1, 1.0, 0,
						nsets, dtype0, dtype1, GPUID);
			}
			if (newxyoff)
				realize_xyoff_pthreads(ddat, ddat1, nsets, dtype0, dtype1, GPUID);

			/* If the model extended beyond POS frame (sky rendering) for any
			 * trial parameter value(s), if it extended beyond any plane-of-
			 * sky fit frames, or if it was too wide in delay-Doppler space,
			 * evaluate model for best-fit parameter value to check if these
			 * problems persist - that is, to update "posbnd" "badposet" and
			 * "badradar" parameters for updated model.
			 * (This needn't be done for "baddiam" "badphoto" flags: if we've
			 * just finished adjusting an ellipsoid dimension or photometric
			 * parameter, realize_mod or realize_photo was called in code block
			 * above in order to realize the changed portion of model, and that
			 * call updated corresponding flag. Also we needn't worry about the
			 * "baddopscale" flag, since realize_dopscale was called above if
			 * Doppler scaling factors were changed.) The call to objective
			 * (*hotparam) first sets *hotparam (the parameter that we just
			 * adjusted) equal to itself (i.e., no change) and then calls
			 * calc_fits to evaluate the model for all datasets.          */
			if (check_posbnd || check_badposet || check_badradar)
				objective_pthreads(hotparamval, verts0, verts1, htype, dtype0,
						dtype1, nframes, nviews, lc_n, GPUID, nsets, nf,
						max_frames, thread1, thread2, gpu0_stream, gpu1_stream);

			/* Launch single-thread kernel to retrieve flags in dev_par */
			bf_get_flags_krnl<<<1,1>>>(dpar, flags);
			checkErrorAfterKernelLaunch("bf_get_flags_krnl");
			gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*6,
					hipMemcpyDeviceToHost));
			/* Display the objective function after each parameter adjustment.  */
			printf("%4d %8.6f %d", p, enderr, iround(par->fpartype[p]));
			if (hflags[0])		printf("  (BAD DIAMS)");
			if (hflags[1])		printf("  (BAD PHOTO)");
			if (hflags[2])		printf("  (BAD POS)");
			if (hflags[3])		printf("  (BAD POSET)");
			if (hflags[4])		printf("  (BAD RADAR)");
			if (hflags[5])		printf("  (BAD DOPSCALE)");
			printf("\n");
			fflush(stdout);

			/* Display reduced chi-square and individual penalty values after
			 * every 20th parameter adjustment. Setting showvals to 1 here
			 * means that these things will be displayed next time objective(x)
			 * is evaluated - at start of NEXT parameter adjustment.  Specifi-
			 * cally, they will be displayed when routine mnbrak evaluates
			 * objective(x) for *unadjusted* parameter value ax (see comment
			 * above).
			 * Also rewrite model and obs files after every 20th parameter
			 * adjustment. Most of obs file doesn't change, but some floating
			 * parameters (i.e. delay correction polynomial coefficients) do.  */
			if (++cntr >= npar_update) {
				cntr = 0;
				showvals = 1;
				calc_fits_pthreads(dpar, dpar1, dmod, dmod1, ddat, ddat1,
						verts0, verts1, nviews, nframes, lc_n, GPUID, htype,
						nsets, nf, max_frames, thread1,	thread2, gpu0_stream,
						gpu1_stream);

				chi2_pthreads(dpar, dpar1, ddat, ddat1, htype, dtype0, dtype1,
						nframes, lc_n, GPUID, 0, nsets, max_frames, thread1,
						thread2, gpu0_stream, gpu1_stream);

//				write_mod( dpar, dmod);
//				write_dat( dpar, ddat);
			}
		}  // End fitpar loop

		/* End of this iteration: Write model and data to disk, and display the
		 * region within each delay-Doppler or Doppler frame for which model
		 * power is nonzero.                                               */
		if (cntr != 0) {
			calc_fits_pthreads(dpar, dpar1, dmod, dmod1, ddat, ddat1, verts0,
					verts1, nviews, nframes, lc_n, GPUID, htype, nsets, nf,
					max_frames, thread1, thread2, gpu0_stream, gpu1_stream);

			chi2_pthreads(dpar, dpar1, ddat, ddat1, htype, dtype0, dtype1,
					nframes, lc_n, GPUID, 0, nsets, max_frames, thread1,
					thread2, gpu0_stream, gpu1_stream);

//			write_mod( dpar, dmod);
//			write_dat( dpar, ddat);
		}
		show_deldoplim_pthread(ddat, ddat1, htype, nsets, nframes, max_frames, GPUID);

		/* Check if we should start a new iteration  */
		if (iter == term_maxiter) {
			/* Just completed last iteration permitted by "term_maxiter" para-
			 * meter, so stop iterating; note that since iter is 1-based, this
			 * test is always false if "term_maxiter" = 0 (its default value)  */
			keep_iterating = 0;

		} else if (first_fitpar > 0) {
			/* Just completed partial iteration (possible for iteration 1): if
			 * "objfunc_start" parameter was given, check if fractional decrea-
			 * se in objective function *relative to objfunc_start* during the
			 * just-completed iteration was larger than term_prec, thus
			 * justifying a new iteration; if it wasn't specified, definitely
			 * proceed to a new iteration.                            */
			if (objfunc_start > 0.0)
				keep_iterating = ((objfunc_start - enderr)/enderr >= term_prec);
			else
				keep_iterating = 1;
			first_fitpar = 0;     /* for all iterations after the first iteration */

		} else if (term_badmodel && (hflags[0] || hflags[1] || hflags[2] ||
				hflags[3] || hflags[4] || hflags[5]) ) {

			/* Just completed a full iteration, stop iterating because "term_
			 * badmodel" parameter is turned on and model has a fatal flaw: it
			 * extends beyond POS frame OR it one or more illegal photometric
			 * parameters OR it has one or more tiny or negative ellipsoid dia-
			 * meters OR it has plane-of-sky fit frames too small to "contain"
			 * model OR it is too wide in delay-Doppler space for (delay-)
			 * Doppler fit frames to be correctly constructed OR it has out-of-
			 * range values for one or more Doppler scaling factors    */
			keep_iterating = 0;

		} else {
			/* Just completed a full iteration and the model has no fatal flaws
			 * (or else the "term_badmodel" parameter is turned off): keep
			 * iterating if fractional decrease objective function during the
			 * just-completed iteration was greater than term_prec         */
			keep_iterating = ((beginerr - enderr)/enderr >= term_prec);
		}

	} while (keep_iterating);

	/* Show final values of reduced chi-square, individual penalty functions,
	 * and the objective function  */
	final_chi2 = chi2_pthreads(dpar, dpar1, ddat, ddat1, htype, dtype0, dtype1,
			nframes, lc_n, GPUID, 1, nsets, max_frames, thread1, thread2,
			gpu0_stream, gpu1_stream);

	final_redchi2 = final_chi2/dat->dof;
	printf("# search completed\n");

	/* Launch single-thread kernel to get these final flags from dev->par:
	 * pen.n, baddiam, badphoto, posbnd, badposet, badradar, baddopscale */
	/* Launch single-thread kernel to retrieve flags in dev_par */
	bf_get_flags_krnl<<<1,1>>>(dpar, flags);
	checkErrorAfterKernelLaunch("bf_get_flags_krnl");
	gpuErrchk(hipMemcpy(hflags, flags, sizeof(unsigned char)*6,
			hipMemcpyDeviceToHost));

	if (par->pen.n > 0 || hflags[0] || hflags[1] || hflags[2]	|| hflags[3] ||
			hflags[4] || hflags[5]) {
		printf("#\n");
		printf("# %15s %e\n", "reduced chi2", final_redchi2);
		if (par->pen.n > 0) {
			par->showstate = 1;
			penalties_gpu(dpar, dmod, ddat);
			par->showstate = 0;
		}
		if (hflags[0])
			printf("# objective func multiplied by %.1f: illegal ellipsoid diameters\n",
					baddiam_factor);
		if (hflags[1])
			printf("# objective func multiplied by %.1f: illegal photometric parameters\n",
					badphoto_factor);
		if (hflags[2])
			printf("# objective func multiplied by %.1f: model extends beyond POS frame\n",
					posbnd_factor);
		if (hflags[3])
			printf("# objective func multiplied by %.1f: "
					"model extends beyond plane-of-sky fit image\n",
					badposet_factor);
		if (hflags[4])
			printf("# objective func multiplied by %.1f: "
					"model is too wide in delay-Doppler space to construct fit image\n",
					badradar_factor);
		if (hflags[5])
			printf("# objective func multiplied by %.1f: illegal Doppler scaling factors\n",
					baddopscale_factor);
		printf("# ----------------------------\n");
		printf("# %15s %e\n", "objective func", enderr);
		printf("#\n");
	}
	intifpossible( dofstring, MAXLEN, dat->dof, SMALLVAL, "%f");
	printf("# final chi2 = %e for %s dof (reduced chi2 = %f)\n",
			final_chi2, dofstring, final_redchi2);
	printf("#\n");
	printf("\nIterations total: %i\n", iter);
	fflush(stdout);

	/* Destroy the streams */
	hipSetDevice(GPU0);
	for (int f=0; f<max_frames; f++)
		hipStreamDestroy(gpu0_stream[f]);

	hipSetDevice(GPU1);
	for (int f=0; f<max_frames; f++)
		hipStreamDestroy(gpu1_stream[f]);

	hipSetDevice(GPU0);

	free(hflags);
	free(htype);
	free(nframes);
	free(lc_n);
	free(nviews);
	free(hfparstep);
	free(hfpartol);
	free(hfparabstol);
	free(hfpartype);
	hipFree(sdev_par);
	hipFree(sdev_mod);
	hipFree(sdev_dat);
	hipFree(fparstep);
	hipFree(fpartol);
	hipFree(fparabstol);
	hipFree(fpartype);
	hipFree(fpntr);
	hipFree(flags);
	hipFree(dtype0);
	hipFree(dtype1);
	hipFree(verts0);
	hipFree(verts1 );
	hipDeviceReset();
	//hipProfilerStop();
	return enderr;
}

/* objective_gpu is a version of objective_cuda that takes an extra
 * argument - the cudaStreams created in bestfit_cuda2. The goal is to
 * reduce overhead from stream creation/destruction to a minimum by having
 * just one set number of streams per program run.  */
__host__ double objective_gpu(
		double x,
		struct vertices_t **verts,
		unsigned char *htype,
		unsigned char *dtype,
		int *nframes,
		int *nviews,
		int *lc_n,
		int nsets,
		int nf,
		hipStream_t *bf_stream)
{
	double err, pens, delta_delcor0, dopscale_factor, radalb_factor,
		optalb_factor, *dlogfactors, *hlogfactors;
	unsigned char *dflags, *hflags;
	int max_frames;

//	gpuErrchk(hipSetDevice(GPU0));
	gpuErrchk(hipMalloc((void**)&dflags, sizeof(unsigned char)*7));
	gpuErrchk(hipMalloc((void**)&dlogfactors, sizeof(double)*7));
	hflags 	 	= (unsigned char *) malloc(7*sizeof(unsigned char));
	hlogfactors	= (double *) malloc(7*sizeof(double));

	/* Initialize local parameters  */
	delta_delcor0 = 0.0;
	dopscale_factor = radalb_factor = optalb_factor = 1.0;

	/* Assign new trial value to the model parameter being adjusted  */
	bf_set_hotparam_val_krnl<<<1,1>>>(x);	//(*hotparam) = x;
	checkErrorAfterKernelLaunch("bf_set_hotparam_val_krnl (in objective_cuda)");
	gpuErrchk(hipMemcpyFromSymbol(&max_frames, HIP_SYMBOL(dmax_frames),
			sizeof(int),	0, hipMemcpyDeviceToHost));

	/* Realize whichever part(s) of the model have changed, then calculate root's
	 * contribution to chi-square.
	 * The code here is somewhat opaque because more than one part of the model
	 * may have changed - if the "vary_delcor0" "vary_dopscale" "vary_radalb" and
	 * /or "vary_optalb" parameter is being used to permit joint parameter ad-
	 * justments. Before calling the vary_params routine, the size/shape and spin
	 * states must be realized (realize_mod and realize_spin); if albedos are
	 * being varied jointly with other parameters, the photometric state must
	 * also be realized (realize_photo); and in either case the 0th-order delay
	 * correction polynomial coefficients and the Doppler scaling factors must be
	 * reset to their saved values via the appropriate calls to realize_delcor
	 * and realize_dopscale, respectively.*/

	if (newsize || newshape)
		realize_mod_gpu(sdev_par, sdev_mod, type, nf, bf_stream);
	if (newspin)
		realize_spin_gpu(sdev_par, sdev_mod, sdev_dat, htype, nframes,
				nviews, nsets, bf_stream);

	if ((newsize && vary_alb_size) || ((newshape || newspin) && vary_alb_shapespin))
		realize_photo_gpu(sdev_par, sdev_mod, 1.0, 1.0, 1, nf);  /* set R to R_save */
	if ((newsize && vary_delcor0_size) || ((newshape || newspin) && vary_delcor0_shapespin)) {
		realize_delcor_gpu(sdev_dat, 0.0, 1, nsets, nframes);  /* set delcor0 to delcor0_save */
	}

	if ((newspin && vary_dopscale_spin) || ((newsize || newshape) && vary_dopscale_sizeshape))
		realize_dopscale_gpu(sdev_par, sdev_dat, 1.0, 1, nsets, dtype);  /* set dopscale to dopscale_save */
	if (call_vary_params) {
		/* Call vary_params to get the trial adjustments to 0th-order delay correc-
		 * tion polynomial coefficients, to Doppler scaling factors,and to radar
		 * and optical albedos, then send them to the branch nodes  */

		vary_params_gpu(sdev_par, sdev_mod, sdev_dat, spar->action,
				&deldop_zmax, &rad_xsec, &opt_brightness, &cos_subradarlat,
				nframes, lc_n, nviews, verts, htype, dtype, nf, nsets,
				bf_stream, max_frames);

		delta_delcor0 = (deldop_zmax - deldop_zmax_save)*KM2US;
		if (cos_subradarlat != 0.0)
			dopscale_factor = cos_subradarlat_save/cos_subradarlat;
		if (rad_xsec != 0.0)
			radalb_factor = rad_xsec_save/rad_xsec;
		if (opt_brightness != 0.0)
			optalb_factor = opt_brightness_save/opt_brightness;
	}

	if ((newsize && vary_alb_size) || ((newshape || newspin) && vary_alb_shapespin))
		realize_photo_gpu(sdev_par, sdev_mod, radalb_factor, optalb_factor, 1, nf);  /* adjust R */
	else if (newphoto)
		realize_photo_gpu(sdev_par, sdev_mod, 1.0, 1.0, 0, nf);  /* set R_save to R */
	if ((newsize && vary_delcor0_size) || ((newshape || newspin) && vary_delcor0_shapespin)) {
		realize_delcor_gpu(sdev_dat, delta_delcor0, 1, nsets, nframes);  /* adjust delcor0 */
	}
	else if (newdelcor) {
		realize_delcor_gpu(sdev_dat, 0.0, 0, nsets, nframes);  /* set delcor0_save to delcor0 */
	}
	if ((newspin && vary_dopscale_spin) || ((newsize || newshape) && vary_dopscale_sizeshape))
		realize_dopscale_gpu(sdev_par, sdev_dat, dopscale_factor, 1, nsets, dtype);  /* adjust dopscale */
	else if (newdopscale)
		realize_dopscale_gpu(sdev_par, sdev_dat, 1.0, 0, nsets, dtype);  /* set dopscale_save to dopscale */
	if (newxyoff)
		realize_xyoff_gpu(sdev_dat, nsets, dtype);

	calc_fits_gpu(sdev_par, sdev_mod, sdev_dat, verts, nviews, nframes, lc_n,
			htype, nsets, nf, bf_stream, max_frames);
	err = chi2_gpu(sdev_par, sdev_dat, htype, dtype, nframes, lc_n, 0, nsets,
			bf_stream, max_frames);


	/* Divide chi-square by DOF to get reduced chi-square.    */
	err /= sdat->dof;
//	printf("(GPU MODE) chi2_gpu error: %g with DOF = %g\n", err, sdat->dof);
	/* If bestfit has set showvals = 1, display reduced chi-square. Then set
	 * spar->showstate = 1, so that when function penalties is called later,
	 * it "knows" that it should display the individual penalty values.
	 * Reset showstate to 0 if showvals = 0.  */
	if (showvals) {
		printf("# %15s %e\n", "reduced chi2", err);
		spar->showstate = 1;
	}
	else
		spar->showstate = 0;

	/* Compute penalties and add to reduced chi-square. Individual penalty values
	 * will be displayed if we set spar->showstate = 1 a few lines back.        */
	pens = penalties_gpu(sdev_par, sdev_mod, sdev_dat);
//	printf("(GPU MODE) penalties: %g\n", pens);
	err += pens;
//	printf("(GPU MODE) err + pens = %g\n", err);
//	showvals = 1;
	/* Double the objective function if there's an ellipsoid component with tiny
	 * or negative diameter, if any optical photometric parameters have invalid
	 * values, if any portion of the model lies outside specified POS window or
	 * outside any plane-of-sky fit image, or if model is too wide in delay-Dopp-
	 * ler space for any (delay-)Doppler fit image to be correctly constructed.
	 * This effectively rules out any models with any of these flaws.         */
	/* NOTE: TO-DO: baddiam may need to come from elsewhere other than spar.
	 * However, bestfit gets called only once and spar/smod/sdat gets copied
	 * only once.
	 * flags[0] = dpar->baddiam;
		flags[1] = dpar->badphoto;
		flags[2] = dpar->posbnd;
		flags[3] = dpar->badposet;
		flags[4] = dpar->badradar;
		flags[5] = dpar->baddopscale;

		dlogfactors[0] = dpar->bad_objfactor;
		dlogfactors[1] = dpar->baddiam_logfactor;
		dlogfactors[2] = dpar->badphoto_logfactor;
		dlogfactors[3] = dpar->posbnd_logfactor;
		dlogfactors[4] = dpar->badposet_logfactor;
		dlogfactors[5] = dpar->badradar_logfactor;
		dlogfactors[6] = dpar->baddopscale_logfactor;
	 */
	ocs_get_flags_krnl<<<1,1>>>(sdev_par, dflags, dlogfactors);
	checkErrorAfterKernelLaunch("bf_get_flags_krnl");
	gpuErrchk(hipMemcpy(hflags, dflags, sizeof(unsigned char)*7,
			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hlogfactors, dlogfactors, sizeof(double)*6,
			hipMemcpyDeviceToHost));

	if (hflags[0]) {
		baddiam_factor = hlogfactors[0] * exp(hlogfactors[1]);
		err *= baddiam_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: illegal ellipsoid diameters\n",
					baddiam_factor);
	}
	if (hflags[1]) {
		badphoto_factor = hlogfactors[0] * exp(hlogfactors[2]);
		err *= badphoto_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: illegal photometric parameters\n",
					badphoto_factor);
	}
	if (hflags[2]) {
		check_posbnd = 1;     /* tells bestfit about this problem */
		posbnd_factor = hlogfactors[0] * exp(hlogfactors[3]);
//		printf("# hlogfactors[0] = %g and hlogfactors[3] = %g\n", hlogfactors[0], hlogfactors[3]);
		err *= posbnd_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: model extends beyond POS frame\n",
					posbnd_factor);
	}
	if (hflags[3]) {
		check_badposet = 1;     /* tells bestfit about this problem */
		badposet_factor = hlogfactors[0] * exp(hlogfactors[4]);
		err *= badposet_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: plane-of-sky fit frame too small\n",
					badposet_factor);
	}
	if (hflags[4]) {
		check_badradar = 1;     /* tells bestfit about this problem */
		badradar_factor = hlogfactors[0] * exp(hlogfactors[5]);
		err *= badradar_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: model too wide in delay-Doppler space\n",
					badradar_factor);
	}
	if (hflags[5]) {
		baddopscale_factor = hlogfactors[0] * exp(hlogfactors[6]);
		err *= baddopscale_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: illegal Doppler scaling factors\n",
					baddopscale_factor);
	}

	/* Reset showvals to 0 if it had been 1 (i.e., turn off display of reduced
	 * chi-square and the individual penalty values).  */
	if (showvals)
		fflush( stdout);
	showvals = 0;

	free(hflags);
	free(hlogfactors);
	hipFree(dflags);
	hipFree(dlogfactors);
//	printf("(GPU MODE) err (return value): %g\n", err);
	return err;
}

/* objective_pthreads i similiar to the _gpu version, but takes additional
 * arguments. It is intended for multi-threaded host/dual-GPU mode.  The
 * extra arguments are *GPUID which identifies what set is assigned to which
 * host thread and gpu, pthreads thread1 and thread2, and 2 separate cudaStream
 * arrays (2 gpus)  */
__host__ double objective_pthreads(
		double x,
		struct vertices_t **verts0,
		struct vertices_t **verts1,
		unsigned char *htype,
		unsigned char *dtype0,
		unsigned char *dtype1,
		int *nframes,
		int *nviews,
		int *lc_n,
		int *GPUID,
		int nsets,
		int nf,
		int max_frames,
		pthread_t thread1,
		pthread_t thread2,
		hipStream_t *gpu0_stream,
		hipStream_t *gpu1_stream)
{
	double err, pens, delta_delcor0, dopscale_factor, radalb_factor,
		optalb_factor, *dlogfactors, *hlogfactors;
	unsigned char *dflags, *hflags;

	gpuErrchk(hipSetDevice(GPU0));
	gpuErrchk(hipMalloc((void**)&dflags, sizeof(unsigned char)*6));
	gpuErrchk(hipMalloc((void**)&dlogfactors, sizeof(double)*7));
	hflags 	 	= (unsigned char *) malloc(6*sizeof(unsigned char));
	hlogfactors	= (double *) malloc(7*sizeof(double));

	/* Initialize local parameters  */
	delta_delcor0 = 0.0;
	dopscale_factor = radalb_factor = optalb_factor = 1.0;

	/* Assign new trial value to the model parameter being adjusted  */
	bf_set_hotparam_val_krnl<<<1,1>>>(x);	//(*hotparam) = x;
	checkErrorAfterKernelLaunch("bf_set_hotparam_val_krnl (in objective_cuda)");

	/* Realize whichever part(s) of the model have changed, then calculate root's
	 * contribution to chi-square.
	 * The code here is somewhat opaque because more than one part of the model
	 * may have changed - if the "vary_delcor0" "vary_dopscale" "vary_radalb" and
	 * /or "vary_optalb" parameter is being used to permit joint parameter ad-
	 * justments. Before calling the vary_params routine, the size/shape and spin
	 * states must be realized (realize_mod and realize_spin); if albedos are
	 * being varied jointly with other parameters, the photometric state must
	 * also be realized (realize_photo); and in either case the 0th-order delay
	 * correction polynomial coefficients and the Doppler scaling factors must be
	 * reset to their saved values via the appropriate calls to realize_delcor
	 * and realize_dopscale, respectively.*/
	if (newsize || newshape)
		realize_mod_pthread(sdev_par, sdev_par1, sdev_mod, sdev_mod1, type, nf,
				thread1, thread2, gpu0_stream, gpu1_stream);
	if (newspin)
		realize_spin_pthread(sdev_par, sdev_par1, sdev_mod, sdev_mod1, sdev_dat,
				sdev_dat1, htype, nframes, nviews, GPUID, nsets, thread1,
				thread2, gpu0_stream, gpu1_stream);

	if ((newsize && vary_alb_size) || ((newshape || newspin) && vary_alb_shapespin))
		realize_photo_pthread(sdev_par, sdev_par1, sdev_mod, sdev_mod1, 1.0,
				1.0, 1, nf, thread1, thread2);
	if ((newsize && vary_delcor0_size) || ((newshape || newspin) && vary_delcor0_shapespin)) {
		/* set delcor0 to delcor0_save */
		realize_delcor_pthreads(sdev_dat, sdev_dat1, 0.0, 1, nsets, nframes, GPUID, htype,
				thread1, thread2);
	}

	if ((newspin && vary_dopscale_spin) || ((newsize || newshape) && vary_dopscale_sizeshape))
		/* set dopscale to dopscale_save */
		realize_dopscale_pthreads(sdev_par, sdev_par1, sdev_dat, sdev_dat1,
				1.0, 1, nsets, dtype0, dtype1,GPUID);
	if (call_vary_params) {
		/* Call vary_params to get the trial adjustments to 0th-order delay
		 * correction polynomial coefficients, to Doppler scaling factors, and
		 * to radar and optical albedos, then send them to the branch nodes  */
		vary_params_pthreads(sdev_par, sdev_par1, sdev_mod, sdev_mod1, sdev_dat,
				sdev_dat1, spar->action, &deldop_zmax, &rad_xsec,
				&opt_brightness,&cos_subradarlat, nframes, lc_n, nviews, GPUID,
				verts0, verts1, htype, dtype0, dtype1, nf, nsets, max_frames,
				thread1, thread2, gpu0_stream, gpu1_stream);

		delta_delcor0 = (deldop_zmax - deldop_zmax_save)*KM2US;
		if (cos_subradarlat != 0.0)
			dopscale_factor = cos_subradarlat_save/cos_subradarlat;
		if (rad_xsec != 0.0)
			radalb_factor = rad_xsec_save/rad_xsec;
		if (opt_brightness != 0.0)
			optalb_factor = opt_brightness_save/opt_brightness;
	}

	if ((newsize && vary_alb_size) || ((newshape || newspin) && vary_alb_shapespin))
		realize_photo_pthread(sdev_par, sdev_par1, sdev_mod, sdev_mod1,
				radalb_factor, optalb_factor, 1, nf, thread1, thread2);  /* adjust R */
	else if (newphoto)
		realize_photo_pthread(sdev_par, sdev_par1, sdev_mod, sdev_mod1, 1.0,
				1.0, 0, nf,	thread1, thread2);  /* set R_save to R */
	if ((newsize && vary_delcor0_size) || ((newshape || newspin) && vary_delcor0_shapespin)) {
		/* adjust delcor0 */
		realize_delcor_pthreads(sdev_dat, sdev_dat1, delta_delcor0, 1, nsets,
				nframes, GPUID, htype, thread1, thread2);
	}
	else if (newdelcor) {
		/* set delcor0_save to delcor0 */
		realize_delcor_pthreads(sdev_dat, sdev_dat1, 0.0, 0, nsets, nframes,
				GPUID, htype, thread1, thread2);
	}
	if ((newspin && vary_dopscale_spin) || ((newsize || newshape) &&
			vary_dopscale_sizeshape))
		/* adjust dopscale */
		realize_dopscale_pthreads(sdev_par, sdev_par1, sdev_dat, sdev_dat1,
				dopscale_factor, 1, nsets, dtype0, dtype1, GPUID);
	else if (newdopscale)
		/* set dopscale_save to dopscale */
		realize_dopscale_pthreads(sdev_par, sdev_par1, sdev_dat, sdev_dat1,
				1.0, 0, nsets, dtype0, dtype1, GPUID);
	if (newxyoff)
		realize_xyoff_pthreads(sdev_dat,sdev_dat1,nsets,dtype0,dtype1,GPUID);

	calc_fits_pthreads(sdev_par, sdev_par1, sdev_mod, sdev_mod1, sdev_dat,
			sdev_dat1, verts0, verts1, nviews, nframes, lc_n, GPUID, htype,
			nsets, nf, max_frames,thread1, thread2, gpu0_stream, gpu1_stream);
	err = chi2_pthreads(sdev_par, sdev_par1, sdev_dat, sdev_dat1, htype,
			dtype0,	dtype1, nframes, lc_n, GPUID, 0, nsets, max_frames, thread1,
			thread2, gpu0_stream, gpu1_stream);

	/* Divide chi-square by DOF to get reduced chi-square.    */
	err /= sdat->dof;

	/* If bestfit has set showvals = 1, display reduced chi-square. Then set
	 * spar->showstate = 1, so that when function penalties is called later,
	 * it "knows" that it should display the individual penalty values.
	 * Reset showstate to 0 if showvals = 0.  */
	if (showvals) {
		printf("# %15s %e\n", "reduced chi2", err);
		spar->showstate = 1;
	}
	else
		spar->showstate = 0;

	/* Compute penalties and add to reduced chi-square. Individual penalty values
	 * will be displayed if we set spar->showstate = 1 a few lines back.        */
	pens = penalties_gpu(sdev_par, sdev_mod, sdev_dat);
	err += pens;

	/* Double the objective function if there's an ellipsoid component with tiny
	 * or negative diameter, if any optical photometric parameters have invalid
	 * values, if any portion of the model lies outside specified POS window or
	 * outside any plane-of-sky fit image, or if model is too wide in delay-Dopp-
	 * ler space for any (delay-)Doppler fit image to be correctly constructed.
	 * This effectively rules out any models with any of these flaws.         */
	/* NOTE: TO-DO: baddiam may need to come from elsewhere other than spar.
	 * However, bestfit gets called only once and spar/smod/sdat gets copied
	 * only once.
	 * flags[0] = dpar->baddiam;
		flags[1] = dpar->badphoto;
		flags[2] = dpar->posbnd;
		flags[3] = dpar->badposet;
		flags[4] = dpar->badradar;
		flags[5] = dpar->baddopscale;

		dlogfactors[0] = dpar->bad_objfactor;
		dlogfactors[1] = dpar->baddiam_logfactor;
		dlogfactors[2] = dpar->badphoto_logfactor;
		dlogfactors[3] = dpar->posbnd_logfactor;
		dlogfactors[4] = dpar->badposet_logfactor;
		dlogfactors[5] = dpar->badradar_logfactor;
		dlogfactors[6] = dpar->baddopscale_logfactor;
	 */
	ocs_get_flags_krnl<<<1,1>>>(sdev_par, dflags, dlogfactors);
	checkErrorAfterKernelLaunch("bf_get_flags_krnl");
	gpuErrchk(hipMemcpy(hflags, dflags, sizeof(unsigned char)*6,
			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hlogfactors, dlogfactors, sizeof(double)*7,
			hipMemcpyDeviceToHost));

	if (hflags[0]) {
		baddiam_factor = hlogfactors[0] * exp(hlogfactors[1]);
		err *= baddiam_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: illegal ellipsoid diameters\n",
					baddiam_factor);
	}
	if (hflags[1]) {
		badphoto_factor = hlogfactors[0] * exp(hlogfactors[2]);
		err *= badphoto_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: illegal photometric parameters\n",
					badphoto_factor);
	}
	if (hflags[2]) {
		check_posbnd = 1;     /* tells bestfit about this problem */
		posbnd_factor = hlogfactors[0] * exp(hlogfactors[3]);
		err *= posbnd_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: model extends beyond POS frame\n",
					posbnd_factor);
	}
	if (hflags[3]) {
		check_badposet = 1;     /* tells bestfit about this problem */
		badposet_factor = hlogfactors[0] * exp(hlogfactors[4]);
		err *= badposet_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: plane-of-sky fit frame too small\n",
					badposet_factor);
	}
	if (hflags[4]) {
		check_badradar = 1;     /* tells bestfit about this problem */
		badradar_factor = hlogfactors[0] * exp(hlogfactors[5]);
		err *= badradar_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: model too wide in delay-Doppler space\n",
					badradar_factor);
	}
	if (hflags[5]) {
		baddopscale_factor = hlogfactors[0] * exp(hlogfactors[6]);
		err *= baddopscale_factor;
		if (showvals)
			printf("# objective func multiplied by %.1f: illegal Doppler scaling factors\n",
					baddopscale_factor);
	}

	/* Reset showvals to 0 if it had been 1 (i.e., turn off display of reduced
	 * chi-square and the individual penalty values).  */
	if (showvals)
		fflush( stdout);
	showvals = 0;

	free(hflags);
	free(hlogfactors);
	hipFree(dflags);
	hipFree(dlogfactors);
	return err;
}
