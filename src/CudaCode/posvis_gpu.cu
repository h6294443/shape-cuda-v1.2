#include "hip/hip_runtime.h"
/*****************************************************************************************
 posvis.c

 Fill in the portion of a plane-of-sky image due to a particular model component: Assign
 each relevant POS pixel a z-value in observer coordinates (distance from the origin
 towards Earth) and a value of cos(scattering angle).

 Return 1 if any portion of this component lies outside the specified POS window,
 0 otherwise.

 If the "src" argument is true, the "observer" is the Sun rather than Earth, and
 "plane-of-sky" becomes "projection as viewed from the Sun."

 Modified 2014 February 20 by CM:
 Allow facets that partly project outside the POS frame to contribute to the POS frame
 (thus avoiding see-through "holes" in the model at the edge of a POS image)

 Modified 2010 May 18 by CM:
 Bug fix: When checking if a POS pixel hasn't already been assigned
 values during a previous call to posvis for a different component,
 check for fac[i][j] < 0 rather than cosa[i][j] == 0.0, since for
 bistatic situations the latter condition will also be true for
 pixels centered on Earth-facing facets that don't face the Sun

 Modified 2009 July 2 by CM:
 Eliminate the check that facets are "active": this term is now being
 interpreted to mean "not lying interior to the model," so the
 check is unnecessary and the determination of active vs. inactive
 status is inaccurate for half-exposed facets at the intersections
 between model components

 Modified 2009 April 3 by CM:
 Compute the "posbnd_logfactor" parameter: if the model extends beyond
 the POS frame, posbnd_logfactor is set to the logarithm of the
 ratio of the area that would have been required to "contain" the
 entire model divided by the area of the actual POS frame
 Work with floating-point pixel numbers (imin_dbl, etc.), at least
 initially, in case the sky rendering for a model with illegal
 parameters would involve huge pixel numbers that exceed the
 limits for valid integers

 Modified 2007 August 4 by CM:
 Add "orbit_offset" and "body" parameters and remove "facet" parameter
 Add body, bodyill, comp, and compill matrices for POS frames

 Modified 2006 June 21 by CM:
 For POS renderings, change res to km_per_pixel

 Modified 2005 September 19 by CM:
 Allow for roundoff error when determining which POS pixels project
 onto each model facet

 Modified 2005 June 27 by CM:
 Renamed "round" function to "iround" to avoid conflicts

 Modified 2005 June 22 by CM:
 Slightly modified some comments

 Modified 2005 January 25 by CM:
 Take care of unused and uninitialized variables

 Modified 2004 December 19 by CM:
 Added more comments
 Put update of rectangular POS area into "POSrect" routine and applied it
 even to facets which lie outside the POS frame

 Modified 2004 Feb 11 by CM:
 Added comments

 Modified 2003 May 5 by CM:
 Removed redundant coordinate transformation of the unit normal n
 for the no-pvs_smoothing case
 *****************************************************************************************/
extern "C" {
#include "../shape/head.h"
#include <limits.h>
}
__device__ int posvis_streams_outbnd, pvst_smooth;

/* Note that the following custom atomic functions must be declared in each
 * file it is needed (consequence of being a static device function) */

__device__ static float atomicMaxf(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__global__ void posvis_init_krnl(
		struct par_t *dpar,
		struct pos_t **pos,
		float4 *ijminmax_overall,
		float3 *oa,
		float3 *usrc,
		int *outbndarr,
		int c,
		int start,
		int src,
		int size,
		int set) {

	/* nfrm_alloc-threaded */
	int f = blockIdx.x * blockDim.x + threadIdx.x + start;

	if (f < size) {
		if (f == start) {
			posvis_streams_outbnd = 0;
			pvst_smooth = dpar->pos_smooth;
		}
		ijminmax_overall[f].w = ijminmax_overall[f].y = HUGENUMBER;
		ijminmax_overall[f].x = ijminmax_overall[f].z = -HUGENUMBER;
		pos[f]->posbnd_logfactor = 0.0;

		dev_mtrnsps3(oa, pos[f]->ae, f);
		if (src) {
			/* We're viewing the model from the sun: at the center of each pixel
			 * in the projected view, we want cos(incidence angle), distance from
			 * the COM towards the sun, and the facet number.                */
			dev_mmmul3(oa, pos[f]->se, oa, f); /* oa takes ast into sun coords           */
		} else {
			/* We're viewing the model from Earth: at the center of each POS pixel
			 * we want cos(scattering angle), distance from the COM towards Earth,
			 * and the facet number.  For bistatic situations (lightcurves) we also
									 want cos(incidence angle) and the unit vector towards the source.     */
			dev_mmmul3(oa, pos[f]->oe, oa, f); /* oa takes ast into obs coords */
			if (pos[f]->bistatic) {
				usrc[f].x = usrc[f].y = 0.0; /* unit vector towards source */
				usrc[f].z = 1.0;
				dev_cotrans9(&usrc[f], pos[f]->se, usrc[f], -1);
				dev_cotrans9(&usrc[f], pos[f]->oe, usrc[f], 1); /* in observer coordinates */
			}
		}
		outbndarr[f] = 0;
	}
}

__global__ void posvis_facet_krnl(
		struct pos_t **pos,
		struct vertices_t **verts,
		float4 *ijminmax_overall,
		float3 orbit_offs,
		float3 *oa,
		float3 *usrc,
		int src,
		int body,
		int comp,
		int nfacets,
		int frm,
		int smooth,
		int *outbndarr,
		int set) {
	/* (nf * nframes)-threaded kernel.  This version eliminates as much double
	 * math as possible */

	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int pxa, i, i1, i2, j, j1, j2, imin, imax, jmin, jmax;
	float imin_dbl, imax_dbl, jmin_dbl, jmax_dbl, old, s, t, z, den;
	int3 fidx;
	float3 n, v0, v1, v2, tv0, tv1, tv2, x;
	int pn;
	float kmpxl;

	if (f < nfacets) {
		pn = pos[frm]->n;
		kmpxl = __double2float_rn(pos[frm]->km_per_pixel);
		/* The following section transfers vertex coordinates from double[3]
		 * storage to float3		 */
		fidx.x = verts[0]->f[f].v[0];
		fidx.y = verts[0]->f[f].v[1];
		fidx.z = verts[0]->f[f].v[2];
		tv0.x = __double2float_rn( verts[0]->v[fidx.x].x[0]);
		tv0.y = __double2float_rn(verts[0]->v[fidx.x].x[1]);
		tv0.z = __double2float_rn(verts[0]->v[fidx.x].x[2]);
		tv1.x = __double2float_rn(verts[0]->v[fidx.y].x[0]);
		tv1.y = __double2float_rn(verts[0]->v[fidx.y].x[1]);
		tv1.z = __double2float_rn(verts[0]->v[fidx.y].x[2]);
		tv2.x = __double2float_rn(verts[0]->v[fidx.z].x[0]);
		tv2.y = __double2float_rn(verts[0]->v[fidx.z].x[1]);
		tv2.z = __double2float_rn(verts[0]->v[fidx.z].x[2]);
		v0.x = v0.y = v0.z = v1.x = v1.y = v1.z = v2.x = v2.y = v2.z = 0.0;

		/* Get the normal to this facet in body-fixed (asteroid) coordinates
		 * and convert it to observer coordinates     */
		n.x = __double2float_rn(verts[0]->f[f].n[0]);
		n.y = __double2float_rn(verts[0]->f[f].n[1]);
		n.z = __double2float_rn(verts[0]->f[f].n[2]);

		dev_cotrans8(&n, oa, n, 1, frm);

		/* Consider this facet further only if its normal points somewhat
		 * towards the observer rather than away         */
		if (n.z > 0.0) {
			/* Convert the three sets of vertex coordinates from body to ob-
			 * server coordinates; orbit_offset is the center-of-mass offset
			 * (in observer coordinates) for this model at this frame's epoch
			 * due to orbital motion, in case the model is half of a binary
			 * system.  */
			dev_cotrans8(&v0, oa, tv0, 1, frm);
			dev_cotrans8(&v1, oa, tv1, 1, frm);
			dev_cotrans8(&v2, oa, tv2, 1, frm);

			v0.x += orbit_offs.x;	v0.y += orbit_offs.x;	v0.z += orbit_offs.x;
			v1.x += orbit_offs.y;	v1.y += orbit_offs.y;	v1.z += orbit_offs.y;
			v2.x += orbit_offs.z;	v2.y += orbit_offs.z;	v2.z += orbit_offs.z;

			/* Find rectangular region (in POS pixels) containing the projected
			 * facet - use floats in case model has illegal parameters and the
			 * pixel numbers exceed the limits for valid integers                         */
			imin_dbl = floor(MIN(v0.x,MIN(v1.x,v2.x)) / kmpxl
							- SMALLVAL + 0.5);
			imax_dbl = floor(MAX(v0.x,MAX(v1.x,v2.x)) / kmpxl
							+ SMALLVAL + 0.5);
			jmin_dbl = floor(MIN(v0.y,MIN(v1.y,v2.y)) / kmpxl
							- SMALLVAL + 0.5);
			jmax_dbl = floor(MAX(v0.y,MAX(v1.y,v2.y)) / kmpxl
							+ SMALLVAL + 0.5);
			imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
			imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
			jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
			jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

			/*  Set the outbnd flag if the facet extends beyond the POS window  */
			if ((imin < (-pn)) || (imax > pn) || (jmin < (-pn))	|| (jmax > pn)) {
				posvis_streams_outbnd = 1;
				atomicExch(&outbndarr[frm], 1);
			}

			/* Figure out if facet projects at least partly within POS window;
			 * if it does, look at each "contained" POS pixel and get the
			 * z-coordinate and cos(scattering angle)           */
			i1 = MAX(imin, -pn);		j1 = MAX(jmin, -pn);
			i2 = MIN(imax,  pn);		j2 = MIN(jmax,  pn);

			if (i1 > pn || i2 < -pn || j1 > pn || j2 < -pn) {

				/* Facet is entirely outside the POS frame: just keep track of
				 * changed POS region     */
				dev_POSrect_gpu(pos, src, imin_dbl, imax_dbl, jmin_dbl, jmax_dbl,
						ijminmax_overall, frm);

			} else {

				dev_POSrect_gpu(pos, src, (float)i1, (float)i2, (float)j1,
						(float)j2, ijminmax_overall, frm);

				/* Facet is at least partly within POS frame: find all POS
				 * pixels whose centers project onto this facet  */
				for (i = i1; i <= i2; i++) {
					x.x = i * kmpxl;
					for (j = j1; j <= j2; j++) {
						x.y = j * kmpxl;

						/* Calculate the pixel address for 1D arrays */
						pxa = (j+pn) * (2*pn + 1) + (i+pn);

						/* Compute parameters s(x,y) and t(x,y) which define a
						 * facet's surface as
						 *         z = z0 + s*(z1-z0) + t*(z2-z1)
						 * where z0, z1, and z2 are the z-coordinates at the
						 * vertices. The conditions 0 <= s <= 1 and
						 * 0 <= t <= s require the POS pixel center to be
						 * "within" the (projected) perimeter of facet f.    */
						den = 1	/ ((v1.x - v0.x) * (v2.y - v1.y)
								 - (v2.x - v1.x) * (v1.y - v0.y));
						s = ((x.x - v0.x) * (v2.y - v1.y)
						  - (v2.x - v1.x) * (x.y - v0.y)) * den;

						if ((s >= -SMALLVAL) && (s <= 1.0 + SMALLVAL)) {

							t = ((v1.x - v0.x) * (x.y - v0.y)
							    - (x.x- v0.x) * (v1.y- v0.y)) * den;
							if ((t >= -SMALLVAL) && (t <= s + SMALLVAL)) {

								/* Compute z-coordinate of pixel center: its
								 * distance measured from the origin towards
								 * Earth.    */
								z = v0.z + s*(v1.z-v0.z) + t*(v2.z-v1.z);

								/* If fac[i][j] is >= 0, pixel [i][j] was al-
								 * ready assigned values during a previous call
								 * to posvis for a different model component.
								 * If so, override only if the current component
								 * is blocking our view of (i.e., is closer to
								 * us than) the previous one.   */

								/* Following line replaces the previous if check
								 * for z > zz[i][j]
								 * atomicMaxf returns the value that was sitting
								 * at zzf[pxa] at time of call.  So if that value
								 * matches the z we compared to*/

								if (src)
									old = atomicMaxf(&pos[frm]->zill_s[pxa], z);
								else
									old = atomicMaxf(&pos[frm]->z_s[pxa], z);

								if (old < z || pos[frm]->fill[i][j] < 0 ||
										pos[frm]->f[i][j] < 0) {

									/* Next line assigns distance of POS pixel
									 * center from COM towards Earth; that is,
									 * by changing zz,it changes pos->z or
									 * pos->zill                */
									/* following line is a first time z calc
									 * for this pixel  */
									if ( (pos[frm]->fill[i][j] < 0) || (pos[frm]->f[i][j] < 0)){
										if (src)	atomicExch(&pos[frm]->zill_s[pxa], z);
										else 		atomicExch(&pos[frm]->z_s[pxa], z);
									}

									if (pvst_smooth) {
										/* Assign temp. normal components as float3 */
										tv0.x = __double2float_rn(verts[0]->v[fidx.x].n[0]);
										tv0.y = __double2float_rn(verts[0]->v[fidx.x].n[1]);
										tv0.z = __double2float_rn(verts[0]->v[fidx.x].n[2]);
										tv1.x = __double2float_rn(verts[0]->v[fidx.y].n[0]);
										tv1.y = __double2float_rn(verts[0]->v[fidx.y].n[1]);
										tv1.z = __double2float_rn(verts[0]->v[fidx.y].n[2]);
										tv2.x = __double2float_rn(verts[0]->v[fidx.z].n[0]);
										tv2.y = __double2float_rn(verts[0]->v[fidx.z].n[1]);
										tv2.z = __double2float_rn(verts[0]->v[fidx.z].n[2]);

										/* Get pvs_smoothed version of facet unit
										 * normal: Take the linear combination
										 * of the three vertex normals; trans-
										 * form from body to observer coordina-
										 * tes; and make sure that it points
										 * somewhat in our direction.         */
										n.x = tv0.x + s * (tv1.x - tv0.x) + t * (tv2.x - tv1.x);
										n.y = tv0.y + s * (tv1.y - tv0.y) + t * (tv2.y - tv1.y);
										n.z = tv0.z + s * (tv1.z - tv0.z) + t * (tv2.z - tv1.z);
										dev_cotrans8(&n, oa, n, 1, frm);
										dev_normalize2(n);
									}

									/* Determine scattering and/or incidence
									 * angles. Next lines change pos->cose/
									 * cosill. If bistatic (lightcurves), where
									 * we are viewing from Earth (src = 0),
									 * pos->cosi is also changed.                 */
									if (n.z > 0.0) {
										if (src)
											atomicExch(&pos[frm]->cosill_s[pxa], n.z);
										else
											atomicExch(&pos[frm]->cose_s[pxa], n.z);
										if ((!src) && (pos[frm]->bistatic)) {
											float temp = dev_dot_f3(n,usrc[frm]);
											atomicExch(&pos[frm]->cosi_s[pxa], temp);
											if (pos[frm]->cosi_s[pxa] <= 0.0)
												pos[frm]->cose_s[pxa] = 0.0;
										}
									}

									/* Next lines change pos->body/bodyill,
									 * pos->comp/compill, pos->f/fill          */
									if (src) {
										pos[frm]->bodyill[i][j] = body;
										pos[frm]->compill[i][j] = comp;
										pos[frm]->fill[i][j] = f;
									} else {
										pos[frm]->body[i][j] = body;
										pos[frm]->comp[i][j] = comp;
										pos[frm]->f[i][j] = f;
									}

								} /* end if (no other facet yet blocks this facet from view) */
							} /* end if 0 <= t <= s (facet center is "in" this POS pixel) */
						} /* end if 0 <= s <= 1 */
					} /* end j-loop over POS rows */
				} /* end i-loop over POS columns */
			} /* end else of if (i1 > pos->n || i2 < -pos->n || j1 > pos->n || j2 < -pos->n) */
		} /* End if (n[2] > 0.0) */
	} /* end if (f < nf) */
}

__global__ void posvis_outbnd_krnl(struct pos_t **pos,
		int *outbndarr, float4 *ijminmax_overall, int size, int start) {
	/* nfrm_alloc-threaded kernel */
	int posn, f = blockIdx.x * blockDim.x + threadIdx.x + start;
	double xfactor, yfactor;
	if (f <size) {
		if (outbndarr[f]) {
			/* ijminmax_overall.w = imin_overall
			 * ijminmax_overall.x = imax_overall
			 * ijminmax_overall.y = jmin_overall
			 * ijminmax_overall.z = jmax_overall	 */
			posn = pos[f]->n;
			xfactor = (MAX( ijminmax_overall[f].x,  posn) -
					MIN( ijminmax_overall[f].w, -posn) + 1) / (2*posn+1);
			yfactor = (MAX( ijminmax_overall[f].z,  posn) -
					MIN( ijminmax_overall[f].y, -posn) + 1) / (2*posn+1);
			pos[f]->posbnd_logfactor = log(xfactor*yfactor);
		}
	}
}

__host__ int posvis_gpu(
		struct par_t *dpar,
		struct mod_t *dmod,
		struct dat_t *ddat,
		struct pos_t **pos,
		struct vertices_t **verts,
		float3 orbit_offset,
		int *posn,
		int *outbndarr,
		int set,
		int nfrm_alloc,
		int src,
		int nf,
		int body, int comp, unsigned char type, hipStream_t *pv_stream) {

	int f, outbnd, smooth, start;
	dim3 BLK,THD, BLKfrm, THD64;
	hipEvent_t start1, stop1;
	float milliseconds;
	float4 *ijminmax_overall;
	float3 *oa, *usrc;

	/* Launch parameters for the facet_streams kernel */
	THD.x = maxThreadsPerBlock;	THD64.x = 64;
	BLK.x = floor((THD.x - 1 + nf) / THD.x);
	BLKfrm.x = floor((THD64.x - 1 + nfrm_alloc)/THD64.x);

	/* Set up the offset addressing for lightcurves if this is a lightcurve */
	if (type == LGHTCRV)	start = 1;	/* fixes the lightcurve offsets */
	else 					start = 0;

	int oasize = nfrm_alloc*3;
	/* Allocate temporary arrays/structs */
	gpuErrchk(hipMalloc((void**)&ijminmax_overall, sizeof(float4) * nfrm_alloc));
	gpuErrchk(hipMalloc((void**)&oa, sizeof(float3) * oasize));
	gpuErrchk(hipMalloc((void**)&usrc, sizeof(float3) * nfrm_alloc));

	if (TIMING) {
		/* Create the timer events */
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventRecord(start1);
	}
	posvis_init_krnl<<<BLKfrm,THD64>>>(dpar, pos, ijminmax_overall, oa, usrc,
			outbndarr, comp, start, src, nfrm_alloc, set);
	checkErrorAfterKernelLaunch("posvis_init_krnl");

	for (f=start; f<nfrm_alloc; f++) {
		/* Now the main facet kernel */
		posvis_facet_krnl<<<BLK,THD, 0, pv_stream[f-start]>>>(pos, verts,
				ijminmax_overall, orbit_offset, oa, usrc,	src, body, comp,
				nf, f, smooth, outbndarr, set);
	}
	checkErrorAfterKernelLaunch("posvis_facet_krnl");

	/* Take care of any posbnd flags */
	posvis_outbnd_krnl<<<BLKfrm,THD64>>>(pos,
			outbndarr, ijminmax_overall, nfrm_alloc, start);
	checkErrorAfterKernelLaunch("posvis_outbnd_krnl");
	gpuErrchk(hipMemcpyFromSymbol(&outbnd, HIP_SYMBOL(posvis_streams_outbnd), sizeof(int), 0,
			hipMemcpyDeviceToHost));

	if (TIMING) {
		hipEventRecord(stop1);
		hipEventSynchronize(stop1);
		milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start1, stop1);
		printf("%i facets in posvis_cuda_2 in %3.3f ms with %i frames.\n", nf, milliseconds, nfrm_alloc);
	}

//	int n = 200;
//	int npixels = 401*401;
//	f = 0;
//	dbg_print_pos_arrays_full(pos, f, npixels, n);

	/* Free temp arrays, destroy streams and timers, as applicable */

	hipFree(ijminmax_overall);
	hipFree(oa);
	hipFree(usrc);


	if (TIMING) {
		hipEventDestroy(start1);
		hipEventDestroy(stop1);
	}
	return outbnd;
}
