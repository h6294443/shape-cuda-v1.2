#include "hip/hip_runtime.h"
extern "C" {
#include "../shape/head.h"
}
__device__ double dev_distance(double x[3], double y[3])
{
	int i;
	double d;

	d = 0.0;
	for (i=0;i<=2;i++)
		d += (x[i]-y[i])*(x[i]-y[i]);
	return sqrt(d);
}
