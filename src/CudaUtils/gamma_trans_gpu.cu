#include "hip/hip_runtime.h"
/***************************************************************************
                                                              gamma_trans.c

Applies a gamma transformation to a data point.

Modified 2004 Feb 13 by CM:
    Removed obsolete "sdev" argument and the commented-out code
    which used to make use of that argument
***************************************************************************/
extern "C" {
#include "../shape/head.h"
}

__device__ int dev_gamma_trans(double *datum, double gamma)
{
  if ((*datum) <= 0.0)
    return 0;
  (*datum) = pow( (*datum), 1/gamma);
  return 1;
}
