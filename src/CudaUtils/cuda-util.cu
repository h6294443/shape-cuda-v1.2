#include "hip/hip_runtime.h"
extern "C" {
#include "../shape/head.h"
}

void checkErrorAfterKernelLaunch(const char *location) {
	hipError_t cudaStatus;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed in %s: %s\n", location, hipGetErrorString(cudaStatus));
	}
}
void deviceSyncAfterKernelLaunch(const char *location) {
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel in %s.\n", cudaStatus, location);
}
__host__ void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}
void pickGPU(int gpuid) {
	hipSetDeviceFlags(hipDeviceMapHost);
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(gpuid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}

__global__ void zero_fit_overflow_krnl32(struct deldop_t *deldop, int f, int size) {
	/* MAXOVERFLOW^2 - threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int x = offset % MAXOVERFLOW;
	int y = offset / MAXOVERFLOW;

	if (offset < size) {
		deldop->frame[f].fit_overflow32[x][y] = 0.0;
	}
}

__global__ void zero_fit_overflow_krnl64(struct deldop_t *deldop, int f, int size) {
	/* MAXOVERFLOW^2 - threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int x = offset % MAXOVERFLOW;
	int y = offset / MAXOVERFLOW;

	if (offset < size) {
		deldop->frame[f].fit_overflow64[x][y] = 0.0;
	}
}

__host__ void zero_fit_overflow(struct deldop_t *deldop, int f) {
	/* Wrapper function */
	dim3 THD, BLK;
	int threads = 0;
	threads = MAXOVERFLOW*MAXOVERFLOW;
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + threads) / THD.x);

	if (FP64)
		zero_fit_overflow_krnl64<<<BLK,THD>>>(deldop, f, threads);
	else
		zero_fit_overflow_krnl32<<<BLK,THD>>>(deldop, f, threads);
	checkErrorAfterKernelLaunch("zero_fit_overflow_krnl");
}
