extern "C" {
#include "../shape/head.h"
}

void checkErrorAfterKernelLaunch(const char *location) {
	hipError_t cudaStatus;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed in %s: %s\n", location, hipGetErrorString(cudaStatus));
	}
}
void deviceSyncAfterKernelLaunch(const char *location) {
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel in %s.\n", cudaStatus, location);
}
__host__ void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}
void pickGPU(int gpuid) {
	hipSetDeviceFlags(hipDeviceMapHost);
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(gpuid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}
