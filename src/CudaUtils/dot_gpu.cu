#include "hip/hip_runtime.h"
extern "C" {
#include "../shape/head.h"
}
__device__ double dev_dot( double x[3], double y[3])
{
	return x[0]*y[0]+x[1]*y[1]+x[2]*y[2];
}
__device__ double dev_dot2( double x[3], double3 y)
{
	/* This version replaces double y[3] with a double3 *y */
	return x[0]*y.x + x[1]*y.y + x[2]*y.z;
}
__device__ double dev_dot3( float3 x, double3 y)
{
	/* This version replaces double x[3] with a float3 *x */
	return x.x*y.x + x.y*y.y + x.z*y.z;
}
__device__ float dev_dot4(float3 x, float3 y)
{
	/* This version just uses two float3's and returns a float */
	return x.x*y.x + x.y*y.y + x.z*y.z;
}
