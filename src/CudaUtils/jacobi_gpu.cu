#include "hip/hip_runtime.h"
extern "C" {
#include "../shape/head.h"
}

#define ROTATE(a,i,j,k,l) g=a[i][j];h=a[k][l];a[i][j]=g-s*(h+g*tau);\
	a[k][l]=h+s*(g-h*tau);

__device__ void dev_jacobi(double a[3][3], int n, double d[3], double v[3][3], int *nrot)
{
	int j,iq,ip,i;
	double tresh,theta,tau,t,sm,s,h,g,c;

	double b[3];
	double z[3];

	for (ip=0;ip<3;ip++) {
		for (iq=0;iq<3;iq++) v[ip][iq]=0.0;
		v[ip][ip]=1.0;
	}
	for (ip=0;ip<3;ip++) {
		b[ip]=d[ip]=a[ip][ip];
		z[ip]=0.0;
	}
	*nrot=0;
	for (i=0;i<50;i++) {
		sm=0.0;
		for (ip=0;ip<3-1;ip++) {
			for (iq=ip+1;iq<3;iq++)
				sm += fabs(a[ip][iq]);
		}
		if (sm == 0.0)
			return;
		if (i < 4)
			tresh=0.2*sm/(9);
		else
			tresh=0.0;
		for (ip=0;ip<2;ip++) {
			for (iq=ip+1;iq<3;iq++) {
				g=100.0*fabs(a[ip][iq]);
				if (i > 4 && fabs(d[ip])+g == fabs(d[ip])
					&& fabs(d[iq])+g == fabs(d[iq]))
					a[ip][iq]=0.0;
				else if (fabs(a[ip][iq]) > tresh) {
					h=d[iq]-d[ip];
					if (fabs(h)+g == fabs(h))
						t=(a[ip][iq])/h;
					else {
						theta=0.5*h/(a[ip][iq]);
						t=1.0/(fabs(theta)+sqrt(1.0+theta*theta));
						if (theta < 0.0) t = -t;
					}
					c=1.0/sqrt(1+t*t);
					s=t*c;
					tau=s/(1.0+c);
					h=t*a[ip][iq];
					z[ip] -= h;
					z[iq] += h;
					d[ip] -= h;
					d[iq] += h;
					a[ip][iq]=0.0;
					for (j=0;j<ip-1;j++) {
						ROTATE(a,j,ip,j,iq)
					}
					for (j=ip+1;j<=iq-1;j++) {
						ROTATE(a,ip,j,j,iq)
					}
					for (j=iq+1;j<=n;j++) {
						ROTATE(a,ip,j,iq,j)
					}
					for (j=0;j<n;j++) {
						ROTATE(v,j,ip,j,iq)
					}
					++(*nrot);
				}
			}
		}
		for (ip=0;ip<3;ip++) {
			b[ip] += z[ip];
			d[ip]=b[ip];
			z[ip]=0.0;
		}
	}
	printf("Too many iterations in routine JACOBI");
}
#undef ROTATE
